#include "hip/hip_runtime.h"
/* This file is the current iteration of research being done to implement the
swept rule for Partial differential equations in one dimension.  This research
is a collaborative effort between teams at MIT, Oregon State University, and
Purdue University.

Copyright (C) 2015 Kyle Niemeyer, niemeyek@oregonstate.edu AND
Daniel Magee, mageed@oregonstate.edu

This program is free software: you can redistribute it and/or modify
it under the terms of the MIT license.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.

You should have received a copy of the MIT license
along with this program.  If not, see <https://opensource.org/licenses/MIT>.
*/

//COMPILE LINE:
// nvcc -o ./bin/HeatOut Heat1D_SweptShared.cu -gencode arch=compute_35,code=sm_35 -lm -restrict -Xcompiler -fopenmp


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include "myVectorTypes.h" //For clamp.

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>

#ifndef REAL
    #define REAL        float
    #define HALF        0.5f
    #define ONE         1.f
    #define TWO         2.f
#else
    #define HALF        0.5
    #define ONE         1.0
    #define TWO         2.0
#endif

using namespace std;

struct heatConstants{

    REAL fourier;
    REAL fourierTwo;
    int base;
	int ht;
    int idxend;

};

heatConstants hostC;

__constant__ heatConstants gpuC;

const REAL th_diff = 8.418e-5;

const REAL ds = 0.001;

__host__
__device__
REAL initFun(int xnode, REAL ds, REAL lx)
{
    REAL a = ((REAL)xnode*ds);
    return 100.f*a*(ONE-a/lx);
}

//Read in the data from the global right/left variables to the shared temper variable.
__device__
__forceinline__
void
readIn(REAL *temp, const REAL *rights, const REAL *lefts, int td, int gd)
{
	int leftidx = gpuC.ht - (tid>>1) + (((tid>>1) & 1) * gpuC.base) + (tid & 1) - 2;
	int rightidx = gpuC.ht + (tid>>1) + (((tid>>1) & 1) * gpuC.base) + (tid & 1);

	temp[leftidx] = rights[gd];
	temp[rightidx] = lefts[gd];
}

__device__
__forceinline__
void
writeOutRight(REAL *temp, REAL *rights, REAL *lefts, int td, int gd, int bd)
{
	int gdskew = (gd + bd) & disc.idxend;
    int leftidx = (td>>1) + (((tid>>1) & 1) * gpuC.base) + (tid & 1) + 1;
	int rightidx = (bd - 1) + (((tid>>1) & 1) * gpuC.base) + (tid & 1) -  (tid>>1);
	rights[gdskew] = temp[rightidx];
	lefts[gd] = temp[leftidx];
}

__device__
__forceinline__
void
writeOutLeft(REAL *temp, REAL *rights, REAL *lefts, int td, int gd, int bd)
{
	int gdskew = (gd - bd) & disc.idxend;
    int leftidx = (td>>1) + (((tid>>1) & 1) * gpuC.base) + (tid & 1) + 1;
	int rightidx = (bd - 1) + (((tid>>1) & 1) * gpuC.base + (tid & 1) -  (tid>>1);
	rights[gdskew] = temp[rightidx];
	lefts[gd] = temp[leftidx];
}

__host__ __device__
__forceinline__
REAL execFunc(const REAL *heat, int idx[3])
{
    #ifdef __CUDA_ARCH__
    return gpuC.fourier*(heat[idx[0]] + heat[idx[2]]) + gpuC.fourierTwo * heat[idx[1]];
    #else 
    return hostC.fourier*(heat[idx[0]] + heat[idx[2]]) + hostC.fourierTwo * heat[idx[1]];
    #endif
}

__global__
void
classicHeat(const REAL *heat_in, REAL *heat_out)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
    int gidz[3];

    #pragma unroll
    for (int k=-1; k<2; k++) gidz[k+1] = clamp(gid+k, 0, gpuC.lastidx);

    heat_out[gid] =  execFunc(heat_in, gidz);

}

__global__
void
upTriangle(const REAL *IC, REAL *outRight, REAL *outLeft)
{
	extern __shared__ REAL temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tididx = threadIdx.x + 1; //Block Thread ID
	int shft_wr; //Initialize the shift to the written row of temper.
	int shft_rd; //Initialize the shift to the read row (opposite of written)
	int leftidx = (tid>>1) + (((tid>>1) & 1) * blockDim.x) + (tid & 1);
	int rightidx = (blockDim.x - 2) + (((tid>>1) & 1) * blockDim.x) + (tid & 1) -  (tid>>1);
    int gidout = (gid + blockDim.x) & lastidx;

    //Assign the initial values to the first row in temper, each warp (in this
	//case each block) has it's own version of temper shared among its threads.
	temper[tid] = IC[gid];

    __syncthreads();

	//The initial conditions are timslice 0 so start k at 1.

	for (int k = 1; k<(blockDim.x>>1); k++)
	{
		//Bitwise even odd. On even iterations write to first row.
		shft_wr = blockDim.x * (k & 1);
		//On even iterations write to second row (starts at element 32)
		shft_rd = blockDim.x * ((k + 1) & 1);

		//Each iteration the triangle narrows.  When k = 1, 30 points are
		//computed, k = 2, 28 points.
		if (tid < (blockDim.x-k) && tid >= k)
		{
			temper[tid + shft_wr] = execFunc(temper[tidm+shft_rd], temper[tidp+shft_rd], temper[tid+shft_rd]);
		}

		//Make sure the threads are synced
		__syncthreads(); 

	}

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	writeOutRight(temper, outRight, outLeft, threadIdx.x, gid, blockDim.x);

}

// Down triangle is only called at the end when data is passed left.  It's never split.
// It returns IC which is a full 1D result at a certain time.
__global__
void
downTriangle(REAL *IC, const REAL *inRight, const REAL *inLeft)
{
	extern __shared__ REAL temper[];

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tid1 = tid + 1;
	int tid2 = tid + 2;
	int base = blockDim.x + 2;
	int height = base>>1;
	int shft_rd;
	int shft_wr;
	int leftidx = height - (tid>>1) + (((tid>>1) & 1) * base) + (tid & 1) - 2;
	int rightidx = height + (tid>>1) + (((tid>>1) & 1) * base) + (tid & 1);
    int lastidx = ((blockDim.x*gridDim.x)-1);

	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.

	temper[leftidx] = inRight[gid];
	temper[rightidx] = inLeft[gid];

    __syncthreads();
    //k needs to insert the relevant left right values around the computed values
	//every timestep.  Since it grows larger the loop is reversed.

	for (int k = (height-1); k>1; k--)
	{
		// This tells you if the current row is the first or second.
		shft_wr = base * ((k+1) & 1);
		// Read and write are opposite rows.
		shft_rd = base * (k & 1);

		if (tid1 < (base-k) && tid1 >= k)
		{
			temper[tid1 + shft_wr] = execFunc(temper[tid+shft_rd], temper[tid2+shft_rd], temper[tid1+shft_rd]);
		}
        __syncthreads();
	}

    if (gid == 0)
    {
        temper[tid] = execFunc(temper[tid2+base], temper[tid2+base], temper[tid1+base]);
    }
    else if (gid == lastidx)
    {
        temper[tid] = execFunc(temper[tid+base], temper[tid+base], temper[tid1+base]);
    }
    else
    {
        temper[tid] = execFunc(temper[tid+base], temper[tid2+base], temper[tid1+base]);
    }
    __syncthreads();

    IC[gid] = temper[tid];
}

//Full refers to whether or not there is a node run on the CPU.
__global__
void
wholeDiamond(const REAL *inRight, const REAL *inLeft, REAL *outRight, REAL *outLeft, const bool split)
{
    extern __shared__ REAL temper[];

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tididx = threadIdx.x + 1;

	int tid_top[3], tid_bottom[3];

    int k = gpuC.ht;
	tid_bottom[1] = tididx;
    tid_top[1] = tididx + gpuC.base;

	// Initialize temper.
    if (split)
    {
        gid += blockDim.x;
        tid_bottom[0] = tididx - 1;
        tid_bottom[2] = tididx + 1;
    }
    else
    {
        tid_bottom[0] = (gid == 0) ? (tididx + 1 : tididx - 1);
        tid_bottom[2] = (gid == gpuC.lastidx) ? (tididx - 1 : tididx + 1);
    }

    readIn(temper, inRight, inLeft, threadIdx.x, gid);
 
    tid_top[0] = tid_bottom[0] + gpuC.base;
    tid_top[2] = tid_bottom[2] + gpuC.base;

    __syncthreads();

	while (k<blockdim.x)
	{
        if (tididx > (blockDim.x-k) && tididx <= k)
        {
			temper[tid_top[1]] = execFunc(temper, tid_bottom);
		}
        k++
        __syncthreads();

        if (tididx > (blockDim.x-k) && tididx <= k)
		{
			temper[tididx] = execFunc(temper, tid_top);
		}
        k++
        __syncthreads();      
	}

    //-------------------TOP PART------------------------------------------
    k -= 2 

	while (k<0)
	{
        if (tididx > (blockDim.x-k) && tididx <= k)
        {
			temper[tid_top[1]] = execFunc(temper, tid_bottom);
		}
        k--
        __syncthreads();

        if (tididx > (blockDim.x-k) && tididx <= k)
		{
			temper[tididx] = execFunc(temper, tid_top);
		}
        k--
        __syncthreads();      
	}

	if (split)
	{
		writeOutLeft(temper, outRight, outLeft, threadIdx.x, gid, blockDim.x);
	}
	else
	{
		writeOutRight(temper, outRight, outLeft, threadIdx.x, gid, blockDim.x);
	}
}

//Split one is always first.  Passing left like the downTriangle.  downTriangle
//should be rewritten so it isn't split.  Only write on a non split pass.
//Needs to be changed to not justify temper between triangles.
__global__
void
splitDiamond(const REAL *inRight, const REAL *inLeft, REAL *outRight, REAL *outLeft)
{
    extern __shared__ REAL temper[];

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
    int lastidx = ((blockDim.x*gridDim.x)-1);
	int base = blockDim.x + 2;
	int height = base>>1;
    int ht1 = height-1;
	int shft_rd;
	int shft_wr;
	int leftidx = height - (tid>>1) + (((tid>>1) & 1) * base) + (tid & 1) - 2;
	int rightidx = height + (tid>>1) + (((tid>>1) & 1) * base) + (tid & 1);
    int tid1 = tid + 1;
    int tid2 = ((gid == ht1) ? tid : tid+2);
    int tid0 = ((gid == height) ? tid+2 : tid);
    int gidout = (gid - blockDim.x) & lastidx;

	// Initialize temper.

    temper[leftidx] = inRight[gid];
	temper[rightidx] = inLeft[gid];

    //Wind it up!

    __syncthreads();

    for (int k = ht1; k>0; k--)
    {
        // This tells you if the current row is the first or second.
        shft_wr = base * ((k+1) & 1);
        // Read and write are opposite rows.
        shft_rd = base * (k & 1);

        if (tid1 < (base-k) && tid1 >= k)
        {
            temper[tid1 + shft_wr] = execFunc(temper[tid0+shft_rd], temper[tid2+shft_rd], temper[tid1+shft_rd]);
        }

        __syncthreads();
    }

    REAL trade = temper[tid1];
    __syncthreads();
    temper[tid] = trade;
    __syncthreads();

    //-------------------TOP PART------------------------------------------
    leftidx = (tid>>1) + (((tid>>1) & 1) * blockDim.x) + (tid & 1);
    rightidx = (blockDim.x - 2) + (((tid>>1) & 1) * blockDim.x) + (tid & 1) -  (tid>>1);

    tid0--;
    tid2--;

	for (int k = 1; k<ht1; k++)
	{
		//Bitwise even odd. On even iterations write to first row.
		shft_wr = blockDim.x * (k & 1);
		//On even iterations write to second row (starts at element 32)
		shft_rd = blockDim.x * ((k + 1) & 1);


        if (tid < (blockDim.x-k) && tid >= k)
        {
            temper[tid + shft_wr] = execFunc(temper[tid0+shft_rd], temper[tid2+shft_rd], temper[tid+shft_rd]);
        }

		//Make sure the threads are synced
		__syncthreads();
    }

	outRight[gid] = temper[rightidx];
	outLeft[gidout] = temper[leftidx];
}

__host__
void
CPU_diamond(REAL *temper, int tpb)
{
    int bck, fwd, shft_rd, shft_wr;
    int base = tpb + 2;
    int ht = tpb/2;

    //Splitting it is the whole point!
    for (int k = ht; k>0; k--)
    {
        // This tells you if the current row is the first or second.
        shft_wr = base * ((k+1) & 1);
        // Read and write are opposite rows.
        shft_rd = base * (k & 1);

        for(int n = k; n<(base-k); n++)
        {
            bck = n - 1;
            fwd = n + 1;
            //Double trailing index.
            if(n == ht)
            {
                temper[n + shft_wr] = execFuncHost(temper[bck+shft_rd], temper[bck+shft_rd], temper[n+shft_rd]);
            }
            //Double leading index.
            else if(n == ht+1)
            {
                temper[n + shft_wr] = execFuncHost(temper[fwd+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
            }
            else
            {
                temper[n + shft_wr] = execFuncHost(temper[bck+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
            }
        }
    }

    for (int k = 0; k<tpb; k++) temper[k] = temper[k+1];
    //Top part.
    ht--;
    for (int k = 1; k<ht; k++)
    {
        // This tells you if the current row is the first or second.
        shft_wr = tpb * (k & 1);
        // Read and write are opposite rows.
        shft_rd = tpb * ((k+1) & 1);

        for(int n = k; n<(tpb-k); n++)
        {
            bck = n - 1;
            fwd = n + 1;
            //Double trailing index.
            if(n == ht)
            {
                temper[n + shft_wr] = execFuncHost(temper[bck+shft_rd], temper[bck+shft_rd], temper[n+shft_rd]);
            }
            //Double leading index.
            else if(n == ht+1)
            {
                temper[n + shft_wr] = execFuncHost(temper[fwd+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
            }
            else
            {
                temper[n + shft_wr] = execFuncHost(temper[bck+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
            }
        }
    }
}

//Classic Discretization wrapper.
double
classicWrapper(const int bks, int tpb, const int dv, const double dt, const double t_end,
    REAL *IC, REAL *T_f, const double freq, ofstream &fwr)
{
    REAL *dheat_in, *dheat_out;

    hipMalloc((void **)&dheat_in, sizeof(REAL)*dv);
    hipMalloc((void **)&dheat_out, sizeof(REAL)*dv);

    // Copy the initial conditions to the device array.
    hipMemcpy(dheat_in,IC, sizeof(REAL)*dv, hipMemcpyHostToDevice);

    const double t_fullstep = dt+dt;
    double twrite = freq;
    classicHeat <<< bks,tpb >>> (dheat_in, dheat_out);
    classicHeat <<< bks,tpb >>> (dheat_out, dheat_in);

    double t_eq = t_fullstep;

    while (t_eq < t_end)
    {
        classicHeat <<< bks,tpb >>> (dheat_in, dheat_out);
        classicHeat <<< bks,tpb >>> (dheat_out, dheat_in);
        t_eq += t_fullstep;

        if (t_eq > twrite)
        {
            hipMemcpy(T_f, dheat_in, sizeof(REAL)*dv, hipMemcpyDeviceToHost);
            fwr << " Temperature " << t_eq << " ";

            for (int k = 0; k<dv; k++)   fwr << T_f[k] << " ";

            fwr << endl;

            t_eq += t_fullstep;

            twrite += freq;
        }
    }

    cout << t_eq << " " << t_end << " " << t_fullstep << endl;

    hipMemcpy(T_f, dheat_in, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

    hipFree(dheat_in);
    hipFree(dheat_out);

    return t_eq;

}

//The Swept Rule wrapper.
double
sweptWrapper(const int bks, int tpb, const int dv, const double dt, const double t_end, const int cpu,
    REAL *IC, REAL *T_f, const double freq, ofstream &fwr)
{
    const int base = (tpb + 2);
    const int ht = base/2;
    const size_t smem = (base*2)*sizeof(REAL);
    const int cpuLoc = dv-tpb;

    int indices[4][tpb];
    for (int k = 0; k<tpb; k++)
    {
        indices[0][k] = ht - k/2 + ((k/2 & 1) * base) + (k & 1) - 2; //left
        indices[1][k] = ht + k/2 + ((k/2 & 1) * base) + (k & 1); //right

        indices[2][k] = k/2 + ((k/2 & 1) * tpb) + (k & 1); //left
        indices[3][k] = (tpb - 2) + ((k/2 & 1) * tpb) + (k & 1) -  k/2; //right
    }

	REAL *d_IC, *d0_right, *d0_left, *d2_right, *d2_left;

	hipMalloc((void **)&d_IC, sizeof(REAL)*dv);
	hipMalloc((void **)&d0_right, sizeof(REAL)*dv);
	hipMalloc((void **)&d0_left, sizeof(REAL)*dv);
	hipMalloc((void **)&d2_right, sizeof(REAL)*dv);
	hipMalloc((void **)&d2_left, sizeof(REAL)*dv);

	// Copy the initial conditions to the device array.
	hipMemcpy(d_IC,IC,sizeof(REAL)*dv,hipMemcpyHostToDevice);
	// Start the counter and start the clock.
	const double t_fullstep = dt*(double)tpb;

	upTriangle <<< bks,tpb,smem >>>(d_IC,d0_right,d0_left);

    double t_eq;
    double twrite = freq;

	// Call the kernels until you reach the iteration limit.

    if (cpu)
    {
        REAL *h_right, *h_left;
        REAL *tmpr = (REAL*)malloc(smem);
        hipHostAlloc((void **) &h_right, tpb*sizeof(REAL), hipHostMallocDefault);
        hipHostAlloc((void **) &h_left, tpb*sizeof(REAL), hipHostMallocDefault);

        t_eq = t_fullstep;

        hipStream_t st1, st2, st3;
        hipStreamCreate(&st1);
        hipStreamCreate(&st2);
        hipStreamCreate(&st3);

        //Split Diamond Begin------

        wholeDiamond <<< bks-1, tpb, smem, st1 >>>(d0_right, d0_left, d2_right, d2_left, false);

        hipMemcpyAsync(h_left, d0_left, tpb*sizeof(REAL), hipMemcpyDeviceToHost, st2);
        hipMemcpyAsync(h_right, d0_right, tpb*sizeof(REAL), hipMemcpyDeviceToHost, st3);

        hipStreamSynchronize(st2);
        hipStreamSynchronize(st3);

        for (int k = 0; k<tpb; k++) 
        {		
            tmpr[indices[0][k]] = h_right[k];		
            tmpr[indices[1][k]] = h_left[k];			
        }

        CPU_diamond(tmpr, tpb);

        for (int k = 0; k<tpb; k++) 
        {		
            h_left[k] = tmpr[indices[2][k]];		
            h_right[k] = tmpr[indices[3][k]];		
        }
        
        hipMemcpyAsync(d2_right, h_right, tpb*sizeof(REAL), hipMemcpyHostToDevice,st2);
        hipMemcpyAsync(d2_left+cpuLoc, h_left, tpb*sizeof(REAL), hipMemcpyHostToDevice,st3);

        //Split Diamond End------

    	while(t_eq < t_end)
    	{

            wholeDiamond <<< bks,tpb,smem >>>(d2_right,d2_left,d0_right,d0_left,true);

            //Split Diamond Begin------

            wholeDiamond <<< bks-1, tpb, smem, st1 >>>(d0_right, d0_left, d2_right, d2_left, false);

            hipMemcpyAsync(h_left, d0_left, tpb*sizeof(REAL), hipMemcpyDeviceToHost, st2);
            hipMemcpyAsync(h_right, d0_right, tpb*sizeof(REAL), hipMemcpyDeviceToHost, st3);

            hipStreamSynchronize(st2);
            hipStreamSynchronize(st3);

            for (int k = 0; k<tpb; k++) 
            {		
                tmpr[indices[0][k]] = h_right[k];		
                tmpr[indices[1][k]] = h_left[k];		
            }

            CPU_diamond(tmpr, tpb);

            for (int k = 0; k<tpb; k++) 
            {		
                h_left[k] = tmpr[indices[2][k]];		
                h_right[k] = tmpr[indices[3][k]];		
            }

            hipMemcpyAsync(d2_right, h_right, tpb*sizeof(REAL), hipMemcpyHostToDevice,st2);
            hipMemcpyAsync(d2_left+cpuLoc, h_left, tpb*sizeof(REAL), hipMemcpyHostToDevice,st3);

            //Split Diamond End------

		    //So it always ends on a left pass since the down triangle is a right pass.

		    t_eq += t_fullstep;

            if (t_eq > twrite)
    		{
    			downTriangle <<< bks,tpb,smem >>>(d_IC,d2_right,d2_left);

    			hipMemcpy(T_f, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

    			fwr << "Temperature " << t_eq << " ";

    			for (int k = 0; k<dv; k++)	fwr << T_f[k] << " ";

    			fwr << endl;

                upTriangle <<< bks,tpb,smem >>>(d_IC,d0_right,d0_left);

    			splitDiamond <<< bks,tpb,smem >>>(d0_right,d0_left,d2_right,d2_left);

                t_eq += t_fullstep;

    			twrite += freq;
    		}
        }
        hipHostFree(h_right);
        hipHostFree(h_left);
        free(tmpr);
	}
    else
    {
        splitDiamond <<< bks,tpb,smem >>>(d0_right,d0_left,d2_right,d2_left);
        t_eq = t_fullstep;

        while(t_eq < t_end)
        {
            wholeDiamond <<< bks,tpb,smem >>>(d2_right,d2_left,d0_right,d0_left,true);

            splitDiamond <<< bks,tpb,smem >>>(d0_right,d0_left,d2_right,d2_left);

            //So it always ends on a left pass since the down triangle is a right pass.
            t_eq += t_fullstep;

            if (t_eq > twrite)
    		{
    			downTriangle <<< bks,tpb,smem >>>(d_IC,d2_right,d2_left);

    			hipMemcpy(T_f, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);
    			fwr << "Temperature " << t_eq << " ";

    			for (int k = 0; k<dv; k++)	fwr << T_f[k] << " ";

    			fwr << endl;

    			upTriangle <<< bks,tpb,smem >>>(d_IC,d0_right,d0_left);

    			splitDiamond <<< bks,tpb,smem >>>(d0_right,d0_left,d2_right,d2_left);

                t_eq += t_fullstep;

    			twrite += freq;
    		}
        }
    }

	downTriangle <<< bks,tpb,smem >>>(d_IC,d2_right,d2_left);

    cout << t_eq << " " << t_end << " " << t_fullstep << endl;

	hipMemcpy(T_f, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

	hipFree(d_IC);
	hipFree(d0_right);
	hipFree(d0_left);
    hipFree(d2_right);
	hipFree(d2_left);

    return t_eq;
}

int main(int argc, char *argv[])
{
    //That is there are less than 8 arguments.

    if (argc < 9)
    {
    	cout << "The Program takes 9 inputs, #Divisions, #Threads/block, deltat, finish time, output frequency..." << endl;
        cout << "Classic/Swept, CPU sharing Y/N, Variable Output File, Timing Output File (optional)" << endl;
    	exit(-1);
    }
    cout.precision(10);
	// Choose the GPGPU.  This is device 0 in my machine which has 2 devices.
	hipSetDevice(0);
    if (sizeof(REAL)>6) hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

    int dv = atoi(argv[1]); //Number of spatial points
	const int tpb = atoi(argv[2]); //Threads per Blocks
    const double dt =  atof(argv[3]);
	const double tf = atof(argv[4]) - 0.5*dt; //Finish time
    const double freq = atof(argv[5]);
    const int scheme = atoi(argv[6]); //1 for Swept 0 for classic
    const int share = atoi(argv[7]);
	const int bks = dv/tpb; //The number of blocks
    const double lx = ds * ((double)dv - 1.0);
    fou = th_diff*dt/(ds*ds);  //Fourier number
    char const *prec;
    prec = (sizeof(REAL)<6) ? "Single": "Double";

    cout << "Heat --- #Blocks: " << bks << " | Length: " << lx << " | Precision: " << prec << " | Fo: " << fou << endl;

	//dv and tpb must be powers of two.  dv must be larger than tpb and divisible by
	//tpb.

	if ((dv & (tpb-1) !=0) || (tpb&31) != 0)
    {
        cout << "INVALID NUMERIC INPUT!! "<< endl;
        cout << "2nd ARGUMENT MUST BE A POWER OF TWO >= 32 AND FIRST ARGUMENT MUST BE DIVISIBLE BY SECOND" << endl;
        exit(-1);
    }

	// Initialize arrays.
    REAL *IC, *T_final;

	hipHostAlloc((void **) &IC, dv*sizeof(REAL), hipHostMallocDefault);
	hipHostAlloc((void **) &T_final, dv*sizeof(REAL), hipHostMallocDefault);

    // IC = (REAL *) malloc(dv*sizeof(REAL));
    // T_final = (REAL *) malloc(dv*sizeof(REAL));

	for (int k = 0; k<dv; k++)
	{
		IC[k] = initFun(k, ds, lx);
	}

	// Call out the file before the loop and write out the initial condition.
	ofstream fwr;
	fwr.open(argv[8], ios::trunc);
    fwr.precision(10);

	// Write out x length and then delta x and then delta t.
	// First item of each line is timestamp.
	fwr << lx << " " << dv << " " << ds << " " << endl << "Temperature " << 0 << " ";

	for (int k = 0; k<dv; k++)
	{
		fwr << IC[k] << " ";
	}
	fwr << endl;

    //Transfer data to GPU.
	// This puts the Fourier number in constant memory.
	hipMemcpyToSymbol(HIP_SYMBOL(fo),&fou,sizeof(REAL));

	// Start the counter and start the clock.
	hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

    // Call the kernels until you reach the iteration limit.
	double tfm;
    if (scheme)
    {
        cout << "Swept" << endl;
        tfm = sweptWrapper(bks, tpb, dv, dt, tf, share, IC, T_final, freq, fwr);
    }
    else
    {
        cout << "Classic" << endl;
        tfm = classicWrapper(bks, tpb, dv, dt, tf, IC, T_final, freq, fwr);
    }

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime( &timed, start, stop);

	timed *= 1.e3;

    double n_timesteps = tfm/dt;

    double per_ts = timed/n_timesteps;

    cout << n_timesteps << " timesteps" << endl;
	cout << "Averaged " << per_ts << " microseconds (us) per timestep" << endl;

    if (argc>8)
    {
        ofstream ftime;
        ftime.open(argv[9],ios::app);
    	ftime << dv << "\t" << tpb << "\t" << per_ts << endl;
    	ftime.close();
    }

	fwr << "Temperature " << tfm << " ";
	for (int k = 0; k<dv; k++)	fwr << T_final[k] << " ";

	fwr.close();

	// Free the memory and reset the device.

	hipEventDestroy( start );
	hipEventDestroy( stop );
    hipDeviceReset();
    hipHostFree(IC);
    hipHostFree(T_final);

	return 0;
}
