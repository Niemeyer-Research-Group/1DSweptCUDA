/* This file is the current iteration of research being done to implement the
swept rule for Partial differential equations in one dimension.  This research
is a collaborative effort between teams at MIT, Oregon State University, and
Purdue University.

Copyright (C) 2015 Kyle Niemeyer, niemeyek@oregonstate.edu AND
Daniel Magee, mageed@oregonstate.edu

This program is free software: you can redistribute it and/or modify
it under the terms of the MIT license.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.

You should have received a copy of the MIT license
along with this program.  If not, see <https://opensource.org/licenses/MIT>.
*/

//COMPILE LINE!
// nvcc -o ./bin/KSRegOut KS1D_SweptRegister.cu -gencode arch=compute_35,code=sm_35 -lm -restrict --ptxas-options=-v

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <hip/device_functions.h>
#include <hip/hip_vector_types.h>

#include <iostream>
#include <ostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>

#ifndef REAL
    #define REAL        float
    #define TWO         2.f
	#define FOUR        4.f
	#define SIX			6.f
#else
    #define TWO         2.0
	#define FOUR        4.0
	#define SIX			6.0
#endif

#define BASE            36
#define HEIGHT          18
#define WARPSIZE        32
#define TWOBASE         72

#ifndef WPB
    #define WPB             8
#endif

using namespace std;

const REAL dx = 0.5;

struct discConstants{

	REAL dx_i4; // 1/(4*dx)
	REAL dx2_i; // 1/(dx^2)
	REAL dx4_i; // 1/(dx^4)
	REAL dt; // dt
	REAL dt_half; // dt/2
};

__constant__ discConstants disc;

__host__
REAL initFun(REAL xnode)
{
	return TWO * cos(19.0*xnode*M_PI/128.0);
}

__device__
__forceinline__
REAL fourthDer(REAL tfarLeft, REAL tLeft, REAL tCenter, REAL tRight, REAL tfarRight)
{
	return disc.dx4_i * (tfarLeft - FOUR*tLeft + SIX*tCenter - FOUR*tRight + tfarRight);
}

__device__
__forceinline__
REAL secondDer(REAL tLeft, REAL tRight, REAL tCenter)
{
	return disc.dx2_i * (tLeft + tRight - TWO*tCenter);
}

__device__
__forceinline__
REAL convect(REAL tLeft, REAL tRight)
{
	return disc.dx_i4 * (tRight*tRight - tLeft*tLeft);
}

__device__
__forceinline__
REAL stutterStep(REAL tfarLeft, REAL tLeft, REAL tCenter, REAL tRight, REAL tfarRight)
{
	return tCenter - disc.dt_half * (convect(tLeft, tRight) + secondDer(tLeft, tRight, tCenter) +
		fourthDer(tfarLeft, tLeft, tCenter, tRight, tfarRight));
}

__device__
__forceinline__
REAL finalStep(REAL tfarLeft, REAL tLeft, REAL tCenter, REAL tRight, REAL tfarRight)
{
	return (-disc.dt * (convect(tLeft, tRight) + secondDer(tLeft, tRight, tCenter) +
			fourthDer(tfarLeft, tLeft, tCenter, tRight, tfarRight)));
}

__global__
void
swapKernel(const REAL *passing_side, REAL *bin, int direction)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
    int lastidx = ((blockDim.x*gridDim.x)-1);
    int gidout = (gid + direction*blockDim.x) & lastidx;

    bin[gidout] = passing_side[gid];

}

__global__
void
upTriangle(const REAL *IC, REAL *right, REAL *left)
{
	__shared__ REAL temper[WPB][TWOBASE];

	int gid = blockDim.x*blockIdx.x*blockDim.y + threadIdx.y*blockDim.x +
        threadIdx.x; //Global Thread ID

	int wid = threadIdx.x; //Thread id in warp.
    int wtag = threadIdx.y; //Warp id in block.
    int widx = wid + 2;
    int widTop = widx+BASE;

	int leftidx = (((wid>>2) & 1) * BASE) + ((wid>>2)<<1) + (wid & 3) + 2;
	int rightidx = 30 + (((wid>>2) & 1) * BASE) + (wid & 3) - ((wid>>2)<<1);

    REAL vel[2];
    //Assign the initial values to the first row in temper, each block
    //has it's own version of temper shared among its threads.
	vel[0] = IC[gid];

	__syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    temper[wtag][widx] = vel[0];
    temper[wtag][widTop] = vel[1];

	__syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid > 3 && wid < 28) temper[wtag][widx] = vel[0];

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid > 5 && wid < 26) temper[wtag][widTop] = vel[1];

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid > 7 && wid < 24) temper[wtag][widx] = vel[0];

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid > 9 && wid < 22) temper[wtag][widTop] = vel[1];

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid > 11 && wid < 20) temper[wtag][widx] = vel[0];

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid > 13 && wid < 18) temper[wtag][widTop] = vel[1];

	//Make sure the threads are synced
	__syncthreads();

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[wtag][rightidx];
	left[gid] = temper[wtag][leftidx];
}

__global__
void
downTriangle(REAL *IC, const REAL *right, const REAL *left)
{
    __shared__ REAL temper[WPB][TWOBASE];

    int gid = blockDim.x*blockIdx.x*blockDim.y + threadIdx.y*blockDim.x +
        threadIdx.x; //Global Thread ID

	int wid = threadIdx.x; //Thread id in warp.
    int wtag = threadIdx.y; //Warp id in block.
    int widx = wid + 2;
    int widTop = wid+BASE;

    int leftidx = HEIGHT + (((wid>>2) & 1) * BASE) + (wid & 3) - (4 + ((wid>>2) << 1));
	int rightidx = HEIGHT + (((wid>>2) & 1) * BASE) + ((wid>>2)<<1) + (wid & 3);

	temper[wtag][leftidx] = right[gid];
	temper[wtag][rightidx] = left[gid];
    REAL vel[2];

    //stutter first
    vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 14 || wid > 17) vel[1] = temper[wtag][widTop];

    __syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid < 12 || wid > 19) vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 10 || wid > 21) vel[1] = temper[wtag][widTop];

    __syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid < 8 || wid > 23) vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 6 || wid > 25) vel[1] = temper[wtag][widTop];

    __syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid < 4 || wid > 27) vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 1 || wid > 30) temper[wtag][widTop] = vel[1];

    __syncthreads();

    //This is where to do it with shared mem.
    vel[0] += finalStep(temper[wtag][widTop-2],temper[wtag][widTop-1],temper[wtag][widTop],
        temper[wtag][widTop+1],temper[wtag][widTop+2]);

    IC[gid] = vel[0];
}

__global__
void
wholeDiamond(REAL *right, REAL *left)
{
    __shared__ REAL temper[WPB][TWOBASE];

    int gid = blockDim.x*blockIdx.x*blockDim.y + threadIdx.y*blockDim.x +
        threadIdx.x; //Global Thread ID

	int wid = threadIdx.x; //Thread id in warp.
    int wtag = threadIdx.y; //Warp id in block.
    int widx = wid+2;
    int widTop = widx+BASE;

    int leftidx = HEIGHT + (((wid>>2) & 1) * BASE) + (wid & 3) - (4 + ((wid>>2) << 1));
	int rightidx = HEIGHT + (((wid>>2) & 1) * BASE) + ((wid>>2)<<1) + (wid & 3);

	temper[wtag][leftidx] = right[gid];
	temper[wtag][rightidx] = left[gid];
    REAL vel[2];

    //stutter first
    vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 14 || wid > 17) vel[1] = temper[wtag][widTop];

    __syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid < 12 || wid > 19) vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 10 || wid > 21) vel[1] = temper[wtag][widTop];

    __syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid < 8 || wid > 23) vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 6 || wid > 25) vel[1] = temper[wtag][widTop];

    __syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid < 4 || wid > 27) vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 1 || wid > 30) temper[wtag][widTop] = vel[1];

    __syncthreads();

    //This is where to do it with shared mem.
    vel[0] += finalStep(temper[wtag][widTop-2],temper[wtag][widTop-1],temper[wtag][widTop],
        temper[wtag][widTop+1],temper[wtag][widTop+2]);

    leftidx = (((wid>>2) & 1) * BASE) + ((wid>>2)<<1) + (wid & 3) + 2;
    rightidx = 30 + (((wid>>2) & 1) * BASE) + (wid & 3) - ((wid>>2)<<1);

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    temper[wtag][widx] = vel[0];
    temper[wtag][widTop] = vel[1];

	__syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid > 3 && wid < 28) temper[wtag][widx] = vel[0];

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid > 5 && wid < 26) temper[wtag][widTop] = vel[1];

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid > 7 && wid < 24) temper[wtag][widx] = vel[0];

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid > 9 && wid < 22) temper[wtag][widTop] = vel[1];

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid > 11 && wid < 20) temper[wtag][widx] = vel[0];

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid > 13 && wid < 18) temper[wtag][widTop] = vel[1];

	//Make sure the threads are synced
	__syncthreads();

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[wtag][rightidx];
	left[gid] = temper[wtag][leftidx];
}

//The host routine.
double
sweptWrapper(const int bks, const int dv, REAL dt, const REAL t_end,
	REAL *IC, REAL *T_f, const REAL freq, ofstream &fwr)
{

	REAL *d_IC, *d_right, *d_left, *d_bin;
	hipMalloc((void **)&d_IC, sizeof(REAL)*dv);
	hipMalloc((void **)&d_right, sizeof(REAL)*dv);
	hipMalloc((void **)&d_left, sizeof(REAL)*dv);
	hipMalloc((void **)&d_bin, sizeof(REAL)*dv);

	// Copy the initial conditions to the device array.
	hipMemcpy(d_IC,IC,sizeof(REAL)*dv,hipMemcpyHostToDevice);

    dim3 tpb(WARPSIZE, WPB);
    cout << tpb.x << " " << tpb.y << " " << tpb.z << endl;

    const int tpbSwap = WARPSIZE*WPB;
	//Start the counter and start the clock.
	//
	//Every other step is a full timestep and each cycle is half tpb steps.
	const double t_fullstep = 0.25 * dt * (double)WARPSIZE;
	double twrite = freq;

	upTriangle <<< bks,tpb >>> (d_IC,d_right,d_left);

	swapKernel <<< bks,tpbSwap >>> (d_right, d_bin, 1);
	swapKernel <<< bks,tpbSwap >>> (d_bin, d_right, 0);

	//Split
	wholeDiamond <<< bks,tpb >>> (d_right,d_left);

	swapKernel <<< bks,tpbSwap >>> (d_left, d_bin, -1);
	swapKernel <<< bks,tpbSwap >>> (d_bin, d_left, 0);

	double t_eq = t_fullstep;

	// Call the kernels until you reach the iteration limit.
	while(t_eq < t_end)
	{

		wholeDiamond <<< bks,tpb >>> (d_right,d_left);

		swapKernel <<< bks,tpbSwap >>> (d_right, d_bin, 1);
		swapKernel <<< bks,tpbSwap >>> (d_bin, d_right, 0);

		//So it always ends on a left pass since the down triangle is a right pass.

		//Split
		wholeDiamond <<< bks,tpb >>> (d_right,d_left);

		swapKernel <<< bks,tpbSwap >>> (d_left, d_bin, -1);
		swapKernel <<< bks,tpbSwap >>> (d_bin, d_left, 0);

		t_eq += t_fullstep;

	 	if (t_eq > twrite)
		{
			downTriangle <<< bks,tpb >>>(d_IC,d_right,d_left);

			hipMemcpy(T_f, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

			fwr << " Velocity " << t_eq << " ";

			for (int k = 0; k<dv; k++)	fwr << T_f[k] << " ";

			fwr << endl;

			upTriangle <<< bks,tpb >>>(d_IC,d_right,d_left);

			swapKernel <<< bks,tpbSwap >>> (d_right, d_bin, 1);
			swapKernel <<< bks,tpbSwap >>> (d_bin, d_right, 0);

			//Split
			wholeDiamond <<< bks,tpb >>>(d_right,d_left);

			swapKernel <<< bks,tpbSwap >>> (d_left, d_bin, -1);
			swapKernel <<< bks,tpbSwap >>> (d_bin, d_left, 0);

			t_eq += t_fullstep;

			twrite += freq;
		}

	}

	downTriangle <<< bks,tpb >>>(d_IC,d_right,d_left);

	hipMemcpy(T_f, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

	hipFree(d_IC);
	hipFree(d_right);
	hipFree(d_left);
	hipFree(d_bin);

	return t_eq;

}

int main( int argc, char *argv[])
{

	if (argc < 6)
	{
		cout << "The Program takes 9 inputs, #Divisions, deltat, finish time, output frequency..." << endl;
        cout << "Variable Output File, Timing Output File (optional)" << endl;
		exit(-1);
	}

	// Choose the GPGPU.  This is device 0 in my machine which has 2 devices.
	hipSetDevice(0);
	if (sizeof(REAL)>6) hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

	const int dv = atoi(argv[1]); //Number of spatial points
    const REAL dt = atof(argv[2]); //delta T timestep
	const float tf = atof(argv[3]); //Finish time
    const float freq = atof(argv[4]); //Output frequency
    // const int tst = atoi(argv[7]); CPU/GPU share
    const int bks = dv/(WARPSIZE*WPB); //The number of blocks
	const float lx = dv*dx;
	char const *prec;
	prec = (sizeof(REAL)<6) ? "Single": "Double";

	cout << "KS --- #Blocks: " << bks << " | Length: " << lx << " | Precision: " << prec << " | dt/dx: " << dt/dx << " argc: " << argc << endl;

	//Conditions for main input.  Unit testing kinda.
	//dv and tpb must be powers of two.  dv must be larger than tpb and divisible by
	//tpb.

	discConstants dsc = {
		1.0/(FOUR*dx),
		1.0/(dx*dx),
		1.0/(dx*dx*dx*dx),
		dt,
		dt*0.5
	};

	// Initialize arrays.
    REAL *IC, *T_final;

	hipHostAlloc((void **) &IC, dv*sizeof(REAL), hipHostMallocDefault);
	hipHostAlloc((void **) &T_final, dv*sizeof(REAL), hipHostMallocDefault);

    // IC = (REAL *) malloc(dv*sizeof(REAL));
    // T_final = (REAL *) malloc(dv*sizeof(REAL));

	// Inital condition
	for (int k = 0; k<dv; k++)
	{
		IC[k] = initFun((float)k*dx);
	}

	// Call out the file before the loop and write out the initial condition.
	ofstream fwr;
	fwr.open(argv[5],ios::trunc);

	// Write out x length and then delta x and then delta t.
	// First item of each line is timestamp.
	fwr << lx << " " << dv << " " << dx << " " << endl << " Velocity " << 0 << " ";

	for (int k = 0; k<dv; k++) fwr << IC[k] << " ";

	fwr << endl;
	// Transfer data to GPU.

	// This puts the constant part of the equation in constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(disc),&dsc,sizeof(dsc));

	// Start the counter and start the clock.
	hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

	cout << "Swept" << endl;
	double tfm = sweptWrapper(bks, dv, dsc.dt, tf, IC, T_final, freq, fwr);

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime( &timed, start, stop);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }


	timed *= 1.e3;

	double n_timesteps = tfm/dt;

    double per_ts = timed/n_timesteps;

    cout << n_timesteps << " timesteps" << endl;
	cout << "Averaged " << per_ts << " microseconds (us) per timestep" << endl;

    if (argc>5)
    {
        ofstream ftime;
        ftime.open(argv[6],ios::app);
    	ftime << dv << "\t" << WPB*WARPSIZE << "\t" << per_ts << endl;
    	ftime.close();
    }

	fwr << " Velocity " << tfm << " ";
	for (int k = 0; k<dv; k++) fwr << T_final[k] << " ";

    fwr << endl;

	fwr.close();

	hipDeviceSynchronize();
	// Free the memory and reset the device.

	hipEventDestroy( start );
	hipEventDestroy( stop );

	hipDeviceReset();

	hipHostFree(IC);
    hipHostFree(T_final);
	// free(IC);
	// free(T_final);

	return 0;

}
