#include "hip/hip_runtime.h"
/* This file is the current iteration of research being done to implement the
swept rule for Partial differential equations in one dimension.  This research
is a collaborative effort between teams at MIT, Oregon State University, and
Purdue University.

Copyright (C) 2015 Kyle Niemeyer, niemeyek@oregonstate.edu AND
Daniel Magee, mageed@oregonstate.edu

This program is free software: you can redistribute it and/or modify
it under the terms of the MIT license.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.

You should have received a copy of the MIT license
along with this program.  If not, see <https://opensource.org/licenses/MIT>.
*/

//COMPILE LINE!
// nvcc -o ./bin/KSOut KS1D_SweptShared.cu -gencode arch=compute_35,code=sm_35 -lm -restrict -Xcompiler -fopenmp --ptxas-options=-v

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

#include <iostream>
#include <ostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>

#ifndef REAL
    #define REAL        float
    #define TWO         2.f
	#define FOUR        4.f
	#define SIX			6.f
#else
    #define TWO         2.0
	#define FOUR        4.0
	#define SIX			6.0
#endif

#define BASE            36
#define HEIGHT          18
#define WARPSIZE        32
#define TPB             256
#define WPB             8
#define TWOBASE         72

using namespace std;

const REAL dx = 0.5;

struct discConstants{

	REAL dx_i4; // 1/(4*dx)
	REAL dx2_i; // 1/(dx^2)
	REAL dx4_i; // 1/(dx^4)
	REAL dt; // dt
	REAL dt_half; // dt/2
};

__constant__ discConstants disc;

__host__
REAL initFun(REAL xnode)
{
	return TWO * cos(19.0*xnode*M_PI/128.0);
}

__device__
__forceinline__
REAL fourthDer(REAL tfarLeft, REAL tLeft, REAL tCenter, REAL tRight, REAL tfarRight)
{
	return disc.dx4_i * (tfarLeft - FOUR*tLeft + SIX*tCenter - FOUR*tRight + tfarRight);
}

__device__
__forceinline__
REAL secondDer(REAL tLeft, REAL tRight, REAL tCenter)
{
	return disc.dx2_i * (tLeft + tRight - TWO*tCenter);
}

__device__
__forceinline__
REAL convect(REAL tLeft, REAL tRight)
{
	return disc.dx_i4 * (tRight*tRight - tLeft*tLeft);
}

__device__
__forceinline__
REAL stutterStep(REAL tfarLeft, REAL tLeft, REAL tCenter, REAL tRight, REAL tfarRight)
{
	return tCenter - disc.dt_half * (convect(tLeft, tRight) + secondDer(tLeft, tRight, tCenter) +
		fourthDer(tfarLeft, tLeft, tCenter, tRight, tfarRight));
}

__device__
__forceinline__
REAL finalStep(REAL tfarLeft, REAL tLeft, REAL tCenter, REAL tRight, REAL tfarRight)
{
	return (-disc.dt * (convect(tLeft, tRight) + secondDer(tLeft, tRight, tCenter) +
			fourthDer(tfarLeft, tLeft, tCenter, tRight, tfarRight)));
}

__global__
void
swapKernel(const REAL *passing_side, REAL *bin, int direction)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
    int lastidx = ((blockDim.x*gridDim.x)-1);
    int gidout = (gid + direction*blockDim.x) & lastidx;

    bin[gidout] = passing_side[gid];

}

//Classic
__global__
void
classicKS(const REAL *ks_in, REAL *ks_out, bool final)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
    int lastidx = ((blockDim.x*gridDim.x)-1);

	if (final)
	{
		ks_out[gid] += finalStep(ks_in[(gid-2)&lastidx], ks_in[(gid-1)&lastidx],
			ks_in[gid], ks_in[(gid+1)&lastidx], ks_in[(gid+2)&lastidx]);
	}
	else
	{
		ks_out[gid] = stutterStep(ks_in[(gid-2)&lastidx], ks_in[(gid-1)&lastidx], ks_in[gid],
			ks_in[(gid+1)&lastidx], ks_in[(gid+2)&lastidx]);
	}
}

__global__
void
upTriangle(const REAL *IC, REAL *right, REAL *left)
{
	__shared__ REAL temper[WPB][TWOBASE];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int wid = threadIdx.x & 31; //Thread id in warp.
    int wtag = threadIdx.x/TPB; //Warp id in block.
    int widx = wid + 2;
    int widTop = widx+BASE;

	int leftidx = (((wid>>2) & 1) * BASE) + ((wid>>2)<<1) + (wid & 3) + 2;
	int rightidx = 30 + (((wid>>2) & 1) * BASE) + (wid & 3) - ((tid>>2)<<1);

    REAL vel[2];
    //Assign the initial values to the first row in temper, each block
    //has it's own version of temper shared among its threads.
	vel[0] = IC[gid];

	__syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    temper[wtag][widx] = vel[0];
    temper[wtag][widTop] = vel[1];

	__syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid > 3 && wid < 28) temper[wtag][widx] = vel[0];

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid > 5 && wid < 26) temper[wtag][widTop] = vel[1];

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid > 7 && wid < 24) temper[wtag][widx] = vel[0];

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid > 9 && wid < 22) temper[wtag][widTop] = vel[1];

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid > 11 && wid < 20) temper[wtag][widx] = vel[0];

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid > 13 && wid < 18) temper[wtag][widTop] = vel[1];

	//Make sure the threads are synced
	__syncthreads();

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[wtag][rightidx];
	left[gid] = temper[wtag][leftidx];
}

__global__
void
downTriangle(REAL *IC, const REAL *right, const REAL *left)
{
    __shared__ REAL temper[WPB][TWOBASE];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int wid = threadIdx.x & 31; //Thread id in warp.
    int wtag = threadIdx.x/TPB; //Warp id in block.
    int widx = wid + 2;
    int widTop = wid+BASE;

    int leftidx = HEIGHT + (((wid>>2) & 1) * BASE) + (wid & 3) - (4 + ((wid>>2) << 1));
	int rightidx = HEIGHT + (((wid>>2) & 1) * BASE) + ((wid>>2)<<1) + (wid & 3);

	temper[wtag][leftidx] = right[gid];
	temper[wtag][rightidx] = left[gid];

    //stutter first
    vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 14 || wid > 17) vel[1] = temper[wtag][widTop];

    __syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid < 12 || wid > 19) vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 10 || wid > 21) vel[1] = temper[wtag][widTop];

    __syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid < 8 || wid > 23) vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 6 || wid > 25) vel[1] = temper[wtag][widTop];

    __syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid < 4 || wid > 27) vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 1 || wid > 30) temper[wtag][widTop] = vel[1];

    __syncthreads();

    //This is where to do it with shared mem.
    vel[0] += finalStep(temper[wtag][widTop-2],temper[wtag][widTop-1],temper[wtag][widTop],
        temper[wtag][widTop+1],temper[wtag][widTop+2]);

    IC[gid] = vel[0];
}


__global__
void
wholeDiamond(REAL *right, REAL *left)
{
    __shared__ REAL temper[WPB][TWOBASE];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int wid = threadIdx.x & 31; //Thread id in warp.
    int wtag = threadIdx.x/TPB; //Warp id in block.
    int widx = wid+2;
    int widxTop = widx+BASE;

    int leftidx = HEIGHT + (((wid>>2) & 1) * BASE) + (wid & 3) - (4 + ((wid>>2) << 1));
	int rightidx = HEIGHT + (((wid>>2) & 1) * BASE) + ((wid>>2)<<1) + (wid & 3);

	temper[wtag][leftidx] = right[gid];
	temper[wtag][rightidx] = left[gid];

    //stutter first
    vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 14 || wid > 17) vel[1] = temper[wtag][widTop];

    __syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid < 12 || wid > 19) vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 10 || wid > 21) vel[1] = temper[wtag][widTop];

    __syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid < 8 || wid > 23) vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 6 || wid > 25) vel[1] = temper[wtag][widTop];

    __syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid < 4 || wid > 27) vel[0] = temper[wtag][widx];

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid < 1 || wid > 30) temper[wtag][widTop] = vel[1];

    __syncthreads();

    //This is where to do it with shared mem.
    vel[0] += finalStep(temper[wtag][widTop-2],temper[wtag][widTop-1],temper[wtag][widTop],
        temper[wtag][widTop+1],temper[wtag][widTop+2]);

    leftidx = (((wid>>2) & 1) * BASE) + ((wid>>2)<<1) + (wid & 3) + 2;
    rightidx = 30 + (((wid>>2) & 1) * BASE) + (wid & 3) - ((tid>>2)<<1);

    __syncthreads();

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    temper[wtag][widx] = vel[0];
    temper[wtag][widTop] = vel[1];

	__syncthreads();

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid > 3 && wid < 28) temper[wtag][widx] = vel[0];

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid > 5 && wid < 26) temper[wtag][widTop] = vel[1];

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid > 7 && wid < 24) temper[wtag][widx] = vel[0];

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid > 9 && wid < 22) temper[wtag][widTop] = vel[1];

    vel[0] += finalStep(__shfl_up(vel[1],2),__shfl_up(vel[1],1),vel[1],
        __shfl_down(vel[1],1),__shfl_down(vel[1],2));

    if (wid > 11 && wid < 20) temper[wtag][widx] = vel[0];

    vel[1] = stutterStep(__shfl_up(vel[0],2),__shfl_up(vel[0],1),vel[0],
        __shfl_down(vel[0],1),__shfl_down(vel[0],2));

    if (wid > 13 && wid < 18) temper[wtag][widTop] = vel[1];

	//Make sure the threads are synced
	__syncthreads();

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[wtag][rightidx];
	left[gid] = temper[wtag][leftidx];
}

double
classicWrapper(const int bks, int tpb, const int dv, const REAL dt, const REAL t_end,
    REAL *IC, REAL *T_f, const REAL freq, ofstream &fwr)
{
    REAL *dks_in, *dks_out;

    hipMalloc((void **)&dks_in, sizeof(REAL)*dv);
    hipMalloc((void **)&dks_out, sizeof(REAL)*dv);

    // Copy the initial conditions to the device array.
    hipMemcpy(dks_in,IC,sizeof(REAL)*dv,hipMemcpyHostToDevice);

    double t_eq = 0.0;
    double twrite = freq;

    while (t_eq <= t_end)
    {
        classicKS <<< bks,tpb >>> (dks_in, dks_out, false);
        classicKS <<< bks,tpb >>> (dks_out, dks_in, true);
        t_eq += dt;

        if (t_eq > twrite)
        {
            hipMemcpy(T_f, dks_in, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

			fwr << " Velocity " << t_eq << " ";
            for (int k = 0; k<dv; k++)
            {
                fwr << T_f[k] << " ";
            }
            fwr << endl;

            twrite += freq;
        }
    }

    hipMemcpy(T_f, dks_in, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

    hipFree(dks_in);
    hipFree(dks_out);

    return t_eq;
}

//The host routine.
double
sweptWrapper(const int bks, int tpb, const int dv, REAL dt, const REAL t_end,
	REAL *IC, REAL *T_f, const REAL freq, ofstream &fwr)
{

	REAL *d_IC, *d_right, *d_left, *d_bin;
	hipMalloc((void **)&d_IC, sizeof(REAL)*dv);
	hipMalloc((void **)&d_right, sizeof(REAL)*dv);
	hipMalloc((void **)&d_left, sizeof(REAL)*dv);
	hipMalloc((void **)&d_bin, sizeof(REAL)*dv);

	// Copy the initial conditions to the device array.
	hipMemcpy(d_IC,IC,sizeof(REAL)*dv,hipMemcpyHostToDevice);
	//Start the counter and start the clock.
	//
	//Every other step is a full timestep and each cycle is half tpb steps.
	const double t_fullstep = 0.25 * dt * (double)tpb;
	double twrite = freq;

	const size_t smem1 = 2*tpb*sizeof(REAL);
	const size_t smem2 = (2*tpb+8)*sizeof(REAL);

	upTriangle <<< bks,tpb,smem1 >>> (d_IC,d_right,d_left);

	swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
	swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

	//Split
	wholeDiamond <<< bks,tpb,smem2 >>> (d_right,d_left);

	swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
	swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

	double t_eq = t_fullstep;

	// Call the kernels until you reach the iteration limit.
	while(t_eq < t_end)
	{

		wholeDiamond <<< bks,tpb,smem2 >>> (d_right,d_left);

		swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
		swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

		//So it always ends on a left pass since the down triangle is a right pass.

		//Split
		wholeDiamond <<< bks,tpb,smem2 >>> (d_right,d_left);

		swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
		swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

		t_eq += t_fullstep;


	 	if (t_eq > twrite)
		{
			downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);

			hipMemcpy(T_f, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

			fwr << " Velocity " << t_eq << " ";

			for (int k = 0; k<dv; k++)	fwr << T_f[k] << " ";

			fwr << endl;

			upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);

			swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
			swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

			//Split
			wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);

			swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
			swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

			t_eq += t_fullstep;

			twrite += freq;
		}

	}

	downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);

	hipMemcpy(T_f, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

	hipFree(d_IC);
	hipFree(d_right);
	hipFree(d_left);
	hipFree(d_bin);

	return t_eq;

}

int main( int argc, char *argv[])
{

	if (argc < 9)
	{
		cout << "The Program takes 9 inputs, #Divisions, #Threads/block, deltat, finish time, output frequency..." << endl;
        cout << "Classic/Swept, CPU sharing Y/N (Ignored), Variable Output File, Timing Output File (optional)" << endl;
		exit(-1);
	}

	// Choose the GPGPU.  This is device 0 in my machine which has 2 devices.
	hipSetDevice(0);
	if (sizeof(REAL)>6) hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

	const int dv = atoi(argv[1]); //Number of spatial points
	const int tpb = atoi(argv[2]); //Threads per Block
    const REAL dt = atof(argv[3]); //delta T timestep
	const float tf = atof(argv[4]); //Finish time
    const float freq = atof(argv[5]); //Output frequency
    const int scheme = atoi(argv[6]); //1 for Swept 0 for classic
    // const int tst = atoi(argv[7]); CPU/GPU share
    const int bks = dv/tpb; //The number of blocks
	const float lx = dv*dx;
	char const *prec;
	prec = (sizeof(REAL)<6) ? "Single": "Double";

	cout << "KS --- #Blocks: " << bks << " | Length: " << lx << " | Precision: " << prec << " | dt/dx: " << dt/dx << endl;

	//Conditions for main input.  Unit testing kinda.
	//dv and tpb must be powers of two.  dv must be larger than tpb and divisible by
	//tpb.

	if ((dv & (tpb-1) !=0) || (tpb&31) != 0)
    {
        cout << "INVALID NUMERIC INPUT!! "<< endl;
        cout << "2nd ARGUMENT MUST BE A POWER OF TWO >= 32 AND FIRST ARGUMENT MUST BE DIVISIBLE BY SECOND" << endl;
        exit(-1);
    }

	discConstants dsc = {
		1.0/(FOUR*dx),
		1.0/(dx*dx),
		1.0/(dx*dx*dx*dx),
		dt,
		dt*0.5
	};

	// Initialize arrays.
    REAL *IC, *T_final;

	hipHostAlloc((void **) &IC, dv*sizeof(REAL), hipHostMallocDefault);
	hipHostAlloc((void **) &T_final, dv*sizeof(REAL), hipHostMallocDefault);

    // IC = (REAL *) malloc(dv*sizeof(REAL));
    // T_final = (REAL *) malloc(dv*sizeof(REAL));

	// Inital condition
	for (int k = 0; k<dv; k++)
	{
		IC[k] = initFun((float)k*dx);
	}

	// Call out the file before the loop and write out the initial condition.
	ofstream fwr;
	fwr.open(argv[8],ios::trunc);

	// Write out x length and then delta x and then delta t.
	// First item of each line is timestamp.
	fwr << lx << " " << dv << " " << dx << " " << endl << " Velocity " << 0 << " ";

	for (int k = 0; k<dv; k++) fwr << IC[k] << " ";

	fwr << endl;
	// Transfer data to GPU.

	// This puts the constant part of the equation in constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(disc),&dsc,sizeof(dsc));

	// Start the counter and start the clock.
	hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

	// Call the kernels until you reach the iteration limit.
	double tfm;
	if (scheme)
    {
		cout << "Swept" << endl;
		tfm = sweptWrapper(bks, tpb, dv, dsc.dt, tf, IC, T_final, freq, fwr);
	}
	else
	{
		cout << "Classic" << endl;
		tfm = classicWrapper(bks, tpb, dv, dsc.dt, tf, IC, T_final, freq, fwr);
	}

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime( &timed, start, stop);

	timed *= 1.e3;

	double n_timesteps = tfm/dt;

    double per_ts = timed/n_timesteps;

    cout << n_timesteps << " timesteps" << endl;
	cout << "Averaged " << per_ts << " microseconds (us) per timestep" << endl;

    if (argc>8)
    {
        ofstream ftime;
        ftime.open(argv[9],ios::app);
    	ftime << dv << "\t" << tpb << "\t" << per_ts << endl;
    	ftime.close();
    }

	fwr << " Velocity " << tfm << " ";
	for (int k = 0; k<dv; k++) fwr << T_final[k] << " ";

    fwr << endl;

	fwr.close();

	hipDeviceSynchronize();
	// Free the memory and reset the device.

	hipEventDestroy( start );
	hipEventDestroy( stop );
	hipDeviceReset();

	hipHostFree(IC);
    hipHostFree(T_final);
	// free(IC);
	// free(T_final);

	return 0;

}
