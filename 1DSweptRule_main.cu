#include "hip/hip_runtime.h"
/* This file is the current iteration of research being done to implement the
swept rule for Partial differential equations in one dimension.  This research
is a collaborative effort between teams at MIT, Oregon State University, and
Purdue University.

Copyright (C) 2015 Kyle Niemeyer, niemeyek@oregonstate.edu AND
Daniel Magee, mageed@oregonstate.edu

This program is free software: you can redistribute it and/or modify
it under the terms of the MIT license.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.

You should have received a copy of the MIT license
along with this program.  If not, see <https://opensource.org/licenses/MIT>.
*/

/*
Note that this code alters the original scheme. Paper available here:
http://www.sciencedirect.com/science/article/pii/S0021999115007664
The nodes never calculate a full diamond in a single kernel call and the boundary
values only get passed one direction, right.  This is a slightly simpler
application that passes the shared values in each node to the GPU global memory
more often.  This cuts down on some of the logic required in the full scheme and
makes results easier to output at various points in the solution.
*/

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "hip/device_functions.h"

#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <ostream>
#include <cstring>
#include <fstream>
#include "SwR_1DShared.cuh"

using namespace std;

// Define Given Parameters.  Material is aluminum.
#define DIVISIONS  1024.
#define LENX       5.
#define TS         .01
//#define ITERLIMIT  50000
#define REAL       float
#define TH_DIFF    8.418e-5
//#define THREADBLK  32

// Declare constant Fourier number that will go in Device constant memory.
__constant__ REAL fo;

//The host routine.
int main()
{
	// Choose the GPGPU.  This is device 0 in my machine which has 2 devices.
	hipSetDevice(0);
	const int dv = int(DIVISIONS); //Setting it to an int helps with arrays
	const int bks = dv/THREADBLK; //The number of blocks since threads/block = 32.
	// Threads/block will be experimented on.
	const REAL ds = LENX/(DIVISIONS-1); //The x division length.
	REAL fou = TS*TH_DIFF/(ds*ds); //The Fourier number.

	// Initialize arrays.
	REAL IC[dv];
	REAL T_final[dv];
	REAL *d_IC, *d_right, *d_left;

	// Some initial condition for the bar temperature, an exponential decay
	// function.
	for (int k = 0; k<dv; k++)
	{
		IC[k] = 500.f*expf((-ds*k)/LENX);
	}

	cout << fou << endl;
	// Call out the file before the loop and write out the initial condition.
	ofstream fwr;
	fwr.open("1DHeatEQResult.dat",ios::trunc);
	// Write out x length and then delta x and then delta t.
	// First item of each line is timestamp.
	fwr << LENX << " " << DIVISIONS << " " << TS << " " << endl << 0 << " ";

	for (int k = 0; k<dv; k++)
	{
		fwr << IC[k] << " ";
	}

	fwr << endl;

	// Transfer data to GPU.

	// This puts the Fourier number in constant memory.
	hipMemcpyToSymbol(HIP_SYMBOL(fo),&fou,sizeof(REAL));

	// This initializes the device arrays on the device in global memory.
	// They're all the same size.  Conveniently.
	hipMalloc((void **)&d_IC, sizeof(REAL)*dv);
	hipMalloc((void **)&d_right, sizeof(REAL)*dv);
	hipMalloc((void **)&d_left, sizeof(REAL)*dv);

	// Copy the initial conditions to the device array.
	hipMemcpy(d_IC,IC,sizeof(REAL)*dv,hipMemcpyHostToDevice);

	// Start the counter and start the clock.
	REAL t_eq = 0.;
	REAL t_fullstep = TS*(THREADBLK+1);
	double wall0 = clock();

	// Call the kernels until you reach the iteration limit.
	while(t_eq < 1e5)
	{

		upTriangle <<< bks,THREADBLK >>>(d_IC,d_right,d_left);

		downTriangle <<< bks,THREADBLK >>>(d_IC,d_right,d_left);


		t_eq += t_fullstep;

		/* Since the procedure does not store the temperature values, the user
		could input some time interval for which they want the temperature
		values and this loop could copy the values over from the device and
		write them out.  This way the user could see the progression of the
		solution over time, identify an area to be investigated and re-run a
		shorter version of the simulation starting with those intiial conditions.

		-------------------------------------
	 	if (true)
		{
		hipMemcpy(T_final, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);
		fwr << t_eq << " ";

		for (int k = 0; k<dv; k++)
		{
				fwr << T_final[k] << " ";
			}
			fwr << endl;
		}
		-------------------------------------
		*/


	}

	// Show the time and write out the final condition.
	double wall1 = clock();
	double timed = (wall1-wall0)/CLOCKS_PER_SEC;

	cout << "That took: " << timed << " seconds" << endl;


	hipMemcpy(T_final, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);
	fwr << t_eq << " ";
	for (int k = 0; k<dv; k++)
	{
		fwr << T_final[k] << " ";
	}

	fwr.close();

	// Free the memory and reset the device.
	hipFree(d_IC);
	hipFree(d_right);
	hipFree(d_left);
	hipDeviceReset();

	return 0;
}
