
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "hip/device_functions.h"

#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <ostream>
#include <cstring>
#include <fstream>

using namespace std;

// Define Given Parameters.  Material is aluminum.
#define DIVISIONS  1024.
#define LENX       50.
#define TS         .5
#define ITERLIMIT  50000
#define REAL       float
#define TH_DIFF    8.418e-5

__constant__ REAL fo;

__global__ void upTriangle(REAL *IC, REAL *right, REAL *left)
{

	__shared__ REAL temper[64];
	__shared__ REAL sR[32];
	__shared__ REAL sL[32];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tid = threadIdx.x;
	int shft_wr;
	int shft_rd;

	temper[tid] = IC[gid];
	__syncthreads();
	int itr = -1;

	for (int k = 1; k<16; k++)
	{
		shft_wr = (k & 1);
		shft_rd = 32*((shft_wr+1) & 1);

		if (tid <= (31-k) && tid >= k)
		{
			temper[tid + (32*shft_wr)] = fo * (temper[tid+shft_rd-1] + temper[tid+shft_rd+1]) + (1-2.*fo) * temper[tid+shft_rd];
		}

		__syncthreads();

		if (shft_wr)
		{
			sL[k+itr] = temper[k-1];
			sL[k+itr+1] = temper[k];
			sL[k+itr+2] = temper[32+k];
			sL[k+itr+3] = temper[33+k];
			sR[k+itr] = temper[31-k];
			sR[k+itr+1] = temper[32-k];
			sR[k+itr+2] = temper[62-k];
			sR[k+itr+3] = temper[63-k];
			itr += 2;
		}

	}

	right[gid] = sR[tid];
	left[gid] = sL[tid];

}


__global__ void downTriangle(REAL *IC, REAL *right, REAL *left)
{

	//Now temper needs to accommodate a longer row by 2, one on each side.
	//since it has two rows that's 4 extra floats.  The last row will still be
	//32 numbers long.
	__shared__ REAL temper[68];
	__shared__ REAL sR[32];
	__shared__ REAL sL[32];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tid = threadIdx.x;
	int shft_rd;
	int shft_wr;

	// Pass to the left so all checks are for block 0.
	// The left ridge is kept by the block.
	sR[tid] = left[gid];

	// The right ridge is passed, each block 1-end gets the right of 0-end-1
	// Block 0 gets the right of the last block.
	if (blockIdx.x > 0)
	{
		sL[tid] = right[gid-blockDim.x];
	}
	else
	{
		sL[tid] = right[blockDim.x*(gridDim.x-1) + tid];

	}

	__syncthreads();

	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.
	// I think I kinda lost the thread here so to speak.
	temper[15] = sL[0];
	temper[16] = sL[1];
	temper[17] = sR[0];
	temper[18] = sR[1];
	int itr = 2;
	int itr2 = 18;
	// k needs to first insert the right and left into the temper and then put the timestep in between them.
	for (int k = 17; k>1; k--)
	{
		// This tells you if the current row is the first or second.
		shft_wr = (k & 1);
		// Read and write are opposite rows.
		shft_rd = 34*((shft_wr+1) & 1);

		if (blockIdx.x > 0)
		{
			if (tid <= (33-k) && tid >= (k-2))
			{
				temper[tid + 1 + (34*shft_wr)] = fo * (temper[tid+shft_rd] + temper[tid+shft_rd+2]) + (1-2.*fo) * temper[tid+shft_rd+1];
			}

		}
		//Split part
		else
		{
			if (tid <= (33-k) && tid >= (k-2))
			{
				if (tid == 15)
				{
					temper[tid + 1 + (34*shft_wr)] = 2. * fo * (temper[tid+shft_rd]-temper[tid+shft_rd+1]) + temper[tid+shft_rd+1];
				}
				else if (tid == 16)
				{
					temper[tid + 1 + (34*shft_wr)] = 2. * fo * (temper[tid+shft_rd+2]-temper[tid+shft_rd+1]) + temper[tid+shft_rd+1];
				}
				else
				{
					temper[tid + 1 + (34*shft_wr)] = fo * (temper[tid+shft_rd] + temper[tid+shft_rd+2]) + (1-2.*fo) * temper[tid+shft_rd+1];
				}
			}

		}

		// Fill edges.  Thread 0 never gets used for both operations so the calculation and the
		// filling are conceptually coincident.
		if (k>2 && tid == 0)
		{
			temper[(k-3)+(34*shft_wr)] = sL[itr];
			temper[(k-2)+(34*shft_wr)] = sL[itr+1];
			temper[itr2+(34*shft_wr)] = sR[itr];
			itr2++;
			temper[itr2+(34*shft_wr)] = sR[itr+1];
			itr+=2;

		}

		__syncthreads();


	}
	//Now there is only global fill to handle.

	if (blockIdx.x > 0)
	{
		//True if it ends on the first row! The first and last of temper on the final row are empty.
		IC[gid - 16] = temper[tid+1];
	}
	else
	{
		if (tid>15)
		{
			IC[gid - 16] = temper[tid+1];
		}
		else
		{
			IC[(blockDim.x * gridDim.x) + (tid - 16) ] = temper[tid+1];


		}
	}

}


int main()
{

	hipSetDevice(0);
	const int dv = int(DIVISIONS);
	const int bks = dv/32;
	const REAL ds = LENX/(DIVISIONS-1);
	REAL fou = TS*TH_DIFF/(ds*ds);

	REAL IC[dv];
	REAL T_final[dv];
	REAL *d_IC, *d_right, *d_left;

	for (int k = 0; k<dv; k++)
	{
		IC[k] = 500.f*expf((-ds*k)/LENX);
	}

	// Call out the file before the loop and write out the initial condition.
	ofstream fwr;
	fwr.open("1DHeatEQResult.dat",ios::trunc);
	// Write out x length and then delta x and then delta t.  First item of each line is timestamp.
	fwr << LENX << " " << DIVISIONS << " " << TS << " " << endl << 0 << " ";

	for (int k = 0; k<dv; k++)
	{
		fwr << IC[k] << " ";
	}

	fwr << endl;

	// Put what you need on the GPU.

	hipMemcpyToSymbol(HIP_SYMBOL(fo),&fou,sizeof(REAL));

	hipMalloc((void **)&d_IC, sizeof(REAL)*dv);
	hipMalloc((void **)&d_right, sizeof(REAL)*dv);
	hipMalloc((void **)&d_left, sizeof(REAL)*dv);

	hipMemcpy(d_IC,IC,sizeof(REAL)*dv,hipMemcpyHostToDevice);

	// Some for loop
	REAL t_eq = 0.;
	double wall0 = clock();

	for(unsigned int k = 0; k < ITERLIMIT; k++)
	{

		upTriangle <<< bks,32 >>>(d_IC,d_right,d_left);

		downTriangle <<< bks,32 >>>(d_IC,d_right,d_left);

		t_eq += (TS*17);

		// Some condition about when to stop and write out values.
		// if (true)
		// {
		// 	cudaMemcpy(T_final, d_IC, sizeof(REAL)*dv, cudaMemcpyDeviceToHost);
		// 	fwr << t_eq << " ";
		//
		// 	for (int k = 0; k<dv; k++)
		// 	{
		// 		fwr << T_final[k] << " ";
		// 	}
		// 	fwr << endl;
		// }


	}

	double wall1 = clock();
	double timed = (wall1-wall0)/CLOCKS_PER_SEC;

	cout << "That took: " << timed << " seconds" << endl;

	hipMemcpy(T_final, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);
	fwr << t_eq << " ";
	for (int k = 0; k<dv; k++)
	{
		fwr << T_final[k] << " ";
	}

	fwr.close();
	// End loop and write out data.

	hipFree(d_IC);
	hipFree(d_right);
	hipFree(d_left);
	hipDeviceReset();

	return 0;
}
