#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "hip/device_functions.h"

#include <iostream>
#include <cmath>
#include <ostream>
#include <fstream>
#include <math.h>

using namespace std;

// Define Given Parameters.  NOTE TIMELIMIT is approximate.
#define DIVISIONS  1024.
#define LENX       50.
#define TS         .5
#define TIMELIMIT  5000.
#define REAL       float
#define TH_DIFF    8.418e-5

__constant__ REAL fo;

__global__ void upTriangle(REAL *IC, REAL *right, REAL *left)
{

	__shared__ REAL temper[64];
	__shared__ REAL sR[32];
	__shared__ REAL sL[32];

	unsigned int gid = blockDim.x * blockIdx.x + threadIdx; //Global Thread ID
	unsigned int tid = threadIdx.x;

	Temper[tid] = IC[gid];
	__syncthreads();
	int itr = -1;

	for (unsigned int k = 1; k<16; k++)
	{
		int sw = (k & 1);
		int shft = sw*2-1;

		if (tid <= (32-k-1) && >= k)
		{
			temper[tid + (32*sw)] = fo * (IC[tid+(32*shft)-1] + IC[tid+(32*shft)+1]) + (1-2.*fo) * IC[tid+(32*shft)];
		}

		__syncthreads();
		if (sw)
		{
			sL[k+itr] = temper[k-1];
			sL[k+itr+1] = temper[k];
			sL[k+itr+2] = temper[32+k];
			sL[k+itr+3] = temper[33+k];
			sR[k+itr] = temper[31-k];
			sR[k+itr+1] = temper[32-k];
			sR[k+itr+2] = temper[61-k];
			sR[k+itr+3] = temper[62-k];
			itr += 2;
		}

	}

	right[gid] = sR[tid];
	left[gid] = sL[tid];
	__syncthreads();

	}

}

__global__ void downTriangle(REAL *IC)
{



}


int main()
{
	const int dv = int(DIVISIONS);
	const int bks = dv/32;
	const REAL ds = LENX/(DIVISIONS-1);
	REAL fou = TS*TH_DIFF/(ds*ds);

	REAL IC[dv];
	REAL *d_IC[dv], *d_right[dv], *d_left[dv];

	for (int k = 0; k<dv; k++)
	{
		IC[k] = 500.f*expf((-ds*k)/LENX);
	}

	// Call out the file before the loop and write out the initial condition.
	ofstream fwr;
	fwr.open("1DHeatEQResult.dat");
	// Write out x length and then delta x and then delta t.  First item of each line is timestamp.
	filewrite << LENX << " " << ds << " " << TS << " " << 0 <<endl;

	for (int k = 0; k<dv; k++)
	{
		fwr << IC[k] << " ";
	}

	fwr << endl;

	// Put what you need on the GPU.

	hipMemcpyToSymbol(HIP_SYMBOL(fo),&fou,sizeof(REAL));

	hipMalloc((void **)&d_IC, sizeof(REAL)*dv);
	hipMalloc((void **)&d_right, sizeof(REAL)*dv);
	hipMalloc((void **)&d_left, sizeof(REAL)*dv);

	hipMemcpy(d_IC,IC,sizeof(REAL)*dv,hipMemcpyHostToDevice);

	// Some for loop

	upTriangle <<< blk,32 >>>(d_IC,d_right,d_left);


	downTriangle <<< blk,32 >>>(d_IC2,d_IC);

	// Some condition about when to stop or copy memory.



	// End loop and write out data.



	hipFree(d_IC);
	hipFree(d_IC2);
	hipFree(d_coll);

	return 1;

}
