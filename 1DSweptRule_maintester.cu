/* This file is the current iteration of research being done to implement the
swept rule for Partial differential equations in one dimension.  This research
is a collaborative effort between teams at MIT, Oregon State University, and
Purdue University.

Copyright (C) 2015 Kyle Niemeyer, niemeyek@oregonstate.edu AND
Daniel Magee, mageed@oregonstate.edu

This program is free software: you can redistribute it and/or modify
it under the terms of the MIT license.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.

You should have received a copy of the MIT license
along with this program.  If not, see <https://opensource.org/licenses/MIT>.
*/

/*
Note that this code alters the original scheme. Paper available here:
http://www.sciencedirect.com/science/article/pii/S0021999115007664
The nodes never calculate a full diamond in a single kernel call and the boundary
values only get passed one direction, right.  This is a slightly simpler
application that passes the shared values in each node to the GPU global memory
more often.  This cuts down on some of the logic required in the full scheme and
makes results easier to output at various points in the solution.
*/

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "hip/device_functions.h"

#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <ostream>
#include <cstring>
#include <fstream>

using namespace std;

// Define Given Parameters.  Material is aluminum.
#define DIVISIONS  1024.
#define LENX       5.
#define TS         .01
//#define ITERLIMIT  50000
#define REAL       float
#define TH_DIFF    8.418e-5
#define THREADBLK  32

// Declare constant Fourier number that will go in Device constant memory.
__constant__ REAL fo;

__global__ void upTriangle(REAL *IC, REAL *right, REAL *left)
{

	REAL temper[16];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tid = threadIdx.x % 32; //Warp or node ID.  Fix this.
	int mid = tid/2;

	temper[0] = IC[gid];

	//The initial conditions are timslice 0 so start k at 1.
	#pragma unroll
	for (int k = 1; k<16; k++)
	{
		temper[k] = fo * (__shfl_down(temper[k-1],1) + __shfl_up(temper[k-1],1)) + (1.-2.*fo) * temper[k-1];
		//Maybe it works.
	}

	//Doesn't work.
	left[gid] = __shfl_up(temper[mid],mid);
	//Try backward.
	right[gid] = __shfl_down(temper[15-mid],15-mid);

}

//The upside down triangle.  This function essentially takes right and left and
//returns IC.

//IMPORTANT note: k and tid were in sync in the first function, but here they're
//out of sync in the loop.  This is because we can't use tid = 33 or 32 and the
//shared temperature array is that long.  BUT in order to fill the arrays, these
//elements must be accessed.  So each element in each row is shifted by +1.
//For instance, thread tid = 16 refers to temper[17].  That being said, tid is
//unique and k is NOT so the index must be referenced by tid.

__global__ void downTriangle(REAL *IC, REAL *right, REAL *left)
{

	//Now temper needs to accommodate a longer row by 2, one on each side.
	//since it has two rows that's 4 extra floats.  The last row will still be
	//32 numbers long.
	__shared__ REAL temper[(2*THREADBLK)+4];
	__shared__ REAL sR[THREADBLK];
	__shared__ REAL sL[THREADBLK];

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int shft_rd;
	int shft_wr;

	// Pass to the left so all checks are for block 0 (this reduces arithmetic).
	// The left ridge is always kept by the block.
	sR[tid] = left[gid];

	// The right ridge is passed, each block 1-end gets the right of 0-end-1
	// Block 0 gets the right of the last block.
	if (blockIdx.x > 0)
	{
		sL[31-tid] = right[gid-blockDim.x];
	}
	else
	{
		sL[31-tid] = right[blockDim.x*(gridDim.x-1) + tid];
	}

	__syncthreads();

	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.
	temper[THREADBLK/2-1] = sL[0];
	temper[THREADBLK/2] = sL[1];
	temper[THREADBLK/2+1] = sR[0];
	temper[THREADBLK/2+2] = sR[1];

	//Now we need two counters since we need to use sL and sR EVERY iteration
	//instead of every other iteration and instead of growing smaller with every
	//iteration this grows larger.
	int itr = 2;
	int itr2 = THREADBLK/2+2;
	//k needs to insert the relevant left right values around the computed values
	//every timestep.  Since it grows larger the loop is reversed.
	for (int k = THREADBLK/2+1; k>1; k--)
	{
		// This tells you if the current row is the first or second.
		shft_wr = (k & 1);
		// Read and write are opposite rows.
		shft_rd = (THREADBLK+2)*((shft_wr+1) & 1);

		//Block 0 is split so it needs a different algorithm.  This algorithm
		//is slightly different than top triangle as described in the note above.
		if (blockIdx.x > 0)
		{
			if (tid <= ((THREADBLK+1)-k) && tid >= (k-2))
			{
				temper[tid + 1 + ((THREADBLK+2)*shft_wr)] = fo * (temper[tid+shft_rd] + temper[tid+shft_rd+2]) + (1.f-2.f*fo) * temper[tid+shft_rd+1];
			}

		}

		//Split part.  This exhibits thread divergence and is suboptimal.
		//So it's ripe to be improved.

		else
		{
			if (tid <= ((THREADBLK+1)-k) && tid >= (k-2))
			{
				if (tid == (THREADBLK/2-1))
				{
					temper[tid + 1 + ((THREADBLK+2)*shft_wr)] = 2.f * fo * (temper[tid+shft_rd]-temper[tid+shft_rd+1]) + temper[tid+shft_rd+1];
				}
				else if (tid == THREADBLK/2)
				{
					temper[tid + 1 + ((THREADBLK+2)*shft_wr)] = 2.f * fo * (temper[tid+shft_rd+2]-temper[tid+shft_rd+1]) + temper[tid+shft_rd+1];
				}
				else
				{
					temper[tid + 1 + ((THREADBLK+2)*shft_wr)] = fo * (temper[tid+shft_rd] + temper[tid+shft_rd+2]) + (1.f-2.f*fo) * temper[tid+shft_rd+1];
				}
			}

		}

		//Fill edges.  Thread 0 never gets used for both operations so the
		//calculation and the filling are conceptually coincident.
		//Threads are synced afterward here because the next timestep is
		//reliant on the entire loop.
		if (k>2 && tid == 0)
		{
			temper[(k-3)+((THREADBLK+2)*shft_wr)] = sL[itr];
			temper[(k-2)+((THREADBLK+2)*shft_wr)] = sL[itr+1];
			temper[itr2+((THREADBLK+2)*shft_wr)] = sR[itr];
			itr2++;
			temper[itr2+((THREADBLK+2)*shft_wr)] = sR[itr+1];
			itr+=2;

		}
		__syncthreads();

	}

	//Now fill the global unified timestep variable with the final calculated
	//temperatures.

	//Blocks 1 to end hold values 16 to end-16.
	if (blockIdx.x > 0)
	{
		//True if it ends on the first row! The first and last of temper on the final row are empty.
		IC[gid - (THREADBLK/2)] = temper[tid+1];
	}
	//Block 0 holds values 0 to 15 and end-15 to end.  In that order.
	else
	{
		if (tid >= THREADBLK/2)
		{
			IC[gid - (THREADBLK/2)] = temper[tid+1];
		}
		else
		{
			IC[(blockDim.x * gridDim.x) + (tid - THREADBLK/2) ] = temper[tid+1];
		}
	}
}


//The host routine.
int main()
{
	//Choose the GPGPU.  This is device 0 in my machine which has 2 devices.
	hipSetDevice(0);
	const int dv = int(DIVISIONS); //Setting it to an int helps with arrays
	const int bks = dv/THREADBLK; //The number of blocks since threads/block = 32.
	//Threads/block will be experimented on.
	const REAL ds = LENX/(DIVISIONS-1); //The x division length.
	REAL fou = TS*TH_DIFF/(ds*ds); //The Fourier number.

	//Initialize arrays.
	REAL IC[dv];
	REAL T_final[dv];
	REAL *d_IC, *d_right, *d_left;

	// Some initial condition for the bar temperature, an exponential decay
	// function.
	for (int k = 0; k<dv; k++)
	{
		IC[k] = 500.f*expf((-ds*k)/LENX);
	}

	cout << fou << endl;
	// Call out the file before the loop and write out the initial condition.
	ofstream fwr;
	fwr.open("1DHeatEQResult.dat",ios::trunc);
	// Write out x length and then delta x and then delta t.
	// First item of each line is timestamp.
	fwr << LENX << " " << DIVISIONS << " " << TS << " " << endl << 0 << " ";

	for (int k = 0; k<dv; k++)
	{
		fwr << IC[k] << " ";
	}

	fwr << endl;

	// Transfer data to GPU.

	// This puts the Fourier number in constant memory.
	hipMemcpyToSymbol(HIP_SYMBOL(fo),&fou,sizeof(REAL));

	// This initializes the device arrays on the device in global memory.
	// They're all the same size.  Conveniently.
	hipMalloc((void **)&d_IC, sizeof(REAL)*dv);
	hipMalloc((void **)&d_right, sizeof(REAL)*dv);
	hipMalloc((void **)&d_left, sizeof(REAL)*dv);

	//Copy the initial conditions to the device array.
	hipMemcpy(d_IC,IC,sizeof(REAL)*dv,hipMemcpyHostToDevice);

	// Start the counter and start the clock.
	REAL t_eq = 0.;
	REAL t_fullstep = TS*(THREADBLK+1);
	double wall0 = clock();

	// Call the kernels until you reach the iteration limit.
	while(t_eq < 1e5)
	{

		upTriangle <<< bks,THREADBLK >>>(d_IC,d_right,d_left);

		downTriangle <<< bks,THREADBLK >>>(d_IC,d_right,d_left);


		t_eq += t_fullstep;

		/* Since the procedure does not store the temperature values, the user
		could input some time interval for which they want the temperature
		values and this loop could copy the values over from the device and
		write them out.  This way the user could see the progression of the
		solution over time, identify an area to be investigated and re-run a
		shorter version of the simulation starting with those intiial conditions.

		-------------------------------------
	 	if (true)
		{
		cudaMemcpy(T_final, d_IC, sizeof(REAL)*dv, cudaMemcpyDeviceToHost);
		fwr << t_eq << " ";

		for (int k = 0; k<dv; k++)
		{
				fwr << T_final[k] << " ";
			}
			fwr << endl;
		}
		-------------------------------------
		*/


	}

	//Show the time and write out the final condition.
	double wall1 = clock();
	double timed = (wall1-wall0)/CLOCKS_PER_SEC;

	cout << "That took: " << timed << " seconds" << endl;


	hipMemcpy(T_final, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);
	fwr << t_eq << " ";
	for (int k = 0; k<dv; k++)
	{
		fwr << T_final[k] << " ";
	}

	fwr.close();

	//Free the memory and reset the device.
	hipFree(d_IC);
	hipFree(d_right);
	hipFree(d_left);
	hipDeviceReset();

	return 0;
}
