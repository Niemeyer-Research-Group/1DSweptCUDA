#include "hip/hip_runtime.h"
/* This file is the current iteration of research being done to implement the
swept rule for Partial differential equations in one dimension.  This research
is a collaborative effort between teams at MIT, Oregon State University, and
Purdue University.

Copyright (C) 2015 Kyle Niemeyer, niemeyek@oregonstate.edu AND
Daniel Magee, mageed@oregonstate.edu

This program is free software: you can redistribute it and/or modify
it under the terms of the MIT license.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.

You should have received a copy of the MIT license
along with this program.  If not, see <https://opensource.org/licenses/MIT>.
*/

//This is to test different triangle version for shared memory.

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "hip/device_functions.h"

//NEW EDGE COLLECTION ALGORITHM!!!!

#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ostream>
#include <cstring>
#include <fstream>

using namespace std;

// Define Given Parameters.  Material is aluminum.
//#define DIVISIONS  1024.
#define TS         .01
//#define ITERLIMIT  50000
#define REAL       float
#define TH_DIFF    8.418e-5
#ifndef FINISH
#define FINISH		1e4
#endif

#define THREADBLK  32

// Declare constant Fourier number that will go in Device constant memory.
__constant__ REAL fo;


//FILLING POST LOOP AND GLOBAL ONLY EDGES.
__global__ void upTriangle_GA(REAL *IC, REAL *right, REAL *left)
{
	/*
	Initialize shared variables.  Each node (warp) will store 32 values on the
	right and left sides of their triangle, 2 on each side for each timeslice.
	Since the base of the triangle is 32 numbers for each node, 16 timeslices
	are evaluated per kernel call.
	Temper stores the temperatures at each timeslice.  Since only the current
	and previous timeslice results need to be held at each iteration.  This
	variable has 64 values, or two rows of 32, linearized.  The current and
	previous value alternate rows at each timeslice.
	*/
	extern __shared__ REAL temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tid = threadIdx.x; //Warp or node ID
	int tidp = tid + 1;
	int tidm = tid - 1;
	int shft_wr; //Initialize the shift to the written row of temper.
	int shft_rd; //Initialize the shift to the read row (opposite of written)
	int leftidx = tid/2 + ((tid/2 & 1) * blockDim.x) + (tid & 1);
	int rightidx = (blockDim.x - 1) + ((tid/2 & 1) * blockDim.x) + (tid & 1) -  tid/2;

	//Assign the initial values to the first row in temper, each warp (in this
	//case each block) has it's own version of temper shared among its threads.
	temper[tid] = IC[gid];

	//The initial conditions are timslice 0 so start k at 1.
	for (int k = 1; k<(blockDim.x/2); k++)
	{
		//Bitwise even odd. On even iterations write to first row.
		shft_wr = blockDim.x * (k & 1);
		//On even iterations write to second row (starts at element 32)
		shft_rd = blockDim.x * ((k + 1) & 1);

		//Each iteration the triangle narrows.  When k = 1, 30 points are
		//computed, k = 2, 28 points.
		if (tid < ((blockDim.x-k) && tid => k)
		{
			temper[tid + shft_wr] = fo * (temper[tidm+shft_rd] + temper[tidp+shft_rd]) + (1.f-2.f*fo) * temper[tid+shft_rd];
		}

		//Make sure the threads are synced
		__syncthreads();

	}

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];

}

__global__ void downTriangle_GA(REAL *IC, REAL *right, REAL *left)
{

	//Now temper needs to accommodate a longer row by 2, one on each side.
	//since it has two rows that's 4 extra floats.  The last row will still be
	//32 numbers long.

	extern __shared__ REAL temper[];
	int tidp = tid + 1;
	int tidm = tid - 1;

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tidp = tid + 1;
	int tidm = tid - 1;
	int base = blockDim.x + 2;
	int height = base/2;
	int shft_rd;
	int shft_wr;
	int leftidx = base/2 - tid/2 ((tid/2 & 1) * base) + (tid & 1) - 2;
	int rightidx = base/2 + tid/2 ((tid/2 & 1) * base) + (tid & 1);
	int gidout = (gid - blockDim.x/2) & ((blockDim.x*gridDim.x)-1)
	int gidin = (gid - blockDim.x) & ((blockDim.x*gridDim.x)-1)

	// Pass to the left so all checks are for block 0 (this reduces arithmetic).
	// The left ridge is always kept by the block.

	// The right ridge is passed, each block 1-end gets the right of 0-end-1
	// Block 0 gets the right of the last block.


	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.

	__syncthreads();
	temper[leftidx] = right[gidin]
	temper[rightidx] = left[gid]

	//k needs to insert the relevant left right values around the computed values
	//every timestep.  Since it grows larger the loop is reversed.
	for (int k = height-1 k>1; k--)
	{
		// This tells you if the current row is the first or second.
		shft_wr = base * ((k+1) & 1);
		// Read and write are opposite rows.
		shft_rd = base * ((k & 1);

		//Block 0 is split so it needs a different algorithm.  This algorithm
		//is slightly different than top triangle as described in the note above.
		if (blockIdx.x > 0)
		{
			if (tidp <= (blockDim.x-k) && tidp >= k)
			{
				temper[tidp + shft_wr)] = fo * (temper[tid+shft_rd] + temper[tid+shft_rd+2]) + (1.f-2.f*fo) * temper[tidp+shft_rd];
			}

		}

		//Split part.  This exhibits thread divergence and is suboptimal.
		//So it's ripe to be improved.

		else
		{
			if (tidp <= ((THREADBLK+1)-k) && tidp >= k)
			{
				if (tid == (height-1))
				{
					temper[tidp + (base*shft_wr)] = 2.f * fo * (temper[tid+shft_rd]-temper[tid+shft_rd+1]) + temper[tidp + shft_rd];
				}
				else if (tid == height)
				{
					temper[tidp + (base*shft_wr)] = 2.f * fo * (temper[tid+shft_rd+2]-temper[tid+shft_rd+1]) + temper[tidp + shft_rd];
				}
				else
				{
					temper[tidp + (base*shft_wr)] = fo * (temper[tid+shft_rd] + temper[tid+shft_rd+2]) + (1.f-2.f*fo) * temper[tidp +shft_rd];
				}
			}

		}

		__syncthreads();
	}

	IC[gidout] = temper[gid]

}

// OLDEST CODE.  GETS AND INSERTS EDGES INSIDE LOOP
// SPLIT

__global__ void upTriangle_SI(REAL *IC, REAL *right, REAL *left)
{

	extern __shared__ REAL share[];

	REAL *temper = (REAL *) share;
	REAL *shRight = (REAL *) &share[2*blockDim.x+4];
	REAL *shLeft = (REAL *) &share[3*blockDim.x+4];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tid = threadIdx.x; //Warp or node ID
	int shft_wr; //Initialize the shift to the written row of temper.
	int shft_rd; //Initialize the shift to the read row (opposite of written)

	//Assign the initial values to the first row in temper, each warp (in this
	//case each block) has it's own version of temper shared among its threads.
	temper[tid] = IC[gid];
	__syncthreads(); // Then make sure each block of threads are synced.

	int itr = -1;

	//The initial conditions are timslice 0 so start k at 1.
	for (int k = 1; k<(blockDim.x/2); k++)
	{
		//Bitwise even odd. On even iterations write to first row.
		shft_wr = blockDim.x * (k & 1);
		//On even iterations write to second row (starts at element 32)
		shft_rd = blockDim.x * ((k + 1) & 1);

		//Each iteration the triangle narrows.  When k = 1, 30 points are
		//computed, k = 2, 28 points.
		if (tid < ((blockDim.x-k) && tid => k)
		{
			temper[tid + shft_wr] = fo * (temper[tidm+shft_rd] + temper[tidp+shft_rd]) + (1.f-2.f*fo) * temper[tid+shft_rd];
		}

		//Make sure the threads are synced
		__syncthreads();
		if (shft_wr && tid < 4)
		{
			shLeft[k+itr+tid] = temper[(tid/2*(THREADBLK-1))+(tid-1)+k];
			shRight[k+itr+tid] = temper[((tid+2)/2*(THREADBLK-1))+(tid&1)-k];
			itr += 2;
		}

		__syncthreads();

	}

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = shRight[tid];
	left[gid] = shLeft[tid];

}

__global__ void downTriangle_SI(REAL *IC, REAL *right, REAL *left)
{

	extern __shared__ REAL share[];

	REAL *temper = (REAL *) share;
	REAL *shRight = (REAL *) &share[2*blockDim.x+4];
	REAL *shLeft = (REAL *) &share[3*blockDim.x+4];

	int base = blockDim.x + 2;

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tidp = tid + 1;
	int height = blockDim.x/2;
	int shft_rd;
	int shft_wr;

	// Pass to the left so all checks are for block 0 (this reduces arithmetic).
	// The left ridge is always kept by the block.
	shRight[tid] = left[gid];

	// The right ridge is passed, each block 1-end gets the right of 0-end-1
	// Block 0 gets the right of the last block.
	if (blockIdx.x > 0)
	{
		shLeft[tid] = right[gid-blockDim.x];
	}
	else
	{
		shLeft[tid] = right[blockDim.x*(gridDim.x-1) + tid];
	}

	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.
	if (tid < 2)
	{
		temper[tid+height-1] = shLeft[tid];
		temper[tidp+height] = shRight[tid];
	}
	__syncthreads();
	//Now we need two counters since we need to use shLeft and shRight EVERY iteration
	//instead of every other iteration and instead of growing smaller with every
	//iteration this grows larger.
	int itr = 2;

	//k needs to insert the relevant left right values around the computed values
	//every timestep.  Since it grows larger the loop is reversed.
	for (int k = height; k>1; k--)
	{
		// This tells you if the current row is the first or second.
		shft_wr = ((k+1) & 1);
		// Read and write are opposite rows.
		shft_rd = base*((shft_wr+1) & 1);

		//Block 0 is split so it needs a different algorithm.  This algorithm
		//is slightly different than top triangle as described in the note above.
		if (blockIdx.x > 0)
		{
			if (tidp <= ((THREADBLK+1)-k) && tidp >= k)
			{
				temper[tidp + (base*shft_wr)] = fo * (temper[tid+shft_rd] + temper[tid+shft_rd+2]) + (1.f-2.f*fo) * temper[tidp+shft_rd];
			}

		}

		//Split part.  This exhibits thread divergence and is suboptimal.
		//So it's ripe to be improved.

		else
		{
			if (tidp <= ((THREADBLK+1)-k) && tidp >= k)
			{
				if (tid == (height-1))
				{
					temper[tidp + (base*shft_wr)] = 2.f * fo * (temper[tid+shft_rd]-temper[tid+shft_rd+1]) + temper[tidp + shft_rd];
				}
				else if (tid == height)
				{
					temper[tidp + (base*shft_wr)] = 2.f * fo * (temper[tid+shft_rd+2]-temper[tid+shft_rd+1]) + temper[tidp + shft_rd];
				}
				else
				{
					temper[tidp + (base*shft_wr)] = fo * (temper[tid+shft_rd] + temper[tid+shft_rd+2]) + (1.f-2.f*fo) * temper[tidp +shft_rd];
				}
			}

		}

		if (tid < 2)
		{
			temper[tid+(k-2)+shft_wr*base] = shLeft[itr+tid];
			temper[tid+(base-k)+shft_wr*base] = shRight[itr+tid];
			itr += 2;
		}

		__syncthreads();

	}

	if (blockIdx.x > 0)
	{
		temper[tidp] = fo * (temper[tid+base] + temper[tid+base+2]) + (1.f-2.f*fo) * temper[tidp+base];
	}
	else
	{
		if (tid == (height-1))
		{
			temper[tidp] = 2.f * fo * (temper[tid+base]-temper[tid+base+1]) + temper[tidp+base];
		}
		else if (tid == height)
		{
			temper[tidp] = 2.f * fo * (temper[tid+base+2]-temper[tid+base+1]) + temper[tidp+base];
		}
		else
		{
			temper[tidp] = fo * (temper[tid+base] + temper[tid+base+2]) + (1.f-2.f*fo) * temper[tidp+base];
		}

	}

	//Now fill the global unified timestep variable with the final calculated
	//temperatures.

	//Blocks 1 to end hold values 16 to end-16.
	if (blockIdx.x > 0)
	{
		//True if it ends on the first row! The first and last of temper on the final row are empty.
		IC[gid - height] = temper[tidp];
	}
	//Block 0 holds values 0 to 15 and end-15 to end.  In that order.
	else
	{
		if (tid >= height)
		{
			IC[gid - (height)] = temper[tidp];
		}
		else
		{
			IC[(blockDim.x * gridDim.x) + (tid - height) ] = temper[tidp];
		}
	}
}

//The host routine.
int main( int argc, char *argv[])
{
	if (argc != 6)
	{
		cout << "The Program takes five inputs, #Divisions, #Threads/block, dt, finish time, and which algorithm" << endl;
		exit(-1);
	}

	// Choose the GPGPU.  This is device 0 in my machine which has 2 devices.
	hipSetDevice(0);

	int dv = atoi(argv[1]); //Setting it to an int helps with arrays
	const int tpb = atoi(argv[2]);
	const int tf = atoi(argv[4]);
	const int bks = dv/tpb; //The number of blocks since threads/block = 32.
	const REAL lx = 5.0*DIVISIONS/1024;
	const REAL ds = lx/((double)DIVISIONS-1.0); //The x division length.
	REAL fou = TS*TH_DIFF/(ds*ds); //The Fourier number.

	//Initialize arrays.
	REAL IC[dv];
	REAL T_final[dv];
	REAL *d_IC, *d_right, *d_left;

	// Some initial condition for the bar temperature, an exponential decay
	// function.
	for (int k = 0; k<dv; k++)
	{
		IC[k] = 500.f*expf((-ds*k)/lx);
	}

	//cout << fou << endl;
	// Call out the file before the loop and write out the initial condition.
	ofstream fwr;
	ofstream ftime;
	ftime.open("1DSweptTiming.txt",ios::app);
	fwr.open("1DHeatEQResult.dat",ios::trunc);
	// Write out x length and then delta x and then delta t.
	// First item of each line is timestamp.
	fwr << lx << " " << DIVISIONS << " " << TS << " " << endl << 0 << " ";

	for (int k = 0; k<dv; k++)
	{
		fwr << IC[k] << " ";
	}

	fwr << endl;

	// Transfer data to GPU.

	// This puts the Fourier number in constant memory.
	hipMemcpyToSymbol(HIP_SYMBOL(fo),&fou,sizeof(REAL));

	// This initializes the device arrays on the device in global memory.
	// They're all the same size.  Conveniently.
	hipMalloc((void **)&d_IC, sizeof(REAL)*dv);
	hipMalloc((void **)&d_right, sizeof(REAL)*dv);
	hipMalloc((void **)&d_left, sizeof(REAL)*dv);

	//Copy the initial conditions to the device array.
	hipMemcpy(d_IC,IC,sizeof(REAL)*dv,hipMemcpyHostToDevice);

	// Start the counter and start the clock.
	REAL t_eq = 0.;
	REAL t_fullstep = TS*(THREADBLK);
	hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

	// Call the kernels until you reach the iteration limit.
	while(t_eq < FINISH)
	{
		upTriangle <<< bks,THREADBLK >>>(d_IC,d_right,d_left);
		downTriangle <<< bks,THREADBLK >>>(d_IC,d_right,d_left);
		t_eq += t_fullstep;
	}

	//Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime( &timed, start, stop);

	ftime << timed << endl;
	cout << "That took: " << timed << " seconds" << endl;


	hipMemcpy(T_final, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);
	fwr << t_eq << " ";
	for (int k = 0; k<dv; k++)
	{
		fwr << T_final[k] << " ";
	}

	fwr.close();

	//Free the memory and reset the device.
	hipFree(d_IC);
	hipFree(d_right);
	hipFree(d_left);
	hipDeviceReset();

	return 0;
}
