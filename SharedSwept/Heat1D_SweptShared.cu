#include "hip/hip_runtime.h"
/* This file is the current iteration of research being done to implement the
swept rule for Partial differential equations in one dimension.  This research
is a collaborative effort between teams at MIT, Oregon State University, and
Purdue University.

Copyright (C) 2015 Kyle Niemeyer, niemeyek@oregonstate.edu AND
Daniel Magee, mageed@oregonstate.edu

This program is free software: you can redistribute it and/or modify
it under the terms of the MIT license.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.

You should have received a copy of the MIT license
along with this program.  If not, see <https://opensource.org/licenses/MIT>.
*/

//COMPILE LINE:
// nvcc -o ./bin/HeatOut Heat1D_SweptShared.cu -gencode arch=compute_35,code=sm_35 -lm -restrict -Xcompiler -fopenmp


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>

#ifndef REAL
#define REAL  float
#endif

using namespace std;

__constant__ REAL fo;

REAL fou;

const REAL th_diff = 8.418e-5;

const REAL ds = .001;

__host__ __device__ REAL initFun(int xnode, REAL ds, REAL lx)
{
    REAL a = ((REAL)xnode*ds);
    return 100.f*a*(1.f-a/lx);
}

__device__ REAL execFunc(REAL tLeft, REAL tRight, REAL tCenter)
{
    return fo*(tLeft+tRight) + (1.f-2.f*fo)*tCenter;
}

__host__ REAL execFuncHost(REAL tLeft, REAL tRight, REAL tCenter)
{
    return fou*(tLeft+tRight) + (1.f-2.f*fou)*tCenter;
}

__global__
void
swapKernel(const REAL *passing_side, REAL *bin, int direction)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
    int lastidx = ((blockDim.x*gridDim.x)-1);
    int gidout = (gid + direction*blockDim.x) & lastidx;

    bin[gidout] = passing_side[gid];

}

__global__
void
classicHeat(REAL *heat_in, REAL *heat_out)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
    int lastidx = ((blockDim.x*gridDim.x)-1);
    if (gid == 0)
    {
        heat_out[gid] = execFunc(heat_in[gid+1],heat_in[gid+1],heat_in[gid]);
    }
    else if (gid == lastidx)
    {
        heat_out[gid] = execFunc(heat_in[gid-1],heat_in[gid-1],heat_in[gid]);
    }
    else
    {
        heat_out[gid] = execFunc(heat_in[gid-1],heat_in[gid+1],heat_in[gid]);
    }
}

__global__
void
upTriangle(const REAL *IC, REAL *right, REAL *left)
{

	extern __shared__ REAL temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tid = threadIdx.x; //Block Thread ID
    int tidp = tid + 1;
	int tidm = tid - 1;
	int shft_wr; //Initialize the shift to the written row of temper.
	int shft_rd; //Initialize the shift to the read row (opposite of written)
	int leftidx = tid/2 + ((tid/2 & 1) * blockDim.x) + (tid & 1);
	int rightidx = (blockDim.x - 2) + ((tid/2 & 1) * blockDim.x) + (tid & 1) -  tid/2;

    //Assign the initial values to the first row in temper, each warp (in this
	//case each block) has it's own version of temper shared among its threads.
	temper[tid] = IC[gid];

    __syncthreads();

	//The initial conditions are timslice 0 so start k at 1.

	for (int k = 1; k<(blockDim.x/2); k++)
	{
		//Bitwise even odd. On even iterations write to first row.
		shft_wr = blockDim.x * (k & 1);
		//On even iterations write to second row (starts at element 32)
		shft_rd = blockDim.x * ((k + 1) & 1);

		//Each iteration the triangle narrows.  When k = 1, 30 points are
		//computed, k = 2, 28 points.
		if (tid < (blockDim.x-k) && tid >= k)
		{
			temper[tid + shft_wr] = execFunc(temper[tidm+shft_rd], temper[tidp+shft_rd], temper[tid+shft_rd]);
		}

		//Make sure the threads are synced
		__syncthreads();

	}

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];

}

// Down triangle is only called at the end when data is passed left.  It's never split.
// It returns IC which is a full 1D result at a certain time.
__global__
void
downTriangle(REAL *IC, const REAL *right, const REAL *left)
{
	extern __shared__ REAL temper[];

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tid1 = tid + 1;
	int tid2 = tid + 2;
	int base = blockDim.x + 2;
	int height = base/2;
	int shft_rd;
	int shft_wr;

	int leftidx = height - tid/2 + ((tid/2 & 1) * base) + (tid & 1) - 2;
	int rightidx = height + tid/2 + ((tid/2 & 1) * base) + (tid & 1);
    int lastidx = ((blockDim.x*gridDim.x)-1);

	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.

	temper[leftidx] = right[gid];
	temper[rightidx] = left[gid];

    __syncthreads();
    //k needs to insert the relevant left right values around the computed values
	//every timestep.  Since it grows larger the loop is reversed.

	for (int k = (height-1); k>1; k--)
	{
		// This tells you if the current row is the first or second.
		shft_wr = base * ((k+1) & 1);
		// Read and write are opposite rows.
		shft_rd = base * (k & 1);

		if (tid1 < (base-k) && tid1 >= k)
		{
			temper[tid1 + shft_wr] = execFunc(temper[tid+shft_rd], temper[tid2+shft_rd], temper[tid1+shft_rd]);
		}
        __syncthreads();
	}

    if (gid == 0)
    {
        temper[tid] = execFunc(temper[tid2+base], temper[tid2+base], temper[tid1+base]);
    }
    else if (gid == lastidx)
    {
        temper[tid] = execFunc(temper[tid+base], temper[tid+base], temper[tid1+base]);
    }
    else
    {
        temper[tid] = execFunc(temper[tid+base], temper[tid2+base], temper[tid1+base]);
    }
    __syncthreads();

    IC[gid] = temper[tid];
}

//Full refers to whether or not there is a node run on the CPU.
__global__
void
wholeDiamond(REAL *right, REAL *left, bool full)
{
    extern __shared__ REAL temper[];

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
    int lastidx = ((blockDim.x*gridDim.x)-1);
	int tid1 = tid + 1;
	int tid2 = tid + 2;
	int base = blockDim.x + 2;
	int height = base/2;
	int shft_rd;
	int shft_wr;
	int leftidx = height - tid/2 + ((tid/2 & 1) * base) + (tid & 1) - 2;
	int rightidx = height + tid/2 + ((tid/2 & 1) * base) + (tid & 1);


    //if (blockIdx.x > (gridDim.x-3)) printf("gid: %i, gidin: %i \n",gid,gidin);
	// Initialize temper.

    if (full)
    {
        temper[leftidx] = right[gid];
        temper[rightidx] = left[gid];
    }
    else
    {
        gid += blockDim.x;
        temper[leftidx] = right[gid];
        temper[rightidx] = left[gid];
    }

    __syncthreads();

	for (int k = (height-1); k>1; k--)
	{
        // This tells you if the current row is the first or second.
		shft_wr = base * ((k+1) & 1);
		// Read and write are opposite rows.
		shft_rd = base * (k & 1);

        if (tid1 < (base-k) && tid1 >= k)
		{
			temper[tid1 + shft_wr] = execFunc(temper[tid+shft_rd], temper[tid2+shft_rd], temper[tid1+shft_rd]);
		}
        __syncthreads();
	}

    //Boundary Conditions!
    if (full)
    {
        if (gid == 0)
        {
            temper[tid] = execFunc(temper[tid2+base], temper[tid2+base], temper[tid1+base]);
        }
        else if (gid == lastidx)
        {
            temper[tid] = execFunc(temper[tid+base], temper[tid+base], temper[tid1+base]);
        }
        else
        {
            temper[tid] = execFunc(temper[tid+base], temper[tid2+base], temper[tid1+base]);
        }
    }
    else
    {
        temper[tid] = execFunc(temper[tid+base], temper[tid2+base], temper[tid1+base]);
    }

    __syncthreads();

    // Then make sure each block of threads are synced.

    //-------------------TOP PART------------------------------------------

    leftidx = tid/2 + ((tid/2 & 1) * blockDim.x) + (tid & 1);
    rightidx = (blockDim.x - 2) + ((tid/2 & 1) * blockDim.x) + (tid & 1) -  tid/2;

    int tidm = tid - 1;

    height -= 1;
	//The initial conditions are timeslice 0 so start k at 1.

    for (int k = 1; k<height; k++)
	{
		//Bitwise even odd. On even iterations write to first row.
		shft_wr = blockDim.x * (k & 1);
		//On even iterations write to second row (starts at element 32)
		shft_rd = blockDim.x * ((k + 1) & 1);

		//Each iteration the triangle narrows.  When k = 1, 30 points are
		//computed, k = 2, 28 points.
		if (tid < (blockDim.x-k) && tid >= k)
		{
			temper[tid + shft_wr] = execFunc(temper[tidm+shft_rd], temper[tid1+shft_rd], temper[tid+shft_rd]);
		}

		//Make sure the threads are synced
		__syncthreads();

	}
    right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];
}

//Split one is always first.  Passing left like the downTriangle.  downTriangle
//should be rewritten so it isn't split.  Only write on a non split pass.
__global__
void
splitDiamond(REAL *right, REAL *left)
{
    extern __shared__ REAL temper[];

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tid1 = tid + 1;
	int tid2 = tid + 2;
	int base = blockDim.x + 2;
	int height = base/2;
	int shft_rd;
	int shft_wr;
	int leftidx = height - tid/2 + ((tid/2 & 1) * base) + (tid & 1) - 2;
	int rightidx = height + tid/2 + ((tid/2 & 1) * base) + (tid & 1);

	// Initialize temper.

    temper[leftidx] = right[gid];
	temper[rightidx] = left[gid];

    //Wind it up!

    __syncthreads();

    for (int k = (height-1); k>0; k--)
    {
        // This tells you if the current row is the first or second.
        shft_wr = base * ((k+1) & 1);
        // Read and write are opposite rows.
        shft_rd = base * (k & 1);

        //Block 0 is split so it needs a different algorithm.  This algorithm
        //is slightly different than top triangle as described in the note above.
        if (blockIdx.x > 0)
        {
            if (tid1 < (base-k) && tid1 >= k)
            {
                temper[tid1 + shft_wr] = execFunc(temper[tid+shft_rd], temper[tid2+shft_rd], temper[tid1+shft_rd]);
            }

        }

        else
        {
            if (tid1 < (base-k) && tid1 >= k)
            {
                if (tid1 == (height-1))
                {
                    temper[tid1 + shft_wr] = execFunc(temper[tid+shft_rd], temper[tid+shft_rd], temper[tid1+shft_rd]);
                }
                else if (tid1 == height)
                {
                    temper[tid1 + shft_wr] = execFunc(temper[tid2+shft_rd], temper[tid2+shft_rd], temper[tid1+shft_rd]);
                }
                else
                {
                    temper[tid1 + shft_wr] = execFunc(temper[tid+shft_rd], temper[tid2+shft_rd], temper[tid1+shft_rd]);
                }
            }

        }

        __syncthreads();
    }

    REAL trade = temper[tid1];
    __syncthreads();
    temper[tid] = trade;
    __syncthreads();

    //-------------------TOP PART------------------------------------------
    leftidx = tid/2 + ((tid/2 & 1) * blockDim.x) + (tid & 1);
    rightidx = (blockDim.x - 2) + ((tid/2 & 1) * blockDim.x) + (tid & 1) -  tid/2;

    int tidm = tid - 1;

    height--;

	for (int k = 1; k<height; k++)
	{
		//Bitwise even odd. On even iterations write to first row.
		shft_wr = blockDim.x * (k & 1);
		//On even iterations write to second row (starts at element 32)
		shft_rd = blockDim.x * ((k + 1) & 1);

		//Each iteration the triangle narrows.  When k = 1, 30 points are
		//computed, k = 2, 28 points.
        if (blockIdx.x > 0)
        {
            if (tid < (blockDim.x-k) && tid >= k)
    		{
    			temper[tid + shft_wr] = execFunc(temper[tidm + shft_rd], temper[tid1 + shft_rd], temper[tid + shft_rd]);
    		}
        }
        else
        {
            if (tid < (blockDim.x-k) && tid >= k)
            {
                if (tid == (height-1))
                {
                    temper[tid + shft_wr] = execFunc(temper[tidm + shft_rd], temper[tidm + shft_rd], temper[tid + shft_rd]);

                }
                else if (tid == height)
                {
                    temper[tid + shft_wr] = execFunc(temper[tid1 + shft_rd], temper[tid1 + shft_rd], temper[tid + shft_rd]);
                }
                else
                {
                    temper[tid + shft_wr] = execFunc(temper[tidm + shft_rd], temper[tid1 + shft_rd], temper[tid + shft_rd]);
                }
            }
        }

		//Make sure the threads are synced
		__syncthreads();
    }

	right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];
}


__host__
void
CPU_diamond(REAL *temper, int tpb)
{
    int bck, fwd, shft_rd, shft_wr;
    int base = tpb + 2;
    int ht = tpb/2;

    //Splitting it is the whole point!
    for (int k = ht; k>0; k--)
    {
        // This tells you if the current row is the first or second.
        shft_wr = base * ((k+1) & 1);
        // Read and write are opposite rows.
        shft_rd = base * (k & 1);

        for(int n = k; n<(base-k); n++)
        {
            bck = n - 1;
            fwd = n + 1;
            //Double trailing index.
            if(n == ht)
            {
                temper[n + shft_wr] = execFuncHost(temper[bck+shft_rd], temper[bck+shft_rd], temper[n+shft_rd]);
            }
            //Double leading index.
            else if(n == ht+1)
            {
                temper[n + shft_wr] = execFuncHost(temper[fwd+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
            }
            else
            {
                temper[n + shft_wr] = execFuncHost(temper[bck+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
            }
        }
    }

    for (int k = 0; k<tpb; k++) temper[k] = temper[k+1];
    //Top part.
    ht--;
    for (int k = 1; k<ht; k++)
    {
        // This tells you if the current row is the first or second.
        shft_wr = tpb * (k & 1);
        // Read and write are opposite rows.
        shft_rd = tpb * ((k+1) & 1);

        for(int n = k; n<(tpb-k); n++)
        {
            bck = n - 1;
            fwd = n + 1;
            //Double trailing index.
            if(n == ht)
            {
                temper[n + shft_wr] = execFuncHost(temper[bck+shft_rd], temper[bck+shft_rd], temper[n+shft_rd]);
            }
            //Double leading index.
            else if(n == ht+1)
            {
                temper[n + shft_wr] = execFuncHost(temper[fwd+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
            }
            else
            {
                temper[n + shft_wr] = execFuncHost(temper[bck+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
            }
        }
    }
}

//Classic Discretization wrapper.
double
classicWrapper(const int bks, int tpb, const int dv, const REAL dt, const float t_end,
    REAL *IC, REAL *T_f, const float freq, ofstream &fwr)
{
    REAL *dheat_in, *dheat_out;

    hipMalloc((void **)&dheat_in, sizeof(REAL)*dv);
    hipMalloc((void **)&dheat_out, sizeof(REAL)*dv);

    // Copy the initial conditions to the device array.
    hipMemcpy(dheat_in,IC,sizeof(REAL)*dv,hipMemcpyHostToDevice);

    double t_eq = 0.0;
    double twrite = freq;

    while (t_eq < t_end)
    {
        classicHeat <<< bks,tpb >>> (dheat_in, dheat_out);
        classicHeat <<< bks,tpb >>> (dheat_out, dheat_in);
        t_eq += 2*dt;

        if (t_eq > twrite)
        {
            hipMemcpy(T_f, dheat_in, sizeof(REAL)*dv, hipMemcpyDeviceToHost);
            fwr << " Temperature " << t_eq << " ";

            for (int k = 0; k<dv; k++)
            {
                fwr << T_f[k] << " ";
            }
            fwr << endl;

            twrite += freq;
        }
    }

    hipMemcpy(T_f, dheat_in, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

    hipFree(dheat_in);
    hipFree(dheat_out);

    return t_eq;

}

//The Swept Rule wrapper.
double
sweptWrapper(const int bks, int tpb, const int dv, const REAL dt, const float t_end, const int cpu,
    REAL *IC, REAL *T_f, const float freq, ofstream &fwr)
{
    const int base = (tpb + 2);
    const int ht = base/2;
    const size_t smem1 = 2*tpb*sizeof(REAL);
    const size_t smem2 = (base*2)*sizeof(REAL);

    int indices[4][tpb];
    for (int k = 0; k<tpb; k++)
    {
        indices[0][k] = ht - k/2 + ((k/2 & 1) * base) + (k & 1) - 2; //left
        indices[1][k] = ht + k/2 + ((k/2 & 1) * base) + (k & 1); //right

        indices[2][k] = k/2 + ((k/2 & 1) * tpb) + (k & 1); //left
        indices[3][k] = (tpb - 2) + ((k/2 & 1) * tpb) + (k & 1) -  k/2; //right
    }


	REAL *d_IC, *d_right, *d_left, *d_bin;

	hipMalloc((void **)&d_IC, sizeof(REAL)*dv);
	hipMalloc((void **)&d_right, sizeof(REAL)*dv);
	hipMalloc((void **)&d_left, sizeof(REAL)*dv);
    hipMalloc((void **)&d_bin, sizeof(REAL)*dv);

	// Copy the initial conditions to the device array.
	hipMemcpy(d_IC,IC,sizeof(REAL)*dv,hipMemcpyHostToDevice);
	// Start the counter and start the clock.
	const double t_fullstep = dt*(double)tpb;

	upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);

    swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
    swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

    double t_eq;
    double twrite = freq;


	// Call the kernels until you reach the iteration limit.

    if (cpu)
    {
        REAL *tmpr = (REAL*)malloc(smem2);
        REAL *h_right, *h_left;
        hipHostAlloc((void **) &h_right, tpb*sizeof(REAL), hipHostMallocDefault);
        hipHostAlloc((void **) &h_left, tpb*sizeof(REAL), hipHostMallocDefault);
        // h_right = (REALfour *) malloc(tpb*sizeof(REALfour));
        // h_left = (REALfour *) malloc(tpb*sizeof(REALfour));

        t_eq = t_fullstep;
        omp_set_num_threads( 2 );

        //Split Diamond Begin------

        hipMemcpy(h_right, d_left, tpb*sizeof(REAL), hipMemcpyDeviceToHost);
        hipMemcpy(h_left, d_right, tpb*sizeof(REAL), hipMemcpyDeviceToHost);

        #pragma omp parallel sections
        {
        #pragma omp section
        {
            for (int k = 0; k<tpb; k++)
            {
                tmpr[indices[0][k]] = h_left[k];
                tmpr[indices[1][k]] = h_right[k];
            }

            CPU_diamond(tmpr, tpb);

            for (int k = 0; k<tpb; k++)
            {
                h_left[k] = tmpr[indices[2][k]];
                h_right[k] = tmpr[indices[3][k]];
            }
        }
        #pragma omp section
        {
            wholeDiamond <<< bks-1,tpb,smem2 >>>(d_right,d_left,false);
        }
        }

        hipMemcpy(d_right, h_right, tpb*sizeof(REAL), hipMemcpyHostToDevice);
        hipMemcpy(d_left, h_left, tpb*sizeof(REAL), hipMemcpyHostToDevice);

        swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
        swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

        //Split Diamond End------

    	while(t_eq < t_end)
    	{

            wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,true);

            swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
            swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

            //Split Diamond Begin------

            hipMemcpy(h_right,d_left, tpb*sizeof(REAL), hipMemcpyDeviceToHost);
            hipMemcpy(h_left, d_right, tpb*sizeof(REAL), hipMemcpyDeviceToHost);
            #pragma omp parallel sections
            {
            #pragma omp section
            {


                for (int k = 0; k<tpb; k++)
                {
                    tmpr[indices[0][k]] = h_left[k];
                    tmpr[indices[1][k]] = h_right[k];
                }

                CPU_diamond(tmpr, tpb);

                for (int k = 0; k<tpb; k++)
                {
                    h_left[k] = tmpr[indices[2][k]];
                    h_right[k] = tmpr[indices[3][k]];
                }
            }
            #pragma omp section
            {
                wholeDiamond <<< bks-1,tpb,smem2 >>>(d_right,d_left,false);

            }
            }

            hipMemcpy(d_right, h_right, tpb*sizeof(REAL), hipMemcpyHostToDevice);
            hipMemcpy(d_left, h_left, tpb*sizeof(REAL), hipMemcpyHostToDevice);

            swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
            swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

            //Split Diamond End------

		    //So it always ends on a left pass since the down triangle is a right pass.

		    t_eq += t_fullstep;

    		/* Since the procedure does not store the temperature values, the user
    		could input some time interval for which they want the temperature
    		values and this loop could copy the values over from the device and
    		write them out.  This way the user could see the progression of the
    		solution over time, identify an area to be investigated and re-run a
    		shorter version of the simulation starting with those intiial conditions.
            */

            if (t_eq > twrite)
    		{
    			downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);

    			hipMemcpy(T_f, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

    			fwr << "Temperature " << t_eq << " ";

    			for (int k = 0; k<dv; k++)
    			{
    				fwr << T_f[k] << " ";
    			}
    			fwr << endl;

                upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);

                swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
                swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

    			splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);

                swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
                swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

                t_eq += t_fullstep;

    			twrite += freq;
    		}
        }
        hipHostFree(h_right);
        hipHostFree(h_left);
        // free(h_right);
        // free(h_left);
        free(tmpr);
	}
    else
    {
        splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);
        t_eq = t_fullstep;
        swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
        swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

        while(t_eq < t_end)
        {

            wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,true);

            swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
            swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

            splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);

            swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
            swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

            //So it always ends on a left pass since the down triangle is a right pass.
            t_eq += t_fullstep;

            if (t_eq > twrite)
    		{
    			downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);

    			hipMemcpy(T_f, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);
    			fwr << "Temperature " << t_eq << " ";

    			for (int k = 0; k<dv; k++)
    			{
    				fwr << T_f[k] << " ";
    			}
    			fwr << endl;

    			upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);

                swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
                swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

    			splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);

                swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
                swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

                t_eq += t_fullstep;

    			twrite += freq;
    		}
        }
    }

	downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);

	hipMemcpy(T_f, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

	hipFree(d_IC);
	hipFree(d_right);
	hipFree(d_left);
    hipFree(d_bin);

    return t_eq;
}

int main( int argc, char *argv[] )
{
    //That is there are less than 8 arguments.
    if (argc < 9)
	{
		cout << "The Program takes 9 inputs, #Divisions, #Threads/block, deltat, finish time, output frequency..." << endl;
        cout << "Classic/Swept, CPU sharing Y/N, Variable Output File, Timing Output File (optional)" << endl;
		exit(-1);
	}

	// Choose the GPGPU.  This is device 0 in my machine which has 2 devices.
	hipSetDevice(0);

    int dv = atoi(argv[1]); //Number of spatial points
	const int tpb = atoi(argv[2]); //Threads per Blocks
    const float dt =  atof(argv[3]);
	const float tf = atof(argv[4]); //Finish time
    const float freq = atof(argv[5]);
    const int scheme = atoi(argv[6]); //1 for Swept 0 for classic
    const int share = atoi(argv[7]);
	const int bks = dv/tpb; //The number of blocks
    const REAL lx = ds * ((REAL)dv - 1.f);
    fou = th_diff*dt/(ds*ds);  //Fourier number

    cout << bks << " Blocks " << lx << " Length" << " Type (float = 4, double = 8) " << sizeof(REAL) << endl;

	//dv and tpb must be powers of two.  dv must be larger than tpb and divisible by
	//tpb.

	if ((dv & (tpb-1) !=0) || (tpb&31) != 0)
    {
        cout << "INVALID NUMERIC INPUT!! "<< endl;
        cout << "2nd ARGUMENT MUST BE A POWER OF TWO >= 32 AND FIRST ARGUMENT MUST BE DIVISIBLE BY SECOND" << endl;
        exit(-1);
    }

	// Initialize arrays.
    REAL *IC, *T_final;
	hipHostAlloc((void **) &IC, dv*sizeof(REAL), hipHostMallocDefault);
	hipHostAlloc((void **) &T_final, dv*sizeof(REAL), hipHostMallocDefault);

    // IC = (REAL *) malloc(dv*sizeof(REAL));
    // T_final = (REAL *) malloc(dv*sizeof(REAL));

	// Some initial condition for the bar temperature, an exponential decay
	// function.
	for (int k = 0; k<dv; k++)
	{
		IC[k] = initFun(k, ds, lx);
	}

	// Call out the file before the loop and write out the initial condition.
	ofstream fwr;
	fwr.open(argv[8],ios::trunc);
	// Write out x length and then delta x and then delta t.
	// First item of each line is timestamp.
	fwr << lx << " " << dv << " " << ds << " " << endl << "Temperature " << 0 << " ";

	for (int k = 0; k<dv; k++)
	{
		fwr << IC[k] << " ";
	}

	fwr << endl;

    //Transfer data to GPU.
	// This puts the Fourier number in constant memory.
	hipMemcpyToSymbol(HIP_SYMBOL(fo),&fou,sizeof(REAL));

	// This initializes the device arrays on the device in global memory.
	// They're all the same size.  Conveniently.

	// Start the counter and start the clock.
	hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

    // Call the kernels until you reach the iteration limit.
	double tfm;
    if (scheme)
    {
        tfm = sweptWrapper(bks, tpb, dv, dt, tf, share, IC, T_final, freq, fwr);
    }
    else
    {
        tfm = classicWrapper(bks, tpb, dv, dt, tf, IC, T_final, freq, fwr);
    }

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime( &timed, start, stop);

	timed *= 1.e3;

    double n_timesteps = tfm/dt;

    double per_ts = timed/n_timesteps;

    cout << n_timesteps << " timesteps" << endl;
	cout << "Averaged " << per_ts << " microseconds (us) per timestep" << endl;

    if (argc>7)
    {
        ofstream ftime;
        ftime.open(argv[9],ios::app);
    	ftime << dv << "\t" << tpb << "\t" << per_ts << endl;
    	ftime.close();
    }
	fwr << "Temperature " << tfm << " ";
	for (int k = 0; k<dv; k++)
	{
		fwr << T_final[k] << " ";
	}

	fwr.close();

	// Free the memory and reset the device.

	hipEventDestroy( start );
	hipEventDestroy( stop );
    hipDeviceReset();
    hipHostFree(IC);
    hipHostFree(T_final);
    // free(IC);
    // free(T_final);

	return 0;

}

//END
