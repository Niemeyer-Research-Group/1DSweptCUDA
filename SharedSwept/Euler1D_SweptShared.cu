#include "hip/hip_runtime.h"
//


//COMPILE LINE:
// nvcc -o ./bin/EulerOut Euler1D_SweptShared.cu -gencode arch=compute_35,code=sm_35 -lm -w -std=c++11 -Xcompiler -fopenmp

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_vector_types.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>

#define REAL        float
#define REALfour    float4
#define REALthree   float3

const REAL gam = 1.4;
const REAL m_gamma = 0.4;
const REAL dx = 0.5;

REALfour bd[2];
REALthree dimz;
//dbd is the boundary condition
__constant__ REALfour dbd[2]; //0 is left 1 is right.
//dimens has three fields x is dt/dx, y is gamma, z is gamma-1
__constant__ REALthree dimens;

//Calculates the pressure at the current node with the rho, u, e state variables.
__device__ __host__
__forceinline__
REAL
pressure(REALfour current)
{
    #ifdef __CUDA_ARCH__
    return dimens.z * (current.z - (0.5 * current.y * current.y/current.x));
    #else
    return dimz.z * (current.z - (0.5 * current.y * current.y/current.x));
    #endif
}

//Calculates the pressure ratio between the right and left side pressure differences.
//(pRight-pCurrent)/(pCurrent-pLeft)
__device__ __host__
__forceinline__
REAL
pressureRatio(REAL cvLeft, REAL cvCenter, REAL cvRight)
{
    return (cvRight- cvCenter)/(cvCenter- cvLeft);
}

//Reconstructs the state variables if the pressure ratio is finite and positive.
//I think it's that internal boundary condition.
__device__ __host__
REALfour
limitor(REALthree cvCurrent, REALthree cvOther, REAL pRatio)
{
    #ifdef __CUDA_ARCH__
    if (isfinite(pRatio) && pRatio > 0) //If it's finite and positive
    {
        REAL fact = (pRatio < 1) ? pRatio : 1.f;
        return make_float4(cvCurrent + 0.5* fact * (cvOther - cvCurrent));

    }
    #else
    if (std::isfinite(pRatio) && pRatio > 0) //If it's finite and positive
    {
        REAL fact = (pRatio < 1) ? pRatio : 1.f;
        return make_float4(cvCurrent + 0.5* fact * (cvOther - cvCurrent));

    }
    #endif

    return make_float4(cvCurrent);

}

//Left and Center then Left and right.
//This is the meat of the flux calculation.  Fields: x is rho, y is u, z is e, w is p.
__device__ __host__
REALthree
eulerFlux(REALfour cvLeft, REALfour cvRight)
{
    #ifndef __CUDA_ARCH__
    using namespace std;
    #endif
    //For the first calculation rho and p remain the same.
    REALthree flux;
    REAL spectreRadius;
    REAL uLeft = cvLeft.y/cvLeft.x;
    REAL uRight = cvRight.y/cvRight.x;
    REAL eLeft = cvLeft.z/cvLeft.x;
    REAL eRight = cvRight.z/cvRight.x;

    flux.x = 0.5 * (cvLeft.x*uLeft + cvRight.x*uRight);
    flux.y = 0.5 * (cvLeft.x*uLeft*uLeft + cvRight.x*uRight*uRight + cvLeft.w + cvRight.w);
    flux.z = 0.5 * (cvLeft.x*uLeft*eLeft + cvRight.x*uRight*eRight + uLeft*cvLeft.w + uRight*cvRight.w);

    REALfour halfState;
    REAL rhoLeftsqrt = sqrtf(cvLeft.x); REAL rhoRightsqrt = sqrtf(cvRight.x);
    halfState.x = rhoLeftsqrt * rhoRightsqrt;
    halfState.y = (rhoLeftsqrt*uLeft + rhoRightsqrt*uRight)/(rhoLeftsqrt+rhoRightsqrt);
    halfState.z = (rhoLeftsqrt*eLeft + rhoRightsqrt*eRight)/(rhoLeftsqrt+rhoRightsqrt); //Seems to be unnecessary.
    halfState.w = pressure(halfState);

    #ifdef __CUDA_ARCH__
    spectreRadius = sqrtf(dimens.y * halfState.w/halfState.x) + fabs(halfState.y);
    #else
    spectreRadius = sqrtf(dimz.y * halfState.w/halfState.x) + fabs(halfState.y);
    #endif

    flux += 0.5 * spectreRadius * (make_float3(cvLeft) - make_float3(cvRight));

    return flux;

}

//This is the predictor step of the finite volume scheme.
__device__ __host__
REALfour
eulerStutterStep(REAL pfarLeft, REALfour stateLeft, REALfour stateCenter, REALfour stateRight, REAL pfarRight)
{
    REALthree fluxL, fluxR, pR;
    REALfour tempStateLeft, tempStateRight;

    //Get the pressure ratios as a structure.
    pR = make_float3(pressureRatio(pfarLeft,stateLeft.w,stateCenter.w),
        pressureRatio(stateLeft.w,stateCenter.w,stateRight.w),
        pressureRatio(stateCenter.w,stateRight.w,pfarRight));

    //This is the temporary state bounded by the limitor function.
    tempStateLeft = limitor(make_float3(stateLeft), make_float3(stateCenter), pR.x);
    tempStateRight = limitor(make_float3(stateCenter), make_float3(stateLeft), 1.0/pR.y);

    //Pressure needs to be recalculated for the new limited state variables.
    tempStateLeft.w = pressure(tempStateLeft);
    tempStateRight.w = pressure(tempStateRight);
    fluxL = eulerFlux(tempStateLeft,tempStateRight);

    //Do the same thing with the right side.
    tempStateLeft = limitor(make_float3(stateCenter), make_float3(stateRight), pR.y);
    tempStateRight = limitor(make_float3(stateRight), make_float3(stateCenter), 1.0/pR.z);
    tempStateLeft.w = pressure(tempStateLeft);
    tempStateRight.w = pressure(tempStateRight);
    fluxR = eulerFlux(tempStateLeft,tempStateRight);

    //Add the change back to the node in question.
    #ifdef __CUDA_ARCH__
    stateCenter += make_float4(0.5 * dimens.x * (fluxL-fluxR));
    #else
    stateCenter += make_float4(0.5 * dimz.x * (fluxL-fluxR));
    #endif
    stateCenter.w = pressure(stateCenter);

    return stateCenter;
}

//Same thing as the predictor step, but this final step adds the result to the original state variables to advance to the next timestep.
//But the predictor variables to find the fluxes.
__device__ __host__
REALfour
eulerFinalStep(REAL pfarLeft, REALfour stateLeft, REALfour stateCenter, REALfour stateCenter_orig, REALfour stateRight, REAL pfarRight)
{
    REALthree fluxL, fluxR, pR;
    REALfour tempStateLeft, tempStateRight;

    pR = make_float3(pressureRatio(pfarLeft,stateLeft.w,stateCenter.w),
        pressureRatio(stateLeft.w,stateCenter.w,stateRight.w),
        pressureRatio(stateCenter.w,stateRight.w,pfarRight));

    tempStateLeft = limitor(make_float3(stateLeft), make_float3(stateCenter), pR.x);
    tempStateRight = limitor(make_float3(stateCenter), make_float3(stateLeft), 1.0/pR.y);
    tempStateLeft.w = pressure(tempStateLeft);
    tempStateRight.w = pressure(tempStateRight);
    fluxL = eulerFlux(tempStateLeft,tempStateRight);

    tempStateLeft = limitor(make_float3(stateCenter), make_float3(stateRight), pR.y);
    tempStateRight = limitor(make_float3(stateRight), make_float3(stateCenter), 1.0/pR.z);
    tempStateLeft.w = pressure(tempStateLeft);
    tempStateRight.w = pressure(tempStateRight);
    fluxR = eulerFlux(tempStateLeft,tempStateRight);

    #ifdef __CUDA_ARCH__
    stateCenter_orig += make_float4(0.5 * dimens.x * (fluxL-fluxR));
    #else
    stateCenter_orig += make_float4(0.5 * dimz.x * (fluxL-fluxR));
    #endif
    stateCenter_orig.w = pressure(stateCenter_orig);

    return stateCenter_orig;
}

//Simple scheme with dirchlet boundary condition.
__global__
void
classicEuler(REALfour *euler_in, REALfour *euler_out)
{

    int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
    int lastidx = ((blockDim.x*gridDim.x)-1);

    REALfour temp[5];
    REALfour persist = euler_in[gid];

    #pragma unroll
	for (int k = -2; k<3; k++)
	{
		temp[k+2] = euler_in[gid+k];
	}

    if (gid == 0)
    {
        euler_out[gid] = dbd[0];
    }
    else if (gid == lastidx)
    {
        euler_out[gid] = dbd[1];
    }
    else if (gid == 1)
    {
        euler_out[gid] = eulerStutterStep(dbd[0].w,dbd[0],temp[2],temp[3],temp[4].w);
    }
    else if (gid == (lastidx-1))
    {
        euler_out[gid] = eulerStutterStep(temp[0].w,temp[1],temp[2],dbd[1],dbd[1].w);
    }
    else
    {
        euler_out[gid] = eulerStutterStep(temp[0].w,temp[1],temp[2],temp[3],temp[4].w);
    }

    __syncthreads();

    #pragma unroll
    for (int k = -2; k<3; k++)
    {
        temp[k+2] = euler_out[gid+k];
    }

    if (gid == 0)
    {
        euler_out[gid] = dbd[0];
    }
    else if (gid == lastidx)
    {
        euler_out[gid] = dbd[1];
    }
    else if (gid == 1)
    {
        euler_out[gid] = eulerFinalStep(dbd[0].w,dbd[0],temp[2],persist,temp[3],temp[4].w);
    }
    else if (gid == (lastidx-1))
    {
        euler_out[gid] = eulerFinalStep(temp[0].w,temp[1],temp[2],persist,dbd[1],dbd[1].w);
    }
    else
    {
        euler_out[gid] = eulerFinalStep(temp[0].w,temp[1],temp[2],persist,temp[3],temp[4].w);
    }

}

__global__
void
upTriangle(REALfour *IC, REALfour *right, REALfour *left)
{

	extern __shared__ REALfour temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tid = threadIdx.x; //Block Thread ID

    int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tid + k + blockDim.x;
		tid_bottom[k+2] = tid + k;
	}

	int leftidx = ((tid/4 & 1) * blockDim.x) + (tid/4)*2 + (tid & 3);
	int rightidx = (blockDim.x - 4) + ((tid/4 & 1) * blockDim.x) + (tid & 3) - (tid/4)*2;

	int step2;

    //Assign the initial values to the first row in temper, each block
    //has it's own version of temper shared among its threads.
	temper[tid] = IC[gid];

	if (tid > 1 && tid <(blockDim.x-2))
	{
		temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
			temper[tid_bottom[3]], temper[tid_bottom[4]].w);
	}

	__syncthreads();

	//The initial conditions are timslice 0 so start k at 1.
	for (int k = 4; k<(blockDim.x/2); k+=4)
	{
		if (tid < (blockDim.x-k) && tid >= k)
		{
			temper[tid] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
				temper[tid], temper[tid_top[3]], temper[tid_top[4]].w);

		}

		step2 = k + 2;
		__syncthreads();

		if (tid < (blockDim.x-step2) && tid >= step2)
		{
			temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
				temper[tid_bottom[3]], temper[tid_bottom[4]].w);
		}

		//Make sure the threads are synced
		__syncthreads();

	}

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];


}

// Down triangle is only called at the end when data is passed left.  It's never split.
// It returns IC which is a full 1D result at a certain time.
__global__
void
downTriangle(REALfour *IC, REALfour *right, REALfour *left)
{
	extern __shared__ REALfour temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tididx = tid + 2;
	int base = blockDim.x + 4;
	int height = base/2;
    int height2 = height-2;
    int lastidx = ((blockDim.x*gridDim.x)-1);
	int step2;

	int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tididx + k + base;
		tid_bottom[k+2] = tididx + k;
	}

	int leftidx = height + ((tid/4 & 1) * base) + (tid & 3) - (4 + (tid/4) * 2);
	int rightidx = height + ((tid/4 & 1) * base) + (tid/4)*2 + (tid & 3);
	int gidin = (gid + blockDim.x) & lastidx;

	temper[leftidx] = right[gid];
	temper[rightidx] = left[gidin];

    if (gid == 0)
    {
        temper[base] = dbd[0];
        temper[base+1] = dbd[0];
    }
    if (gid == lastidx)
    {
        temper[2*base-1] = dbd[1];
        temper[2*base-2] = dbd[1];
    }


    if (tididx < (base-height2) && tididx >= height2)
    {
        temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
            temper[tid_bottom[3]], temper[tid_bottom[4]].w);
    }

    __syncthreads();

	for (int k = (height-4); k>4; k-=4)
	{

		if (tididx < (base-k) && tididx >= k)
		{
			temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
				temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
		}

        step2 = k-2;

        if (tididx < (base-step2) && tididx >= step2)
        {
            temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                temper[tid_bottom[3]], temper[tid_bottom[4]].w);

        }
		//Make sure the threads are synced
		__syncthreads();
	}

    if (gid == 0)
    {
        temper[tididx] = dbd[0];
    }
    else if (gid == lastidx)
    {
        temper[tididx] = dbd[1];
    }
    else
    {
        temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
            temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
    }


    IC[gid] = temper[tididx];
}

//Full refers to whether or not there is a node run on the CPU.
__global__
void
wholeDiamond(REALfour *right, REALfour *left, bool full)
{

    extern __shared__ REALfour temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tididx = tid + 2;
	int base = blockDim.x + 4;
	int height = base/2;
    int height2 = height-2;
	int step2;
    int lastidx = ((blockDim.x*gridDim.x)-1);

	int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tididx + k + base;
		tid_bottom[k+2] = tididx + k;
	}

	int leftidx = height + ((tid/4 & 1) * base) + (tid & 3) - (4 + (tid/4) * 2);
	int rightidx = height + ((tid/4 & 1) * base) + (tid/4)*2 + (tid & 3);
	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.

    if (full)
    {
        int gidin = (gid + blockDim.x) & lastidx;
        temper[leftidx] = right[gid];
        temper[rightidx] = left[gidin];
    }
    else
    {
        temper[leftidx] = right[gid];
        gid += blockDim.x;
        temper[rightidx] = left[gid];
    }

    if (gid < 2)
    {
        temper[base+gid] = dbd[0];
    }
    if (gid == lastidx)
    {
        temper[2*base-1] = dbd[1];
        temper[2*base-2] = dbd[1];
    }


    if (tididx < (base-height2) && tididx >= height2)
    {
        temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
            temper[tid_bottom[3]], temper[tid_bottom[4]].w);
    }

    __syncthreads();

    for (int k = (height-4); k>4; k-=4)
    {
        if (tididx < (base-k) && tididx >= k)
        {
            temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
        }

        step2 = k-2;

        if (tididx < (base-step2) && tididx >= step2)
        {
            temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                temper[tid_bottom[3]], temper[tid_bottom[4]].w);

        }
        //Make sure the threads are synced
        __syncthreads();
    }

    if (gid == 0)
    {
        temper[tididx] = dbd[0];
    }
    else if (gid == lastidx)
    {
        temper[tididx] = dbd[1];
    }
    else
    {
        temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
            temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
    }

    temper[tid] = temper[tididx];

    // -------------------TOP PART------------------------------------------

    leftidx = ((tid/4 & 1) * blockDim.x) + (tid/4)*2 + (tid & 3);
	rightidx = (blockDim.x - 4) + ((tid/4 & 1) * blockDim.x) + (tid & 3) - (tid/4)*2;

    #pragma unroll
    for (int k = -2; k<3; k++)
    {
        tid_top[k+2] = tid + k + blockDim.x;
        tid_bottom[k+2] = tid + k;
    }

    if (tid > 1 && tid <(blockDim.x-2))
	{
		temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
			temper[tid_bottom[3]], temper[tid_bottom[4]].w);
	}
	//The initial conditions are timeslice 0 so start k at 1.

	__syncthreads();

    //The initial conditions are timslice 0 so start k at 1.
	for (int k = 4; k<(blockDim.x/2); k+=4)
	{
		if (tid < (blockDim.x-k) && tid >= k)
		{
			temper[tid] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
				temper[tid], temper[tid_top[3]], temper[tid_top[4]].w);

		}

		step2 = k + 2;
		__syncthreads();

		if (tid < (blockDim.x-step2) && tid >= step2)
		{
			temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
				temper[tid_bottom[3]], temper[tid_bottom[4]].w);
		}

		//Make sure the threads are synced
		__syncthreads();

	}

    right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];

}

//Split one is always first.  Passing left like the downTriangle.  downTriangle
__global__
void
splitDiamond(REALfour *right, REALfour *left)
{
    extern __shared__ REALfour temper[];

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
    int tididx = tid + 2;
	int base = blockDim.x + 4;
	int height = base/2;
    int height2 = height-2;
	int step2;
    int lastidx = ((blockDim.x*gridDim.x)-1);

	int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tididx + k + base;
		tid_bottom[k+2] = tididx + k;
	}

	int leftidx = height + ((tid/4 & 1) * base) + (tid & 3) - (4 + (tid/4) * 2);
	int rightidx = height + ((tid/4 & 1) * base) + (tid/4)*2 + (tid & 3);
    int gidin = (gid - blockDim.x) & lastidx;
	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.

    temper[leftidx] = right[gidin];
	temper[rightidx] = left[gid];

    for (int k = height2; k>0; k-=4)
    {
        if (blockIdx.x > 0)
        {
            if (tididx < (base-k) && tididx >= k)
            {
                temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                    temper[tid_bottom[3]], temper[tid_bottom[4]].w);
            }
        }
        else
        {
            if (tididx < (base-k) && tididx >= k)
            {
                if (tididx == (height-1)) //case 1
                {
                    temper[tididx+base] = dbd[1];
                }
                else if (tididx == height)  //case 2
                {
                    temper[tididx+base] = dbd[0];
                }
                else if (tididx == height2) //case 0
                {
                    temper[tididx+base] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                        dbd[1], dbd[1].w);
                }
                else if (tididx == (height+1)) //case 3
                {
                    temper[tididx+base] = eulerStutterStep(dbd[0].w, dbd[0], temper[tid_bottom[2]],
                        temper[tid_bottom[3]], temper[tid_bottom[4]].w);
                }
                else
                {
                    temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                        temper[tid_bottom[3]], temper[tid_bottom[4]].w);
                }
            }

        }

        step2 = k-2;

        if (blockIdx.x > 0)
        {
            if (tididx < (base-step2) && tididx >= step2)
            {
                temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                    temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
            }
        }
        else
        {
            if (tididx < (base-step2) && tididx >= step2)
            {
                if (tididx == (height-1)) //case 1
                {
                    temper[tididx] = dbd[1];
                }
                else if (tididx == height)  //case 2
                {
                    temper[tididx] = dbd[0];
                }
                else if (tididx == height2) //case 0
                {
                    temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]], temper[tididx],
                        dbd[1], dbd[1].w);
                }
                else if (tididx == (height+1)) //case 3
                {
                    temper[tididx] = eulerFinalStep(dbd[0].w, dbd[0], temper[tid_top[2]],temper[tididx],
                        temper[tid_top[3]], temper[tid_top[4]].w);
                }
                else
                {
                    temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                        temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
                }
            }

        }

        __syncthreads();
    }

    //Justify the result at 0 index.
    temper[tid] = temper[tididx];

    leftidx = ((tid/4 & 1) * blockDim.x) + (tid/4)*2 + (tid & 3);
	rightidx = (blockDim.x - 4) + ((tid/4 & 1) * blockDim.x) + (tid & 3) - (tid/4)*2;

    #pragma unroll
    for (int k = -2; k<3; k++)
    {
        tid_top[k+2] = tid + k + blockDim.x;
        tid_bottom[k+2] = tid + k;
    }

    //The initial conditions are timslice 0 so start k at 1.
    height -= 2;
    height2 -= 2;

    //Single step start to top part.
    if (blockIdx.x > 0)
    {
        if (tid > 1 && tid <(blockDim.x-2))
        {
            temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                temper[tid_bottom[3]], temper[tid_bottom[4]].w);
        }
    }
    else
    {
        if (tid > 1 && tid <(blockDim.x-2))
        {
            if (tid == (height-1)) //case 1
            {
                temper[tid_top[2]] = dbd[1];
            }
            else if (tid == height)  //case 2
            {
                temper[tid_top[2]] = dbd[0];
            }
            else if (tid == height2) //case 0
            {
                temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                    dbd[1], dbd[1].w);
            }
            else if (tid == (height+1)) //case 3
            {
                temper[tid_top[2]] = eulerStutterStep(dbd[0].w, dbd[0], temper[tid_bottom[2]],
                    temper[tid_bottom[3]], temper[tid_bottom[4]].w);
            }
            else
            {
                temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                    temper[tid_bottom[3]], temper[tid_bottom[4]].w);
            }
        }

    }

	__syncthreads();

    //The first new row is complete 2->blk-2 next is 4->blk-4
	for (int k = 4; k<height; k+=4)
	{
        if (blockIdx.x > 0)
        {
            if (tid < (blockDim.x-k) && tid >= k)
            {
                temper[tid] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                    temper[tid], temper[tid_top[3]], temper[tid_top[4]].w);
            }
        }
        else
        {
            if (tid < (blockDim.x-k) && tid >= k)
            {
                if (tid == (height-1)) //case 1
                {
                    temper[tid] = dbd[1];
                }
                else if (tid == height)  //case 2
                {
                    temper[tid] = dbd[0];
                }
                else if (tid == height2) //case 0
                {
                    temper[tid] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],temper[tid],
                        dbd[1], dbd[1].w);
                }
                else if (tid == (height+1)) //case 3
                {
                    temper[tid] = eulerFinalStep(dbd[0].w, dbd[0], temper[tid_top[2]],temper[tid],
                        temper[tid_top[3]], temper[tid_top[4]].w);
                }
                else
                {
                    temper[tid] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                        temper[tid], temper[tid_top[3]], temper[tid_top[4]].w);
                }
            }

        }
		step2 = k + 2;
		__syncthreads();

        if (blockIdx.x > 0)
        {
            if (tid < (blockDim.x-step2) && tid >= step2)
            {
                temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                    temper[tid_bottom[3]], temper[tid_bottom[4]].w);
            }
        }
        else
        {
            if (tid < (blockDim.x-step2) && tid >= step2)
            {
                if (tid == (height-1)) //case 1
                {
                    temper[tid_top[2]] = dbd[1];
                }
                else if (tid == height)  //case 2
                {
                    temper[tid_top[2]] = dbd[0];
                }
                else if (tid == height2) //case 0
                {
                    temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                        dbd[1], dbd[1].w);
                }
                else if (tid == (height+1)) //case 3
                {
                    temper[tid_top[2]] = eulerStutterStep(dbd[0].w, dbd[0], temper[tid_bottom[2]],
                        temper[tid_bottom[3]], temper[tid_bottom[4]].w);
                }
                else
                {
                    temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                        temper[tid_bottom[3]], temper[tid_bottom[4]].w);
                }
            }

        }

		//Make sure the threads are synced
		__syncthreads();

	}

    //After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];
}

using namespace std;
// Do the split diamond on the CPU?
// What's the idea?  Say malloc the pointers and pass values in the wrapper.

__host__
void
CPU_diamond(REALfour *temper, int tpb)
{
    int step2;
    int base = tpb + 4;
    int height = base/2;
    int height2 = height-2;

    //Splitting it is the whole point!
    for (int k = height2; k>0; k-=4)
    {
        for(int n = k; n<(base-k); n++)
        {
            if (n == (height-1)) //case 1
            {
                temper[n+base] = bd[1];
            }
            else if (n == height)  //case 2
            {
                temper[n+base] = bd[0];
            }
            else if (n == height2) //case 0
            {
                temper[n+base] = eulerStutterStep(temper[n-2].w, temper[n-1], temper[n],
                    bd[1], bd[1].w);
            }
            else if (n == (height+1)) //case 3
            {
                temper[n+base] = eulerStutterStep(bd[0].w, bd[0], temper[n],
                    temper[n+1], temper[n+2].w);
            }
            else
            {
                temper[n+base] = eulerStutterStep(temper[n-2].w, temper[n-1], temper[n],
                    temper[n+1], temper[n+2].w);
            }
        }

        step2 = k-2;

        for(int n = step2; n<(base-step2); n++)
        {
            if (n == (height-1)) //case 1
            {
                temper[n] = bd[1];
            }
            else if (n == height)  //case 2
            {
                temper[n] = bd[0];
            }
            else if (n == height2) //case 0
            {
                temper[n] = eulerFinalStep(temper[base+n-2].w, temper[base+n-1], temper[base+n], temper[n],
                    bd[1], bd[1].w);
            }
            else if (n == (height+1)) //case 3
            {
                temper[n] = eulerFinalStep(bd[0].w, bd[0], temper[base+n], temper[n],
                    temper[base+n+1], temper[base+n+2].w);
            }
            else
            {
                temper[n] = eulerFinalStep(temper[base+n-2].w, temper[base+n-1], temper[base+n], temper[n],
                    temper[base+n+1], temper[base+n+2].w);
            }
        }
    }

    for (int k = 0; k<tpb; k++) temper[k] = temper[k+2];

    height -= 2;
    height2 -= 2;

    for(int n = 2; n<(tpb-2); n++)
    {
        if (n == (height-1)) //case 1
        {
            temper[n+tpb] = bd[1];
        }
        else if (n == height)  //case 2
        {
            temper[n+tpb] = bd[0];
        }
        else if (n == height2) //case 0
        {
            temper[n+tpb] = eulerStutterStep(temper[n-2].w, temper[n-1], temper[n],
                bd[1], bd[1].w);
        }
        else if (n == (height+1)) //case 3
        {
            temper[n+tpb] = eulerStutterStep(bd[0].w, bd[0], temper[n],
                temper[n+1], temper[n+2].w);
        }
        else
        {
            temper[n+tpb] = eulerStutterStep(temper[n-2].w, temper[n-1], temper[n],
                temper[n+1], temper[n+2].w);
        }
    }

    //Top part.
    for (int k = 4; k<height; k+=4)
    {
        for(int n = k; n<(tpb-k); n++)
        {
            if (n == (height-1)) //case 1
            {
                temper[n] = bd[1];
            }
            else if (n == height)  //case 2
            {
                temper[n] = bd[0];
            }
            else if (n == height2) //case 0
            {
                temper[n] = eulerFinalStep(temper[tpb+n-2].w, temper[tpb+n-1], temper[tpb+n], temper[n],
                    bd[1], bd[1].w);
            }
            else if (n == (height+1)) //case 3
            {
                temper[n] = eulerFinalStep(bd[0].w, bd[0], temper[tpb+n], temper[n],
                    temper[tpb+n+1], temper[tpb+n+2].w);
            }
            else
            {
                temper[n] = eulerFinalStep(temper[tpb+n-2].w, temper[tpb+n-1], temper[tpb+n], temper[n],
                    temper[tpb+n+1], temper[tpb+n+2].w);
            }
        }

        step2 = k+2;

        for(int n = step2; n<(tpb-step2); n++)
        {
            if (n == (height-1)) //case 1
            {
                temper[n+tpb] = bd[1];
            }
            else if (n == height)  //case 2
            {
                temper[n+tpb] = bd[0];
            }
            else if (n == height2) //case 0
            {
                temper[n+tpb] = eulerStutterStep(temper[n-2].w, temper[n-1], temper[n],
                    bd[1], bd[1].w);
            }
            else if (n == (height+1)) //case 3
            {
                temper[n+tpb] = eulerStutterStep(bd[0].w, bd[0], temper[n],
                    temper[n+1], temper[n+2].w);
            }
            else
            {
                temper[n+tpb] = eulerStutterStep(temper[n-2].w, temper[n-1], temper[n],
                    temper[n+1], temper[n+2].w);
            }
        }
    }
}

//Classic Discretization wrapper.
double
classicWrapper(const int bks, int tpb, const int dv, const REAL dt, const int t_end,
    REALfour *IC, REALfour *T_f, const float freq, ofstream &fwr)
{
    REALfour *dEuler_in, *dEuler_out;

    hipMalloc((void **)&dEuler_in, sizeof(REALfour)*dv);
    hipMalloc((void **)&dEuler_out, sizeof(REALfour)*dv);

    // Copy the initial conditions to the device array.
    hipMemcpy(dEuler_in,IC,sizeof(REALfour)*dv,hipMemcpyHostToDevice);


    double t_eq = 0.0;
    double twrite = freq;

    while (t_eq < t_end)
    {
        classicEuler <<< bks,tpb >>> (dEuler_in, dEuler_out);
        classicEuler <<< bks,tpb >>> (dEuler_out, dEuler_in);
        t_eq += 2*dt;

        if (t_eq > twrite)
        {
            hipMemcpy(T_f, dEuler_in, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);
            fwr << t_eq << " ";

            for (int k = 0; k<dv; k++)
            {
                fwr << T_f[k].x << " ";
            }
            fwr << endl;

            twrite += freq;
        }
    }

    hipMemcpy(T_f, dEuler_in, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);

    hipFree(dEuler_in);
    hipFree(dEuler_out);

    return t_eq;

}

//The wrapper that calls the routine functions.
double
sweptWrapper(const int bks, int tpb, const int dv, REAL dt, const int t_end, const int cpu,
    REALfour *IC, REALfour *T_f, const float freq, ofstream &fwr)
{
    const int base = tpb + 4;
    const int height = base/2;
    const size_t smem1 = 2*tpb*sizeof(REALfour);
    const size_t smem2 = (2*base)*sizeof(REALfour);

    int indices[4][tpb];

    for (int k = 0; k<tpb; k++)
    {
        //Set indices
        indices[0][k] = height + ((k/4 & 1) * base) + (k & 3) - (4 + (k/4) *2); //left
        indices[1][k] = height + ((k/4 & 1) * base) + (k & 3) +  (k/4)*2; // right
        //Get indices
        indices[2][k] = (k/4)*2 + ((k/4 & 1) * tpb) + (k & 3); //left
        indices[3][k] = (tpb - 4) + ((k/4 & 1) * tpb) + (k & 3) -  (k/4)*2; //right
    }

    REALfour *tmpr = (REALfour*)malloc(smem2);
	REALfour *d_IC, *d_right, *d_left;
    REALfour *right, *left;
    hipHostAlloc((void **) &right, tpb*sizeof(REALfour), hipHostMallocDefault);
    hipHostAlloc((void **) &left, tpb*sizeof(REALfour), hipHostMallocDefault);

	hipMalloc((void **)&d_IC, sizeof(REALfour)*dv);
	hipMalloc((void **)&d_right, sizeof(REALfour)*dv);
	hipMalloc((void **)&d_left, sizeof(REALfour)*dv);

	// Copy the initial conditions to the device array.
	hipMemcpy(d_IC,IC,sizeof(REALfour)*dv,hipMemcpyHostToDevice);
	// Start the counter and start the clock.
	const double t_fullstep = 0.25*dt*(double)tpb;

	upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);

    double t_eq;
    double twrite = freq;

	// Call the kernels until you reach the iteration limit.

    if (cpu)
    {
        t_eq = t_fullstep;
        omp_set_num_threads( 2 );

        hipMemcpy(right,d_left, tpb*sizeof(REALfour), hipMemcpyDeviceToHost);
        hipMemcpy(left, d_right+(dv-tpb) , tpb*sizeof(REALfour), hipMemcpyDeviceToHost);

        #pragma omp parallel sections
        {
        #pragma omp section
        {
            for (int k = 0; k<tpb; k++)
            {
                tmpr[indices[0][k]] = left[k];
                tmpr[indices[1][k]] = right[k];
            }

            CPU_diamond(tmpr, tpb);

            for (int k = 0; k<tpb; k++)
            {
                left[k] = tmpr[indices[2][k]];
                right[k] = tmpr[indices[3][k]];
            }
        }
        #pragma omp section
        {
            wholeDiamond <<< bks-1,tpb,smem2 >>>(d_right,d_left,false);
        }
        }

        hipMemcpy(d_right, right, tpb*sizeof(REALfour), hipMemcpyHostToDevice);
        hipMemcpy(d_left, left, tpb*sizeof(REALfour), hipMemcpyHostToDevice);

        while(t_eq < t_end)
        {

            wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,true);

            hipMemcpy(right,d_left, tpb*sizeof(REALfour), hipMemcpyDeviceToHost);
            hipMemcpy(left, d_right+(dv-tpb) , tpb*sizeof(REALfour), hipMemcpyDeviceToHost);
            #pragma omp parallel sections
            {
            #pragma omp section
            {
                for (int k = 0; k<tpb; k++)
                {
                    tmpr[indices[0][k]] = left[k];
                    tmpr[indices[1][k]] = right[k];
                }
                CPU_diamond(tmpr, tpb);

                for (int k = 0; k<tpb; k++)
                {
                    left[k] = tmpr[indices[2][k]];
                    right[k] = tmpr[indices[3][k]];
                }
            }
            #pragma omp section
            {
                wholeDiamond <<< bks-1,tpb,smem2 >>>(d_right,d_left,false);
            }
            }
            hipMemcpy(d_right, right, tpb*sizeof(REALfour), hipMemcpyHostToDevice);
            hipMemcpy(d_left, left, tpb*sizeof(REALfour), hipMemcpyHostToDevice);
            t_eq += t_fullstep;

    	    if (t_eq > twrite)
    		{
    			downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);
    			hipMemcpy(T_f, d_IC, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);
    			fwr << t_eq << " ";

    			for (int k = 0; k<dv; k++)
    			{
    					fwr << T_f[k].x << " ";
    			}

    			fwr << endl;

    			upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);
    			splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);
                twrite += freq;
    		}
        }
	}
    else
    {
        splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);
        t_eq = t_fullstep;

        while(t_eq < t_end)
        {

            wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,true);

            splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);
            //So it always ends on a left pass since the down triangle is a right pass.

            t_eq += t_fullstep;

            if (t_eq > twrite)
            {
                downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);
                hipMemcpy(T_f, d_IC, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);
                fwr << t_eq << " ";

                for (int k = 0; k<dv; k++)
                {
                        fwr << T_f[k].x << " ";
                }

                fwr << endl;

                upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);
                splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);
                twrite += freq;
            }


        }
    }

	downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);

	hipMemcpy(T_f, d_IC, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);

	hipFree(d_IC);
	hipFree(d_right);
	hipFree(d_left);
    hipHostFree(right);
    hipHostFree(left);

    return t_eq;
}

int main( int argc, char *argv[] )
{

    //That is there are less than 8 arguments.
    if (argc < 9)
	{
		cout << "The Program takes 9 inputs, #Divisions, #Threads/block, deltat, finish time, output frequency..." << endl;
        cout << "Classic/Swept, CPU sharing Y/N, Variable Output File, Timing Output File (optional)" << endl;
		exit(-1);
	}

	// Choose the GPGPU.  This is device 0 in my machine which has 2 devices.
	hipSetDevice(0);

    bd[0].x = 1.0; //Density
    bd[1].x = 0.125;
    bd[0].y = 0.0; //Velocity
    bd[1].y = 0.0;
    bd[0].w = 1.0; //Pressure
    bd[1].w = 0.1;
    bd[0].z = bd[0].w/m_gamma; //Energy
    bd[1].z = bd[1].w/m_gamma;


    const int dv = atoi(argv[1]); //Number of spatial points
	const int tpb = atoi(argv[2]); //Threads per Block
    const REAL dt = atof(argv[3]);
	const float tf = atof(argv[4]); //Finish time
    const float freq = atof(argv[5]);
    const int scheme = atoi(argv[6]); //1 for Swept 0 for classic
    const int tst = atoi(argv[7]);
    const int bks = dv/tpb; //The number of blocks
    REAL lx = dx*((float)dv-1.f);

    //Declare the dimensions in constant memory.
    dimz.x = dt/dx; // dt/dx
    dimz.y = gam; dimz.z = m_gamma;

	//Conditions for main input.  Unit testing kinda.
	//dv and tpb must be powers of two.  dv must be larger than tpb and divisible by
	//tpb.

	if ((dv & (tpb-1) !=0) || (tpb&31) != 0)
    {
        cout << "INVALID NUMERIC INPUT!! "<< endl;
        cout << "2nd ARGUMENT MUST BE A POWER OF TWO >= 32 AND FIRST ARGUMENT MUST BE DIVISIBLE BY SECOND" << endl;
        exit(-1);
    }

	// Initialize arrays.
    REALfour *IC, *T_final;
	hipHostAlloc((void **) &IC, dv*sizeof(REALfour), hipHostMallocDefault);
	hipHostAlloc((void **) &T_final, dv*sizeof(REALfour), hipHostMallocDefault);

	// Some initial condition for the bar temperature, an exponential decay
	// function.
	for (int k = 0; k<dv; k++)
	{
        if (k<dv/2)
        {
            IC[k] = bd[0];
        }
        else
        {
            IC[k] = bd[1];
        }
	}

	// Call out the file before the loop and write out the initial condition.
	ofstream fwr;
	fwr.open(argv[8],ios::trunc);
	// Write out x length and then delta x and then delta t.
	// First item of each line is timestamp.
	fwr << lx << " " << dv << " " << dx << " " << endl << 0 << " ";

	for (int k = 0; k<dv; k++)
	{
		fwr << IC[k].x << " ";
	}

	fwr << endl;

    //Transfer data to GPU.
	// This puts the Fourier number in constant memory.
	hipMemcpyToSymbol(HIP_SYMBOL(dimens),&dimz,sizeof(REALthree));
    hipMemcpyToSymbol(HIP_SYMBOL(dbd),&bd,2*sizeof(REALfour));

	// This initializes the device arrays on the device in global memory.
	// They're all the same size.  Conveniently.

	// Start the counter and start the clock.
	hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

    double tfm;
    if (scheme)
    {
        tfm = sweptWrapper(bks, tpb, dv, dt, tf, tst, IC, T_final, freq, fwr);
    }
    else
    {
        tfm = classicWrapper(bks, tpb, dv, dt, tf, IC, T_final, freq, fwr);
    }

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime( &timed, start, stop);

    timed *= 1.e3;

    double n_timesteps = tfm/dt;

    double per_ts = timed/n_timesteps;

    cout << n_timesteps << " timesteps" << endl;
	cout << "Averaged " << per_ts << " microseconds (us) per timestep" << endl;

    if (argc>8)
    {
        ofstream ftime;
        ftime.open(argv[9],ios::app);
    	ftime << dv << "\t" << tpb << "\t" << per_ts << endl;
    	ftime.close();
    }

	fwr << tfm << " ";
	for (int k = 0; k<dv; k++)
	{
		fwr << T_final[k].x << " ";
	}

    fwr << endl;

	fwr.close();

	// Free the memory and reset the device.

	hipEventDestroy( start );
	hipEventDestroy( stop );
    hipDeviceReset();
    free(IC);
    free(T_final);

	return 0;

}
