#include "hip/hip_runtime.h"
//Based on
//https://en.wikipedia.org/wiki/Sod_shock_tube
//http://wonka.physics.ncsu.edu/pub/VH-1/bproblems.php
//http://www.astro.sunysb.edu/mzingale/codes.html
// http://cococubed.asu.edu/code_pages/exact_riemann.shtml


//COMPILE LINE:
// nvcc -o ./bin/EulerOut Euler1D_SweptShared.cu -gencode arch=compute_35,code=sm_35 -lm -w -std=c++11 -Xcompiler -fopenmp

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_vector_types.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>

//#include "SwR_1DShared.h"

#define REAL        float
#define REALfour    float4
#define REALthree   float3

const REAL gam = 1.4;
const REAL m_gamma = 0.4;
const REAL dx = 0.5;

__constant__ REALfour dbd[2]; //0 is left 1 is right.
__constant__ REALthree dimens;

__device__
__forceinline__
REAL
pressure(REALfour current)
{
    return dimens.z * (current.z - (0.5 * current.y * current.y/current.x));
}

//This will need to return the ratio to the execFunc

__device__
__forceinline__
REAL
pressureRatio(REAL cvLeft, REAL cvCenter, REAL cvRight)
{
    return (cvRight- cvCenter)/(cvCenter- cvLeft);
}

__device__
REALfour
limitor(REALthree cvCurrent, REALthree cvOther, REAL pRatio)
{
    if (isfinite(pRatio) && pRatio > 0) //If it's finite and positive
    {
        REAL fact = (pRatio < 1) ? pRatio : 1.f;
        return make_float4(cvCurrent + 0.5* fact * (cvOther - cvCurrent));

    }
    else //If it's nan, inf, negative or zero.
    {
        return make_float4(cvCurrent);
    }
}

//Left and Center then Left and right.
__device__
void
eulerFlux(REALfour cvLeft, REALfour cvRight, REALthree flux)
{
    REAL uLeft = cvLeft.y/cvLeft.x;
    REAL uRight = cvRight.y/cvRight.x;
    REAL eLeft = cvLeft.z/cvLeft.x;
    REAL eRight = cvRight.z/cvRight.x;

    flux.x = 0.5 * (cvLeft.x*uLeft + cvRight.x*uRight);
    flux.y = 0.5 * (cvLeft.x*uLeft*uLeft + cvRight.x*uRight*uRight + cvLeft.w + cvRight.w);
    flux.z = 0.5 * (cvLeft.x*uLeft*eLeft + cvRight.x*uRight*eRight + uLeft*cvLeft.w + uRight*cvRight.w);

    REALfour halfState;
    REAL rhoLeftsqrt = sqrtf(cvLeft.x); REAL rhoRightsqrt = sqrtf(cvRight.x);
    halfState.x = rhoLeftsqrt * rhoRightsqrt;
    halfState.y = (rhoLeftsqrt*uLeft + rhoRightsqrt*uRight)/(rhoLeftsqrt+rhoRightsqrt);
    halfState.z = (rhoLeftsqrt*eLeft + rhoRightsqrt*eRight)/(rhoLeftsqrt+rhoRightsqrt);
    pressure(halfState);

    REAL spectreRadius = sqrtf(dimens.y * halfState.w/halfState.x) + fabs(halfState.y);

    flux += 0.5 * spectreRadius * (make_float3(cvLeft) - make_float3(cvRight));

}


__device__
REALfour
eulerStutterStep(REAL pfarLeft, REALfour stateLeft, REALfour stateCenter, REALfour stateRight, REAL pfarRight)
{
    REALthree fluxL, fluxR, pR;
    REALfour tempStateLeft, tempStateRight;

    pR = make_float3(pressureRatio(pfarLeft,stateLeft.w,stateCenter.w),
        pressureRatio(stateLeft.w,stateCenter.w,stateRight.w),
        pressureRatio(stateCenter.w,stateRight.w,pfarRight));

    tempStateLeft = limitor(make_float3(stateLeft), make_float3(stateCenter), pR.x);
    tempStateRight = limitor(make_float3(stateCenter), make_float3(stateLeft), 1.0/pR.y);
    tempStateLeft.w = pressure(tempStateLeft);
    tempStateRight.w = pressure(tempStateRight);
    eulerFlux(tempStateLeft,tempStateRight,fluxL);

    tempStateLeft = limitor(make_float3(stateCenter), make_float3(stateRight), pR.y);
    tempStateRight = limitor(make_float3(stateRight), make_float3(stateCenter), 1.0/pR.z);
    tempStateLeft.w = pressure(tempStateLeft);
    tempStateRight.w = pressure(tempStateRight);
    eulerFlux(tempStateLeft,tempStateRight,fluxR);

    stateCenter += make_float4(0.5 * dimens.x * (fluxL-fluxR));
    stateCenter.w = pressure(stateCenter);

    return stateCenter;
}

__device__
REALfour
eulerFinalStep(REAL pfarLeft, REALfour stateLeft, REALfour stateCenter, REALfour stateCenter_orig, REALfour stateRight, REAL pfarRight)
{
    REALthree fluxL, fluxR, pR;
    REALfour tempStateLeft, tempStateRight;

    pR = make_float3(pressureRatio(pfarLeft,stateLeft.w,stateCenter.w),
        pressureRatio(stateLeft.w,stateCenter.w,stateRight.w),
        pressureRatio(stateCenter.w,stateRight.w,pfarRight));

    tempStateLeft = limitor(make_float3(stateLeft), make_float3(stateCenter), pR.x);
    tempStateRight = limitor(make_float3(stateCenter), make_float3(stateLeft), 1.0/pR.y);
    tempStateLeft.w = pressure(tempStateLeft);
    tempStateRight.w = pressure(tempStateRight);
    eulerFlux(tempStateLeft,tempStateRight,fluxL);

    tempStateLeft = limitor(make_float3(stateCenter), make_float3(stateRight), pR.y);
    tempStateRight = limitor(make_float3(stateRight), make_float3(stateCenter), 1.0/pR.z);
    tempStateLeft.w = pressure(tempStateLeft);
    tempStateRight.w = pressure(tempStateRight);
    eulerFlux(tempStateLeft,tempStateRight,fluxR);

    stateCenter_orig += make_float4(dimens.x * (fluxL-fluxR));
    stateCenter_orig.w = pressure(stateCenter_orig);

    return stateCenter_orig;
}


// __global__
// void
// classicDisc(REALfour *IC, REALfour *temp)
// {
//
//     int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
//     int lastidx = ((blockDim.x*gridDim.x)-1);
//     int gidp = gid + 1;
//     int gidm = gid - 1;
//
//     if (gid == 0)
//     {
//         temp[gid] = execFunc(IC[gidp], IC[gidp], IC[gid]);
//         printf("IM HERE!\n");
//     }
//     else if (gid == lastidx)
//     {
//
//         temp[gid] = execFunc(IC[gidm], IC[gidm], IC[gid]);
//     }
//     else
//     {
//         temp[gid] = execFunc(IC[gidm], IC[gidp], IC[gid]);
//     }
//
//     IC[gid] = temp[gid];
// }

__global__
void
upTriangle(REALfour *IC, REALfour *right, REALfour *left)
{

	extern __shared__ REALfour temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tid = threadIdx.x; //Block Thread ID

    int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tid + k + blockDim.x;
		tid_bottom[k+2] = tid + k;
	}

	int leftidx = ((tid/4 & 1) * blockDim.x) + (tid/4)*2 + (tid & 3);
	int rightidx = (blockDim.x - 4) + ((tid/4 & 1) * blockDim.x) + (tid & 3) - (tid/4)*2;

	int step2;

    //Assign the initial values to the first row in temper, each block
    //has it's own version of temper shared among its threads.
	temper[tid] = IC[gid];

	if (tid > 1 && tid <(blockDim.x-2))
	{
		temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
			temper[tid_bottom[3]], temper[tid_bottom[4]].w);
	}

	__syncthreads();

	//The initial conditions are timslice 0 so start k at 1.
	for (int k = 4; k<(blockDim.x/2); k+=4)
	{
		if (tid < (blockDim.x-k) && tid >= k)
		{
			temper[tid] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
				temper[tid], temper[tid_top[3]], temper[tid_top[4]].w);

		}

		step2 = k + 2;
		__syncthreads();

		if (tid < (blockDim.x-step2) && tid >= step2)
		{
			temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
				temper[tid_bottom[3]], temper[tid_bottom[4]].w);
		}

		//Make sure the threads are synced
		__syncthreads();

	}

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];


}

// Down triangle is only called at the end when data is passed left.  It's never split.
// It returns IC which is a full 1D result at a certain time.
__global__
void
downTriangle(REALfour *IC, REALfour *right, REALfour *left)
{
	extern __shared__ REALfour temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tididx = tid + 2;
	int base = blockDim.x + 4;
	int height = base/2;
    int height2 = height-2;
    int lastidx = ((blockDim.x*gridDim.x)-1);
	int step2;

	int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tididx + k + base;
		tid_bottom[k+2] = tididx + k;
	}

	int leftidx = height + ((tid/4 & 1) * base) + (tid & 3) - (4 + (tid/4) * 2);
	int rightidx = height + ((tid/4 & 1) * base) + (tid/4)*2 + (tid & 3);
	int gidin = (gid + blockDim.x) & lastidx;

	temper[leftidx] = right[gid];
	temper[rightidx] = left[gidin];

    if (gid == 0)
    {
        temper[base] = dbd[0];
        temper[base+1] = dbd[0];
    }
    if (gid == lastidx)
    {
        temper[2*base-1] = dbd[1];
        temper[2*base-2] = dbd[1];
    }


    if (tididx < (base-height2) && tididx >= height2)
    {
        temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
            temper[tid_bottom[3]], temper[tid_bottom[4]].w);
    }

    __syncthreads();

	for (int k = (height-4); k>4; k-=4)
	{

		if (tididx < (base-k) && tididx >= k)
		{
			temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
				temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
		}

        step2 = k-2;

        if (tididx < (base-step2) && tididx >= step2)
        {
            temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                temper[tid_bottom[3]], temper[tid_bottom[4]].w);

        }
		//Make sure the threads are synced
		__syncthreads();
	}

    if (gid == 0)
    {
        temper[tididx] = dbd[0];
    }
    else if (gid == lastidx)
    {
        temper[tididx] = dbd[1];
    }
    else
    {
        temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
            temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
    }


    IC[gid] = temper[tididx];
}

//Full refers to whether or not there is a node run on the CPU.
__global__
void
wholeDiamond(REALfour *right, REALfour *left, bool full)
{

    extern __shared__ REALfour temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tididx = tid + 2;
	int base = blockDim.x + 4;
	int height = base/2;
    int height2 = height-2;
	int step2;
    int lastidx = ((blockDim.x*gridDim.x)-1);

	int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tididx + k + base;
		tid_bottom[k+2] = tididx + k;
	}

	int leftidx = height + ((tid/4 & 1) * base) + (tid & 3) - (4 + (tid/4) * 2);
	int rightidx = height + ((tid/4 & 1) * base) + (tid/4)*2 + (tid & 3);
	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.

    if (full)
    {
        int gidin = (gid + blockDim.x) & lastidx;
        temper[leftidx] = right[gid];
        temper[rightidx] = left[gidin];
    }
    else
    {
        int gidin = (gid - blockDim.x) & lastidx;
        temper[leftidx] = right[gidin];
        temper[rightidx] = left[gid];
    }

    if (gid == 0)
    {
        temper[base] = dbd[0];
        temper[base+1] = dbd[0];
    }
    if (gid == lastidx)
    {
        temper[2*base-1] = dbd[1];
        temper[2*base-2] = dbd[1];
    }


    if (tididx < (base-height2) && tididx >= height2)
    {
        temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
            temper[tid_bottom[3]], temper[tid_bottom[4]].w);
    }

    __syncthreads();

    for (int k = (height-4); k>4; k-=4)
    {

        if (tididx < (base-k) && tididx >= k)
        {
            temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
        }

        step2 = k-2;

        if (tididx < (base-step2) && tididx >= step2)
        {
            temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                temper[tid_bottom[3]], temper[tid_bottom[4]].w);

        }
        //Make sure the threads are synced
        __syncthreads();
    }

    if (gid == 0)
    {
        temper[tididx] = dbd[0];
    }
    else if (gid == lastidx)
    {
        temper[tididx] = dbd[1];
    }
    else
    {
        temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
            temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
    }

    temper[tid] = temper[tididx];

    // -------------------TOP PART------------------------------------------

    leftidx = ((tid/4 & 1) * blockDim.x) + (tid/4)*2 + (tid & 3);
	rightidx = (blockDim.x - 4) + ((tid/4 & 1) * blockDim.x) + (tid & 3) - (tid/4)*2;

    #pragma unroll
    for (int k = -2; k<3; k++)
    {
        tid_top[k+2] = tid + k + blockDim.x;
        tid_bottom[k+2] = tid + k;
    }

    if (tid > 1 && tid <(blockDim.x-2))
	{
		temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
			temper[tid_bottom[3]], temper[tid_bottom[4]].w);
	}
	//The initial conditions are timeslice 0 so start k at 1.

	__syncthreads();

    //The initial conditions are timslice 0 so start k at 1.
	for (int k = 4; k<(blockDim.x/2); k+=4)
	{
		if (tid < (blockDim.x-k) && tid >= k)
		{
			temper[tid] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
				temper[tid], temper[tid_top[3]], temper[tid_top[4]].w);

		}

		step2 = k + 2;
		__syncthreads();

		if (tid < (blockDim.x-step2) && tid >= step2)
		{
			temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
				temper[tid_bottom[3]], temper[tid_bottom[4]].w);
		}

		//Make sure the threads are synced
		__syncthreads();

	}

    right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];

}

//Split one is always first.  Passing left like the downTriangle.  downTriangle
//should be rewritten so it isn't split.  Only write on a non split pass.
__global__
void
splitDiamond(REALfour *right, REALfour *left)
{
    extern __shared__ REALfour temper[];

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
    int tididx = tid + 2;
	int base = blockDim.x + 4;
	int height = base/2;
    int height2 = height-2;
	int step2;
    int lastidx = ((blockDim.x*gridDim.x)-1);

	int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tididx + k + base;
		tid_bottom[k+2] = tididx + k;
	}

	int leftidx = height + ((tid/4 & 1) * base) + (tid & 3) - (4 + (tid/4) * 2);
	int rightidx = height + ((tid/4 & 1) * base) + (tid/4)*2 + (tid & 3);
    int gidin = (gid - blockDim.x) & lastidx;
	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.

    temper[leftidx] = right[gidin];
	temper[rightidx] = left[gid];

    for (int k = height2; k>0; k-=4)
    {
        if (blockIdx.x > 0)
        {
            if (tididx < (base-k) && tididx >= k)
            {
                temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                    temper[tid_bottom[3]], temper[tid_bottom[4]].w);
            }
        }
        else
        {
            if (tididx < (base-height2) && tididx >= height2)
            {
                if (tididx == (height-1)) //case 1
                {
                    temper[tididx+base] = dbd[1];
                }
                else if (tididx == height)  //case 2
                {
                    temper[tididx+base] = dbd[0];
                }
                else if (tididx == height2) //case 0
                {
                    temper[tididx+base] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                        dbd[1], dbd[1].w);
                }
                else if (tididx == (height+1)) //case 3
                {
                    temper[tididx+base] = eulerStutterStep(dbd[0].w, dbd[0], temper[tid_bottom[2]],
                        temper[tid_bottom[3]], temper[tid_bottom[4]].w);
                }
                else
                {
                    temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                        temper[tid_bottom[3]], temper[tid_bottom[4]].w);
                }
            }

        }

        step2 = k-2;

        if (blockIdx.x > 0)
        {
            if (tididx < (base-step2) && tididx >= step2)
            {
                temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                    temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
            }
        }
        else
        {
            if (tididx < (base-step2) && tididx >= step2)
            {
                if (tididx == (height-1)) //case 1
                {
                    temper[tididx] = dbd[1];
                }
                else if (tididx == height)  //case 2
                {
                    temper[tididx] = dbd[0];
                }
                else if (tididx == height2) //case 0
                {
                    temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]], temper[tididx],
                        dbd[1], dbd[1].w);
                }
                else if (tididx == (height+1)) //case 3
                {
                    temper[tididx] = eulerFinalStep(dbd[0].w, dbd[0], temper[tid_top[2]],temper[tididx],
                        temper[tid_top[3]], temper[tid_top[4]].w);
                }
                else
                {
                    temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                        temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
                }
            }

        }

        __syncthreads();
    }

    temper[tid] = temper[tididx];

    leftidx = ((tid/4 & 1) * blockDim.x) + (tid/4)*2 + (tid & 3);
	rightidx = (blockDim.x - 4) + ((tid/4 & 1) * blockDim.x) + (tid & 3) - (tid/4)*2;

    #pragma unroll
    for (int k = -2; k<3; k++)
    {
        tid_top[k+2] = tid + k + blockDim.x;
        tid_bottom[k+2] = tid + k;
    }

    //The initial conditions are timslice 0 so start k at 1.
    height -= 2;
    height2 -= 2;

    if (blockIdx.x > 0)
    {
        if (tid > 1 && tid <(blockDim.x-2))
        {
            temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                temper[tid_bottom[3]], temper[tid_bottom[4]].w);
        }
    }
    else
    {
        if (tid > 1 && tid <(blockDim.x-2))
        {
            if (tididx == (height-1)) //case 1
            {
                temper[tididx+base] = dbd[1];
            }
            else if (tididx == height)  //case 2
            {
                temper[tididx+base] = dbd[0];
            }
            else if (tididx == height2) //case 0
            {
                temper[tididx+base] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                    dbd[1], dbd[1].w);
            }
            else if (tididx == (height+1)) //case 3
            {
                temper[tididx+base] = eulerStutterStep(dbd[0].w, dbd[0], temper[tid_bottom[2]],
                    temper[tid_bottom[3]], temper[tid_bottom[4]].w);
            }
            else
            {
                temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                    temper[tid_bottom[3]], temper[tid_bottom[4]].w);
            }
        }

    }
	//The initial conditions are timeslice 0 so start k at 1.

	__syncthreads();

    //The initial conditions are timslice 0 so start k at 1.
	for (int k = 4; k<(blockDim.x/2); k+=4)
	{
        if (blockIdx.x > 0)
        {
            if (tid < (blockDim.x-k) && tid >= k)
            {
                temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                    temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
            }
        }
        else
        {
            if (tid < (blockDim.x-k) && tid >= k)
            {
                if (tididx == (height-1)) //case 1
                {
                    temper[tididx] = dbd[1];
                }
                else if (tididx == height)  //case 2
                {
                    temper[tididx] = dbd[0];
                }
                else if (tididx == height2) //case 0
                {
                    temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],temper[tididx],
                        dbd[1], dbd[1].w);
                }
                else if (tididx == (height+1)) //case 3
                {
                    temper[tididx] = eulerFinalStep(dbd[0].w, dbd[0], temper[tid_top[2]],temper[tididx],
                        temper[tid_top[3]], temper[tid_top[4]].w);
                }
                else
                {
                    temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                        temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
                }
            }

        }
		step2 = k + 2;
		__syncthreads();

        if (blockIdx.x > 0)
        {
            if (tid < (blockDim.x-step2) && tid >= step2)
            {
                temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                    temper[tid_bottom[3]], temper[tid_bottom[4]].w);
            }
        }
        else
        {
            if (tid < (blockDim.x-step2) && tid >= step2)
            {
                if (tididx == (height-1)) //case 1
                {
                    temper[tididx+base] = dbd[1];
                }
                else if (tididx == height)  //case 2
                {
                    temper[tididx+base] = dbd[0];
                }
                else if (tididx == height2) //case 0
                {
                    temper[tididx+base] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                        dbd[1], dbd[1].w);
                }
                else if (tididx == (height+1)) //case 3
                {
                    temper[tididx+base] = eulerStutterStep(dbd[0].w, dbd[0], temper[tid_bottom[2]],
                        temper[tid_bottom[3]], temper[tid_bottom[4]].w);
                }
                else
                {
                    temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                        temper[tid_bottom[3]], temper[tid_bottom[4]].w);
                }
            }

        }

		//Make sure the threads are synced
		__syncthreads();

	}

    //After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];
}

//Do the split diamond on the CPU?
// What's the idea?  Say malloc the pointers in the wrapper.
// Calculate left and right idxs in wrapper too, why continually recalculate.
//

// __host__
// void
// CPU_diamond(REALfour *temper, int tpb)
// {
//     int bck, fwd, shft_rd, shft_wr;
//     int base = tpb + 2;
//     int ht = tpb/2;
//
//     //Splitting it is the whole point!
//     for (int k = ht; k>0; k--)
//     {
//         // This tells you if the current row is the first or second.
//         shft_wr = base * ((k+1) & 1);
//         // Read and write are opposite rows.
//         shft_rd = base * (k & 1);
//
//         for(int n = k; n<(base-k); n++)
//         {
//             bck = n - 1;
//             fwd = n + 1;
//             //Double trailing index.
//             if(n == ht)
//             {
//                 temper[n + shft_wr] = execFunc(temper[bck+shft_rd], temper[bck+shft_rd], temper[n+shft_rd]);
//             }
//             //Double leading index.
//             else if(n == ht+1)
//             {
//                 temper[n + shft_wr] = execFunc(temper[fwd+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
//             }
//             else
//             {
//                 temper[n + shft_wr] = execFunc(temper[bck+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
//             }
//         }
//     }
//
//     for (int k = 0; k<tpb; k++) temper[k] = temper[k+1];
//     //Top part.
//     for (int k = 1; k>ht; k++)
//     {
//         // This tells you if the current row is the first or second.
//         shft_wr = base * (k & 1);
//         // Read and write are opposite rows.
//         shft_rd = base * ((k+1) & 1);
//
//         for(int n = k; n<(tpb-k); n++)
//         {
//             bck = n - 1;
//             fwd = n + 1;
//             //Double trailing index.
//             if(n == ht)
//             {
//                 temper[n + shft_wr] = execFunc(temper[bck+shft_rd], temper[bck+shft_rd], temper[n+shft_rd]);
//             }
//             //Double leading index.
//             else if(n == ht+1)
//             {
//                 temper[n + shft_wr] = execFunc(temper[fwd+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
//             }
//             else
//             {
//                 temper[n + shft_wr] = execFunc(temper[bck+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
//             }
//         }
//     }
// }
//
//The host routine.

double
sweptWrapper(const int bks, int tpb, const int dv, REAL dt, const int t_end, const int cpu,
    REALfour *IC, REALfour *T_f, const float freq, std::ofstream &fwr)
{

    const size_t smem1 = 2*tpb*sizeof(REALfour);
    const size_t smem2 = (2*tpb+8)*sizeof(REALfour);

    int indices[4][tpb];
    for (int k = 0; k<tpb; k++)
    {
        indices[0][k] = k/2 + ((k/2 & 1) * tpb) + (k & 1);
        indices[1][k] = (tpb - 2) + ((k/2 & 1) * tpb) + (k & 1) -  k/2;
        indices[2][k] = k/2 + ((k/2 & 1) * tpb) + (k & 1);
        indices[3][k] = (tpb - 1) + ((k/2 & 1) * tpb) + (k & 1) -  k/2;
    }

    REALfour *tmpr = (REALfour*)malloc(smem2);
	REALfour *d_IC, *d_right, *d_left;
    REALfour right[tpb], left[tpb];

	hipMalloc((void **)&d_IC, sizeof(REALfour)*dv);
	hipMalloc((void **)&d_right, sizeof(REALfour)*dv);
	hipMalloc((void **)&d_left, sizeof(REALfour)*dv);

	// Copy the initial conditions to the device array.
	hipMemcpy(d_IC,IC,sizeof(REALfour)*dv,hipMemcpyHostToDevice);
	// Start the counter and start the clock.
	const double t_fullstep = dt*(double)tpb;

	upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);

    double t_eq;

	// Call the kernels until you reach the iteration limit.
    // Done now juse use streams or omp to optimize.

    // if (cpu)
    // {
    //     t_eq = t_fullstep/2;
    //     omp_set_num_threads( 2 );
    //
    // 	while(t_eq < t_end)
    // 	{
    //
    //         #pragma omp parallel sections
    //         {
    //         #pragma omp section
    //         {
    //             hipMemcpy(right,d_left,tpb*sizeof(REAL),hipMemcpyDeviceToHost);
    //             hipMemcpy(left,d_right+dv-tpb,tpb*sizeof(REAL),hipMemcpyDeviceToHost);
    //
    //             for (int k = 0; k<tpb; k++)
    //             {
    //                 tmpr[indices[0][k]] = right[k];
    //                 tmpr[indices[1][k]] = left[k];
    //             }
    //
    //             CPU_diamond(tmpr, tpb);
    //
    //             for (int k = 0; k<tpb; k++)
    //             {
    //                 right[k] = tmpr[indices[2][k]];
    //                 left[k] = tmpr[indices[3][k]];
    //             }
    //         }
    //         #pragma omp section
    //         {
    //             wholeDiamond <<< bks-1,tpb,smem2 >>>(d_right,d_left,false);
    //             hipMemcpy(d_right, right, tpb*sizeof(REAL), hipMemcpyHostToDevice);
    //             hipMemcpy(d_left, left, tpb*sizeof(REAL), hipMemcpyHostToDevice);
    //         }
    //         }
    //
    //         wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,true);
    //
    // 		-------------------------------------
    // 	 	if (true)
    // 		{
    // 			downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);
    // 			hipMemcpy(T_final, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);
    // 			fwr << t_eq << " ";
    //
    // 			for (int k = 0; k<dv; k++)
    // 			{
    // 					fwr << T_final.x[k] << " ";
    // 			}
    // 				fwr << endl;
    //
    // 			upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);
    // 			wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,-1);
    // 		}
    // 		-------------------------------------
    // 		*/
    //     }
	// }
    // else
    // {
        splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);
        t_eq = t_fullstep;

        while(t_eq < t_end)
        {

            wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,true);

            splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);
            //So it always ends on a left pass since the down triangle is a right pass.

            t_eq += t_fullstep;

            /*
            if (true)
            {
                downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);
                hipMemcpy(T_final, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);
                fwr << t_eq << " ";

                for (int k = 0; k<dv; k++)
                {
                        fwr << T_final.x[k] << " ";
                }
                    fwr << endl;

                upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);
                wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,-1);
            }
            -------------------------------------
            */
        }
    //}

	downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);

	hipMemcpy(T_f, d_IC, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);

	hipFree(d_IC);
	hipFree(d_right);
	hipFree(d_left);

    return t_eq;
}

int main( int argc, char *argv[] )
{
    using namespace std;
    if (argc != 7)
	{
		cout << "The Program takes six inputs, #Divisions, #Threads/block, dt, finish time, CPU sharing Y/N, and output frequency" << endl;
		exit(-1);
	}

	// Choose the GPGPU.  This is device 0 in my machine which has 2 devices.
	hipSetDevice(0);

    REALfour bd[2];
    bd[0].x = 1.0; //Density
    bd[1].x = 0.125;
    bd[0].y = 0.0; //Velocity
    bd[1].y = 0.0;
    bd[0].w = 1.0; //Pressure
    bd[1].w = 0.1;
    bd[0].z = bd[0].w/m_gamma; //Energy
    bd[1].z = bd[1].w/m_gamma;

    //Declare the dimensions in constant memory.

    const REAL dt = atof(argv[3]);
    const int dv = atoi(argv[1]); //Number of spatial points
	const int tpb = atoi(argv[2]); //Threads per Blocks
	const float tf = atof(argv[4]); //Finish time
	const int bks = dv/tpb; //The number of blocks
	const int tst = atoi(argv[5]);
    const float freq = atof(argv[6]);
    REAL lx = dx*((float)dv-1.f);

    REALthree dimz;
    dimz.x = dt/dx; // dt/dx
    dimz.y = gam; dimz.z = m_gamma;

	//Conditions for main input.  Unit testing kinda.
	//dv and tpb must be powers of two.  dv must be larger than tpb and divisible by
	//tpb.

	if ((dv & (tpb-1) !=0) || (tpb&31) != 0)
    {
        cout << "INVALID NUMERIC INPUT!! "<< endl;
        cout << "2nd ARGUMENT MUST BE A POWER OF TWO >= 32 AND FIRST ARGUMENT MUST BE DIVISIBLE BY SECOND" << endl;
        exit(-1);
    }

	// Initialize arrays.
    REALfour *IC = (REALfour*)malloc(dv*sizeof(float4));
	REALfour *T_final = (REALfour*)malloc(dv*sizeof(float4));

	// Some initial condition for the bar temperature, an exponential decay
	// function.
	for (int k = 0; k<dv; k++)
	{
        if (k<dv/2)
        {
            IC[k] = bd[0];
        }
        else
        {
            IC[k] = bd[1];
        }
	}

	// Call out the file before the loop and write out the initial condition.
	ofstream fwr, ftime;
	fwr.open("Results/Euler1D_Result.dat",ios::trunc);
	ftime.open("Results/Euler1D_Timing.txt",ios::app);
	// Write out x length and then delta x and then delta t.
	// First item of each line is timestamp.
	fwr << lx << " " << dv << " " << dx << " " << endl << 0 << " ";

	for (int k = 0; k<dv; k++)
	{
		fwr << IC[k].x << " ";
	}

	fwr << endl;

    //Transfer data to GPU.
	// This puts the Fourier number in constant memory.
	hipMemcpyToSymbol(HIP_SYMBOL(dimens),&dimz,sizeof(REALthree));
    hipMemcpyToSymbol(HIP_SYMBOL(dbd),&bd,2*sizeof(REALfour));

	// This initializes the device arrays on the device in global memory.
	// They're all the same size.  Conveniently.

	// Start the counter and start the clock.
	hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

    // Call the kernels until you reach the iteration limit.
	double tfm;

    //--------TEST-----------

    // REALfour *d_IC, *d_temp;
    //
    // hipMalloc((void **)&d_IC, sizeof(REALfour)*dv);
	// hipMalloc((void **)&d_temp, sizeof(REALfour)*dv);
    //
	// // Copy the initial conditions to the device array.
	// hipMemcpy(d_IC,IC,sizeof(REALfour)*dv,hipMemcpyHostToDevice);
    //
    // tfm = 0.0;
    //
    // while (tfm < tf)
    // {
    //     classicDisc <<< bks,tpb >>> (d_IC,d_temp);
    //     tfm += dt;
    //
    // }
    //
    // hipMemcpy(T_final, d_IC, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);
    // hipFree(d_IC);
    // hipFree(d_temp);
    //
    // //--------TEST-----------

	tfm = sweptWrapper(bks,tpb,dv,dt,tf,tst,IC,T_final,freq,fwr);

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime( &timed, start, stop);

	timed = timed * 1.e-3;

	cout << "That took: " << timed << " seconds" << endl;

	ftime << dv << " " << tpb << " " << timed << endl;

	ftime.close();

	fwr << tfm << " ";
	for (int k = 0; k<dv; k++)
	{
		fwr << T_final[k].x << " ";
	}

    fwr << endl;

	fwr.close();

	// Free the memory and reset the device.
	hipDeviceSynchronize();

	hipEventDestroy( start );
	hipEventDestroy( stop );
    hipDeviceReset();
    free(IC);
    free(T_final);

	return 0;

}
