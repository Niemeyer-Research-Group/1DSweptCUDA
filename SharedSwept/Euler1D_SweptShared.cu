#include "hip/hip_runtime.h"
/* This file is the current iteration of research being done to implement the
swept rule for Partial differential equations in one dimension.  This research
is a collaborative effort between teams at MIT, Oregon State University, and
Purdue University.

Copyright (C) 2015 Kyle Niemeyer, niemeyek@oregonstate.edu AND
Daniel Magee, mageed@oregonstate.edu

This program is free software: you can redistribute it and/or modify
it under the terms of the MIT license.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.

You should have received a copy of the MIT license along with this program.
If not, see <https://opensource.org/licenses/MIT>.
*/

//COMPILE LINE:
// nvcc -o ./bin/EulerOut Euler1D_SweptShared.cu -gencode arch=compute_35,code=sm_35 -lm -restrict -Xcompiler -fopenmp

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include "myVectorTypes.h"

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>

#ifndef REAL
    #define REAL        float
    #define REALfour    float4
    #define REALthree   float3
    #define THREEVEC( ... ) make_float3(__VA_ARGS__)
    #define FOURVEC( ... )  make_float4(__VA_ARGS__)
#else
    #define THREEVEC( ... ) make_double3(__VA_ARGS__)
    #define FOURVEC( ... )  make_double4(__VA_ARGS__)
#endif

const REAL gam = 1.4;
const REAL m_gamma = 0.4;
const REAL lx = 1.0;

REALfour bd[2];
REALthree dimz;
//dbd is the boundary condition
__constant__ REALfour dbd[2]; //0 is left 1 is right.
//dimens has three fields x is dt/dx, y is gamma, z is gamma-1
__constant__ REALthree dimens;

//Calculates the pressure at the current node with the rho, u, e state variables.
__device__ __host__
__forceinline__
REAL
pressure(REALfour current)
{
    #ifdef __CUDA_ARCH__
    return dimens.z * (current.z - (0.5 * current.y * current.y/current.x));
    #else
    return dimz.z * (current.z - (0.5 * current.y * current.y/current.x));
    #endif
}

//Calculates the pressure ratio between the right and left side pressure differences.
//(pRight-pCurrent)/(pCurrent-pLeft)
__device__ __host__
__forceinline__
REAL
pressureRatio(REAL cvLeft, REAL cvCenter, REAL cvRight)
{
    return (cvRight- cvCenter)/(cvCenter- cvLeft);
}

//Reconstructs the state variables if the pressure ratio is finite and positive.
//I think it's that internal boundary condition.
__device__ __host__
REALfour
limitor(REALthree cvCurrent, REALthree cvOther, REAL pRatio)
{
    #ifdef __CUDA_ARCH__
    if (isfinite(pRatio) && pRatio > 0) //If it's finite and positive
    {
        REAL fact = ((pRatio < 1) ? pRatio : 1.0);
        return FOURVEC(cvCurrent + 0.5 * fact * (cvOther - cvCurrent));

    }
    #else
    if (std::isfinite(pRatio) && pRatio > 0) //If it's finite and positive
    {
        REAL fact = ((pRatio < 1) ? pRatio : 1.0);
        return FOURVEC(cvCurrent + 0.5 * fact * (cvOther - cvCurrent));

    }
    #endif

    return FOURVEC(cvCurrent);
}

//Left and Center then Left and right.
//This is the meat of the flux calculation.  Fields: x is rho, y is u, z is e, w is p.
__device__ __host__
REALthree
eulerFlux(REALfour cvLeft, REALfour cvRight)
{
    #ifndef __CUDA_ARCH__
    using namespace std;
    #endif
    //For the first calculation rho and p remain the same.
    REALthree flux;
    REAL spectreRadius;

    REAL uLeft = cvLeft.y/cvLeft.x;
    REAL uRight = cvRight.y/cvRight.x;
    REAL eLeft = cvLeft.z/cvLeft.x;
    REAL eRight = cvRight.z/cvRight.x;

    flux.x = 0.5 * (cvLeft.x*uLeft + cvRight.x*uRight);
    flux.y = 0.5 * (cvLeft.x*uLeft*uLeft + cvRight.x*uRight*uRight + cvLeft.w + cvRight.w);
    flux.z = 0.5 * (cvLeft.x*uLeft*eLeft + cvRight.x*uRight*eRight + uLeft*cvLeft.w + uRight*cvRight.w);

    REALfour halfState;
    REAL rhoLeftsqrt = sqrt(cvLeft.x); REAL rhoRightsqrt = sqrt(cvRight.x);
    halfState.x = rhoLeftsqrt * rhoRightsqrt;
    halfState.y = halfState.x * (rhoLeftsqrt*uLeft + rhoRightsqrt*uRight)/(rhoLeftsqrt + rhoRightsqrt);
    halfState.z = halfState.x * (rhoLeftsqrt*eLeft + rhoRightsqrt*eRight)/(rhoLeftsqrt + rhoRightsqrt);
    halfState.w = pressure(halfState);

    halfState.y = halfState.y/halfState.x;

    #ifdef __CUDA_ARCH__
    spectreRadius = sqrt(dimens.y * halfState.w/halfState.x) + fabs(halfState.y);
    #else
    spectreRadius = sqrt(dimz.y * halfState.w/halfState.x) + fabs(halfState.y);
    #endif

    flux += 0.5 * spectreRadius * (THREEVEC(cvLeft) - THREEVEC(cvRight));

    return flux;
}

//This is the predictor step of the finite volume scheme.
__device__ __host__
REALfour
eulerStutterStep(REAL pfarLeft, REALfour stateLeft, REALfour stateCenter, REALfour stateRight, REAL pfarRight)
{
    REALthree fluxL, fluxR, pR;
    REALfour tempStateLeft, tempStateRight;

    //Get the pressure ratios as a structure.
    pR = THREEVEC(pressureRatio(pfarLeft,stateLeft.w,stateCenter.w),
        pressureRatio(stateLeft.w,stateCenter.w,stateRight.w),
        pressureRatio(stateCenter.w,stateRight.w,pfarRight));

    //This is the temporary state bounded by the limitor function.
    tempStateLeft = limitor(THREEVEC(stateLeft), THREEVEC(stateCenter), pR.x);
    tempStateRight = limitor(THREEVEC(stateCenter), THREEVEC(stateLeft), 1.0/pR.y);

    //Pressure needs to be recalculated for the new limited state variables.
    tempStateLeft.w = pressure(tempStateLeft);
    tempStateRight.w = pressure(tempStateRight);
    fluxL = eulerFlux(tempStateLeft,tempStateRight);

    //Do the same thing with the right side.
    tempStateLeft = limitor(THREEVEC(stateCenter), THREEVEC(stateRight), pR.y);
    tempStateRight = limitor(THREEVEC(stateRight), THREEVEC(stateCenter), 1.0/pR.z);
    tempStateLeft.w = pressure(tempStateLeft);
    tempStateRight.w = pressure(tempStateRight);
    fluxR = eulerFlux(tempStateLeft,tempStateRight);

    //Add the change back to the node in question.
    #ifdef __CUDA_ARCH__
    stateCenter += FOURVEC(0.5 * dimens.x * (fluxL-fluxR));
    #else
    stateCenter += FOURVEC(0.5 * dimz.x * (fluxL-fluxR));
    #endif
    stateCenter.w = pressure(stateCenter);

    return stateCenter;
}

//Same thing as the predictor step, but this final step adds the result to the original state variables to advance to the next timestep.
//But the predictor variables to find the fluxes.
__device__ __host__
REALfour
eulerFinalStep(REAL pfarLeft, REALfour stateLeft, REALfour stateCenter, REALfour stateCenter_orig, REALfour stateRight, REAL pfarRight)
{
    REALthree fluxL, fluxR, pR;
    REALfour tempStateLeft, tempStateRight;

    pR = THREEVEC(pressureRatio(pfarLeft,stateLeft.w,stateCenter.w),
        pressureRatio(stateLeft.w,stateCenter.w,stateRight.w),
        pressureRatio(stateCenter.w,stateRight.w,pfarRight));

    tempStateLeft = limitor(THREEVEC(stateLeft), THREEVEC(stateCenter), pR.x);
    tempStateRight = limitor(THREEVEC(stateCenter), THREEVEC(stateLeft), 1.0/pR.y);
    tempStateLeft.w = pressure(tempStateLeft);
    tempStateRight.w = pressure(tempStateRight);
    fluxL = eulerFlux(tempStateLeft,tempStateRight);

    tempStateLeft = limitor(THREEVEC(stateCenter), THREEVEC(stateRight), pR.y);
    tempStateRight = limitor(THREEVEC(stateRight), THREEVEC(stateCenter), 1.0/pR.z);
    tempStateLeft.w = pressure(tempStateLeft);
    tempStateRight.w = pressure(tempStateRight);
    fluxR = eulerFlux(tempStateLeft,tempStateRight);

    #ifdef __CUDA_ARCH__
    stateCenter_orig += FOURVEC(dimens.x * (fluxL-fluxR));
    #else
    stateCenter_orig += FOURVEC(dimz.x * (fluxL-fluxR));
    #endif
    stateCenter_orig.w = pressure(stateCenter_orig);

    return stateCenter_orig;
}

__global__
void
swapKernel(const REALfour *passing_side, REALfour *bin, int direction)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
    int lastidx = ((blockDim.x*gridDim.x)-1);
    int gidout = (gid + direction*blockDim.x) & lastidx;

    bin[gidout] = passing_side[gid];

}

//Simple scheme with dirchlet boundary condition.
__global__
void
classicEuler(const REALfour *euler_in, REALfour *euler_out, bool final, const REALfour *euler_orig)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
    int lastidx = ((blockDim.x*gridDim.x)-1);

    if (final)
    {
        if (gid == 0)
        {
            euler_out[gid] = dbd[0];
        }
        else if (gid == lastidx)
        {
            euler_out[gid] = dbd[1];
        }
        else if (gid == 1)
        {
            euler_out[gid] = eulerFinalStep(dbd[0].w,dbd[0],euler_in[gid],euler_orig[gid],euler_in[(gid+1)],euler_in[(gid+2)].w);
        }
        else if (gid == (lastidx-1))
        {
            euler_out[gid] = eulerFinalStep(euler_in[(gid-2)].w,euler_in[(gid-1)],euler_in[gid],euler_orig[gid],dbd[1],dbd[1].w);
        }
        else
        {
            euler_out[gid] = eulerFinalStep(euler_in[(gid-2)].w,euler_in[(gid-1)],euler_in[gid],euler_orig[gid],euler_in[(gid+1)],euler_in[(gid+2)].w);
        }
    }
    else
    {
        if (gid == 0)
        {
            euler_out[gid] = dbd[0];
        }
        else if (gid == lastidx)
        {
            euler_out[gid] = dbd[1];
        }
        else if (gid == 1)
        {
            euler_out[gid] = eulerStutterStep(dbd[0].w,dbd[0],euler_in[gid],euler_in[(gid+1)],euler_in[(gid+2)].w);
        }
        else if (gid == (lastidx-1))
        {
            euler_out[gid] = eulerStutterStep(euler_in[(gid-2)].w,euler_in[(gid-1)],euler_in[gid],dbd[1],dbd[1].w);
        }
        else
        {
            euler_out[gid] = eulerStutterStep(euler_in[(gid-2)].w,euler_in[(gid-1)],euler_in[gid],euler_in[(gid+1)],euler_in[(gid+2)].w);
        }
    }
}

__global__
void
upTriangle(const REALfour *IC, REALfour *right, REALfour *left)
{

	extern __shared__ REALfour temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tid = threadIdx.x; //Block Thread ID

    int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tid + k + blockDim.x;
		tid_bottom[k+2] = tid + k;
	}

	int leftidx = ((tid/4 & 1) * blockDim.x) + (tid/4)*2 + (tid & 3);
	int rightidx = (blockDim.x - 4) + ((tid/4 & 1) * blockDim.x) + (tid & 3) - (tid/4)*2;

	int step2;

    //Assign the initial values to the first row in temper, each block
    //has it's own version of temper shared among its threads.
	temper[tid] = IC[gid];

    __syncthreads();

	if (tid > 1 && tid <(blockDim.x-2))
	{
		temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
			temper[tid_bottom[3]], temper[tid_bottom[4]].w);
	}

	__syncthreads();

	//The initial conditions are timslice 0 so start k at 1.
	for (int k = 4; k<(blockDim.x/2); k+=4)
	{
		if (tid < (blockDim.x-k) && tid >= k)
		{
			temper[tid] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
				temper[tid], temper[tid_top[3]], temper[tid_top[4]].w);

		}

		step2 = k + 2;
		__syncthreads();

		if (tid < (blockDim.x-step2) && tid >= step2)
		{
			temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
				temper[tid_bottom[3]], temper[tid_bottom[4]].w);
		}

		//Make sure the threads are synced
		__syncthreads();

	}

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];


}

// Down triangle is only called at the end when data is passed left.  It's never split.
// It returns IC which is a full 1D result at a certain time.
__global__
void
downTriangle(REALfour *IC, const REALfour *right, const REALfour *left)
{
	extern __shared__ REALfour temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tididx = tid + 2;
	int base = blockDim.x + 4;
	int height = base/2;
    int height2 = height-2;
    int lastidx = ((blockDim.x*gridDim.x)-1);
	int step2;

	int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tididx + k + base;
		tid_bottom[k+2] = tididx + k;
	}

	int leftidx = height + ((tid/4 & 1) * base) + (tid & 3) - (4 + (tid/4) * 2);
	int rightidx = height + ((tid/4 & 1) * base) + (tid/4)*2 + (tid & 3);

	temper[leftidx] = right[gid];
	temper[rightidx] = left[gid];

    if (gid < 2)
    {
        temper[base+gid] = dbd[0];
    }
    if (gid == lastidx)
    {
        temper[2*base-1] = dbd[1];
        temper[2*base-2] = dbd[1];
    }

    __syncthreads();

    if (tididx < (base-height2) && tididx >= height2)
    {
        temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
            temper[tid_bottom[3]], temper[tid_bottom[4]].w);
    }

    __syncthreads();

	for (int k = (height-4); k>4; k-=4)
	{

		if (tididx < (base-k) && tididx >= k)
		{
			temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
				temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
		}

        step2 = k-2;

        if (tididx < (base-step2) && tididx >= step2)
        {
            temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                temper[tid_bottom[3]], temper[tid_bottom[4]].w);

        }
		//Make sure the threads are synced
		__syncthreads();
	}

    if (gid == 0)
    {
        temper[tididx] = dbd[0];
    }
    else if (gid == lastidx)
    {
        temper[tididx] = dbd[1];
    }
    {
        temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
            temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
    }

    IC[gid] = temper[tididx];
}

//Full refers to whether or not there is a node run on the CPU.
__global__
void
wholeDiamond(REALfour *right, REALfour *left, bool full)
{

    extern __shared__ REALfour temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tididx = tid + 2;
	int base = blockDim.x + 4;
	int height = base/2;
    int height2 = height-2;
	int step2;
    int lastidx = ((blockDim.x*gridDim.x)-1);

	int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tididx + k + base;
		tid_bottom[k+2] = tididx + k;
	}

	int leftidx = height + ((tid/4 & 1) * base) + (tid & 3) - (4 + (tid/4) * 2);
	int rightidx = height + ((tid/4 & 1) * base) + (tid/4)*2 + (tid & 3);

	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.

    if (full)
    {
        temper[leftidx] = right[gid];
        temper[rightidx] = left[gid];

        __syncthreads();
        if (gid < 2)
        {
            temper[base+gid] = dbd[0];
        }
        if (gid == lastidx)
        {
            temper[2*base-1] = dbd[1];
            temper[2*base-2] = dbd[1];
        }
    }
    else
    {
        gid += blockDim.x;
        temper[leftidx] = right[gid];
        temper[rightidx] = left[gid];
    }

    __syncthreads();

    if (tididx < (base-height2) && tididx >= height2)
    {
        temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
            temper[tid_bottom[3]], temper[tid_bottom[4]].w);
    }

    __syncthreads();

    for (int k = (height-4); k>4; k-=4)
    {
        if (tididx < (base-k) && tididx >= k)
        {
            temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
        }

        step2 = k-2;

        if (tididx < (base-step2) && tididx >= step2)
        {
            temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                temper[tid_bottom[3]], temper[tid_bottom[4]].w);

        }
        //Make sure the threads are synced
        __syncthreads();
    }

    if (full)
    {
        if (gid == 0)
        {
            temper[tididx] = dbd[0];
        }
        else if (gid == lastidx)
        {
            temper[tididx] = dbd[1];
        }
        else
        {
            temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
        }
    }
    else
    {
        temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
            temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
    }


    REALfour trade = temper[tididx];
    __syncthreads();
    temper[tid] = trade;
    __syncthreads();

    // -------------------TOP PART------------------------------------------

    leftidx = ((tid/4 & 1) * blockDim.x) + (tid/4)*2 + (tid & 3);
	rightidx = (blockDim.x - 4) + ((tid/4 & 1) * blockDim.x) + (tid & 3) - (tid/4)*2;

    #pragma unroll
    for (int k = -2; k<3; k++)
    {
        tid_top[k+2] = tid + k + blockDim.x;
        tid_bottom[k+2] = tid + k;
    }
    __syncthreads();

    if (tid > 1 && tid <(blockDim.x-2))
	{
		temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
			temper[tid_bottom[3]], temper[tid_bottom[4]].w);
	}
	//The initial conditions are timeslice 0 so start k at 1.

	__syncthreads();

    //The initial conditions are timslice 0 so start k at 1.
	for (int k = 4; k<(blockDim.x/2); k+=4)
	{
		if (tid < (blockDim.x-k) && tid >= k)
		{
			temper[tid] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
				temper[tid], temper[tid_top[3]], temper[tid_top[4]].w);

		}

		step2 = k + 2;
		__syncthreads();

		if (tid < (blockDim.x-step2) && tid >= step2)
		{
			temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
				temper[tid_bottom[3]], temper[tid_bottom[4]].w);
		}

		//Make sure the threads are synced
		__syncthreads();

	}

    right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];

}

//Split one is always first.
__global__
void
splitDiamond(REALfour *right, REALfour *left)
{
    extern __shared__ REALfour temper[];

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
    int tididx = tid + 2;
	int base = blockDim.x + 4;
	int height = base/2;
    int height2 = height - 2;
	int step2;

	int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tididx + k + base;
		tid_bottom[k+2] = tididx + k;
	}

	int leftidx = height + ((tid/4 & 1) * base) + (tid & 3) - (4 + (tid/4) * 2);
	int rightidx = height + ((tid/4 & 1) * base) + (tid/4)*2 + (tid & 3);
	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.

    temper[leftidx] = right[gid];
	temper[rightidx] = left[gid];

    __syncthreads();

    for (int k = height2; k>0; k-=4)
    {
        if (blockIdx.x > 0)
        {
            if (tididx < (base-k) && tididx >= k)
            {
                temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                    temper[tid_bottom[3]], temper[tid_bottom[4]].w);
            }
        }
        else
        {
            if (tididx < (base-k) && tididx >= k)
            {
                if (tididx == (height-1)) //case 1
                {
                    temper[tid_top[2]] = dbd[1];
                }
                else if (tididx == height)  //case 2
                {
                    temper[tid_top[2]] = dbd[0];
                }
                else if (tididx == height2) //case 0
                {
                    temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                        dbd[1], dbd[1].w);
                }
                else if (tididx == (height+1)) //case 3
                {
                    temper[tid_top[2]] = eulerStutterStep(dbd[0].w, dbd[0], temper[tid_bottom[2]],
                        temper[tid_bottom[3]], temper[tid_bottom[4]].w);
                }
                else
                {
                    temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                        temper[tid_bottom[3]], temper[tid_bottom[4]].w);
                }
            }

        }

        step2 = k-2;
        __syncthreads(); //This

        if (blockIdx.x > 0)
        {
            if (tididx < (base-step2) && tididx >= step2)
            {
                temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                    temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
            }
        }
        else
        {
            if (tididx < (base-step2) && tididx >= step2)
            {
                if (tididx == (height-1)) //case 1
                {
                    temper[tididx] = dbd[1];
                }
                else if (tididx == height)  //case 2
                {
                    temper[tididx] = dbd[0];
                }
                else if (tididx == height2) //case 0
                {
                    temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]], temper[tididx],
                        dbd[1], dbd[1].w);
                }
                else if (tididx == (height+1)) //case 3
                {
                    temper[tididx] = eulerFinalStep(dbd[0].w, dbd[0], temper[tid_top[2]],temper[tididx],
                        temper[tid_top[3]], temper[tid_top[4]].w);
                }
                else
                {
                    temper[tididx] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                        temper[tididx], temper[tid_top[3]], temper[tid_top[4]].w);
                }
            }

        }

        __syncthreads();
    }

    //Justify the result at 0 index.
    REALfour trade = temper[tididx];
    __syncthreads();
    temper[tid] = trade;
    __syncthreads();

    leftidx = ((tid/4 & 1) * blockDim.x) + (tid/4)*2 + (tid & 3);
	rightidx = (blockDim.x - 4) + ((tid/4 & 1) * blockDim.x) + (tid & 3) - (tid/4)*2;

    #pragma unroll
    for (int k = -2; k<3; k++)
    {
        tid_top[k+2] = tid + k + blockDim.x;
        tid_bottom[k+2] = tid + k;
    }

    //The initial conditions are timslice 0 so start k at 1.
    height -= 2;
    height2 -= 2;

    __syncthreads();

    //Single step start to top part.
    if (blockIdx.x > 0)
    {
        if (tid > 1 && tid <(blockDim.x-2))
        {
            temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                temper[tid_bottom[3]], temper[tid_bottom[4]].w);
        }
    }
    else
    {
        if (tid > 1 && tid <(blockDim.x-2))
        {
            if (tid == (height-1)) //case 1
            {
                temper[tid_top[2]] = dbd[1];
            }
            else if (tid == height)  //case 2
            {
                temper[tid_top[2]] = dbd[0];
            }
            else if (tid == height2) //case 0
            {
                temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                    dbd[1], dbd[1].w);
            }
            else if (tid == (height+1)) //case 3
            {
                temper[tid_top[2]] = eulerStutterStep(dbd[0].w, dbd[0], temper[tid_bottom[2]],
                    temper[tid_bottom[3]], temper[tid_bottom[4]].w);
            }
            else
            {
                temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                    temper[tid_bottom[3]], temper[tid_bottom[4]].w);
            }
        }

    }

	__syncthreads();

    //The first new row is complete 2->blk-2 next is 4->blk-4
	for (int k = 4; k<height; k+=4)
	{
        if (blockIdx.x > 0)
        {
            if (tid < (blockDim.x-k) && tid >= k)
            {
                temper[tid] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                    temper[tid], temper[tid_top[3]], temper[tid_top[4]].w);
            }
        }
        else
        {
            if (tid < (blockDim.x-k) && tid >= k)
            {
                if (tid == (height-1)) //case 1
                {
                    temper[tid] = dbd[1];
                }
                else if (tid == height)  //case 2
                {
                    temper[tid] = dbd[0];
                }
                else if (tid == height2) //case 0
                {
                    temper[tid] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]], temper[tid],
                        dbd[1], dbd[1].w);
                }
                else if (tid == (height+1)) //case 3
                {
                    temper[tid] = eulerFinalStep(dbd[0].w, dbd[0], temper[tid_top[2]], temper[tid],
                        temper[tid_top[3]], temper[tid_top[4]].w);
                }
                else
                {
                    temper[tid] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
                        temper[tid], temper[tid_top[3]], temper[tid_top[4]].w);
                }
            }

        }
		step2 = k + 2;
		__syncthreads();

        if (blockIdx.x > 0)
        {
            if (tid < (blockDim.x-step2) && tid >= step2)
            {
                temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                    temper[tid_bottom[3]], temper[tid_bottom[4]].w);
            }
        }
        else
        {
            if (tid < (blockDim.x-step2) && tid >= step2)
            {
                if (tid == (height-1)) //case 1
                {
                    temper[tid_top[2]] = dbd[1];
                }
                else if (tid == height)  //case 2
                {
                    temper[tid_top[2]] = dbd[0];
                }
                else if (tid == height2) //case 0
                {
                    temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                        dbd[1], dbd[1].w);
                }
                else if (tid == (height+1)) //case 3
                {
                    temper[tid_top[2]] = eulerStutterStep(dbd[0].w, dbd[0], temper[tid_bottom[2]],
                        temper[tid_bottom[3]], temper[tid_bottom[4]].w);
                }
                else
                {
                    temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
                        temper[tid_bottom[3]], temper[tid_bottom[4]].w);
                }
            }
        }

		//Make sure the threads are synced
		__syncthreads();
	}

    //After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];
}

using namespace std;

__host__
void
CPU_diamond(REALfour *temper, int tpb)
{
    int step2;
    int base = tpb + 4;
    int height = base/2;
    int height2 = height-2;

    //Splitting it is the whole point!
    for (int k = height2; k>0; k-=4)
    {
        for(int n = k; n<(base-k); n++)
        {
            if (n == (height-1)) //case 1
            {
                temper[n+base] = bd[1];
            }
            else if (n == height)  //case 2
            {
                temper[n+base] = bd[0];
            }
            else if (n == height2) //case 0
            {
                temper[n+base] = eulerStutterStep(temper[n-2].w, temper[n-1], temper[n],
                    bd[1], bd[1].w);
            }
            else if (n == (height+1)) //case 3
            {
                temper[n+base] = eulerStutterStep(bd[0].w, bd[0], temper[n],
                    temper[n+1], temper[n+2].w);
            }
            else
            {
                temper[n+base] = eulerStutterStep(temper[n-2].w, temper[n-1], temper[n],
                    temper[n+1], temper[n+2].w);
            }
        }

        step2 = k-2;

        for(int n = step2; n<(base-step2); n++)
        {
            if (n == (height-1)) //case 1
            {
                temper[n] = bd[1];
            }
            else if (n == height)  //case 2
            {
                temper[n] = bd[0];
            }
            else if (n == height2) //case 0
            {
                temper[n] = eulerFinalStep(temper[base+n-2].w, temper[base+n-1], temper[base+n], temper[n],
                    bd[1], bd[1].w);
            }
            else if (n == (height+1)) //case 3
            {
                temper[n] = eulerFinalStep(bd[0].w, bd[0], temper[base+n], temper[n],
                    temper[base+n+1], temper[base+n+2].w);
            }
            else
            {
                temper[n] = eulerFinalStep(temper[base+n-2].w, temper[base+n-1], temper[base+n], temper[n],
                    temper[base+n+1], temper[base+n+2].w);
            }
        }
    }

    for (int k = 0; k<tpb; k++) temper[k] = temper[k+2];

    height -= 2;
    height2 -= 2;

    for(int n = 2; n<(tpb-2); n++)
    {
        if (n == (height-1)) //case 1
        {
            temper[n+tpb] = bd[1];
        }
        else if (n == height)  //case 2
        {
            temper[n+tpb] = bd[0];
        }
        else if (n == height2) //case 0
        {
            temper[n+tpb] = eulerStutterStep(temper[n-2].w, temper[n-1], temper[n],
                bd[1], bd[1].w);
        }
        else if (n == (height+1)) //case 3
        {
            temper[n+tpb] = eulerStutterStep(bd[0].w, bd[0], temper[n],
                temper[n+1], temper[n+2].w);
        }
        else
        {
            temper[n+tpb] = eulerStutterStep(temper[n-2].w, temper[n-1], temper[n],
                temper[n+1], temper[n+2].w);
        }
    }

    //Top part.
    for (int k = 4; k<height; k+=4)
    {
        for(int n = k; n<(tpb-k); n++)
        {
            if (n == (height-1)) //case 1
            {
                temper[n] = bd[1];
            }
            else if (n == height)  //case 2
            {
                temper[n] = bd[0];
            }
            else if (n == height2) //case 0
            {
                temper[n] = eulerFinalStep(temper[tpb+n-2].w, temper[tpb+n-1], temper[tpb+n], temper[n],
                    bd[1], bd[1].w);
            }
            else if (n == (height+1)) //case 3
            {
                temper[n] = eulerFinalStep(bd[0].w, bd[0], temper[tpb+n], temper[n],
                    temper[tpb+n+1], temper[tpb+n+2].w);
            }
            else
            {
                temper[n] = eulerFinalStep(temper[tpb+n-2].w, temper[tpb+n-1], temper[tpb+n], temper[n],
                    temper[tpb+n+1], temper[tpb+n+2].w);
            }
        }

        step2 = k+2;

        for(int n = step2; n<(tpb-step2); n++)
        {
            if (n == (height-1)) //case 1
            {
                temper[n+tpb] = bd[1];
            }
            else if (n == height)  //case 2
            {
                temper[n+tpb] = bd[0];
            }
            else if (n == height2) //case 0
            {
                temper[n+tpb] = eulerStutterStep(temper[n-2].w, temper[n-1], temper[n],
                    bd[1], bd[1].w);
            }
            else if (n == (height+1)) //case 3
            {
                temper[n+tpb] = eulerStutterStep(bd[0].w, bd[0], temper[n],
                    temper[n+1], temper[n+2].w);
            }
            else
            {
                temper[n+tpb] = eulerStutterStep(temper[n-2].w, temper[n-1], temper[n],
                    temper[n+1], temper[n+2].w);
            }
        }
    }
}

REAL
__host__ __inline__
energy(REAL p, REAL rho, REAL u)
{
    return (p/(m_gamma*rho) + 0.5*rho*u*u);
}

//Classic Discretization wrapper.
double
classicWrapper(const int bks, int tpb, const int dv, const REAL dt, const REAL t_end,
    REALfour *IC, REALfour *T_f, const float freq, ofstream &fwr)
{
    REALfour *dEuler_in, *dEuler_out, *dEuler_orig;

    hipMalloc((void **)&dEuler_in, sizeof(REALfour)*dv);
    hipMalloc((void **)&dEuler_out, sizeof(REALfour)*dv);
    hipMalloc((void **)&dEuler_orig, sizeof(REALfour)*dv);

    // Copy the initial conditions to the device array.
    hipMemcpy(dEuler_in,IC,sizeof(REALfour)*dv,hipMemcpyHostToDevice);

    double t_eq = 0.0;
    double twrite = freq;

    while (t_eq < t_end)
    {
        swapKernel <<< bks,tpb >>> (dEuler_in, dEuler_orig, 0);
        classicEuler <<< bks,tpb >>> (dEuler_in, dEuler_out, false, dEuler_orig);
        classicEuler <<< bks,tpb >>> (dEuler_out, dEuler_in, true, dEuler_orig);
        t_eq += dt;

        if (t_eq > twrite)
        {
            hipMemcpy(T_f, dEuler_in, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);

            fwr << " Density " << t_eq << " ";
            for (int k = 0; k<dv; k++) fwr << T_f[k].x << " ";
            fwr << endl;

            fwr << " Velocity " << t_eq << " ";
            for (int k = 0; k<dv; k++) fwr << T_f[k].y/T_f[k].x << " ";
            fwr << endl;

            fwr << " Energy " << t_eq << " ";
            for (int k = 0; k<dv; k++) fwr << energy(T_f[k].w, T_f[k].x, T_f[k].y/T_f[k].x) << " ";
            fwr << endl;

            fwr << " Pressure " << t_eq << " ";
            for (int k = 0; k<dv; k++) fwr << T_f[k].w << " ";
            fwr << endl;

            twrite += freq;
        }
    }

    hipMemcpy(T_f, dEuler_in, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);

    hipFree(dEuler_in);
    hipFree(dEuler_out);
    hipFree(dEuler_orig);

    return t_eq;

}

//The wrapper that calls the routine functions.
double
sweptWrapper(const int bks, int tpb, const int dv, REAL dt, const REAL t_end, const int cpu,
    REALfour *IC, REALfour *T_f, const float freq, ofstream &fwr)
{
    const int base = tpb + 4;
    const int height = base/2;
    const size_t smem1 = 2*tpb*sizeof(REALfour);
    const size_t smem2 = (2*base)*sizeof(REALfour);

    int indices[4][tpb];

    for (int k = 0; k<tpb; k++)
    {
        //Set indices
        indices[0][k] = height + ((k/4 & 1) * base) + (k & 3) - (4 + (k/4) *2); //left
        indices[1][k] = height + ((k/4 & 1) * base) + (k & 3) +  (k/4)*2; // right
        //Get indices
        indices[2][k] = (k/4)*2 + ((k/4 & 1) * tpb) + (k & 3); //left
        indices[3][k] = (tpb - 4) + ((k/4 & 1) * tpb) + (k & 3) -  (k/4)*2; //right
    }

	REALfour *d_IC, *d_right, *d_left, *d_bin;

	hipMalloc((void **)&d_IC, sizeof(REALfour)*dv);
	hipMalloc((void **)&d_right, sizeof(REALfour)*dv);
	hipMalloc((void **)&d_left, sizeof(REALfour)*dv);
    hipMalloc((void **)&d_bin, sizeof(REALfour)*dv);

	// Copy the initial conditions to the device array.
	hipMemcpy(d_IC,IC,sizeof(REALfour)*dv,hipMemcpyHostToDevice);
	// Start the counter and start the clock.
	const double t_fullstep = 0.25*dt*(double)tpb;

	upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);

    swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
    swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

    double t_eq;
    double twrite = freq;

	// Call the kernels until you reach the iteration limit.

    if (cpu)
    {
        REALfour *tmpr = (REALfour*)malloc(smem2);
        REALfour *h_right, *h_left;
        hipHostAlloc((void **) &h_right, tpb*sizeof(REALfour), hipHostMallocDefault);
        hipHostAlloc((void **) &h_left, tpb*sizeof(REALfour), hipHostMallocDefault);

        // h_right = (REALfour *) malloc(tpb*sizeof(REALfour));
        // h_left = (REALfour *) malloc(tpb*sizeof(REALfour));

        t_eq = t_fullstep;
        omp_set_num_threads( 2 );

        //Split Diamond Begin------

        hipMemcpy(h_right, d_left, tpb*sizeof(REALfour), hipMemcpyDeviceToHost);
        hipMemcpy(h_left, d_right , tpb*sizeof(REALfour), hipMemcpyDeviceToHost);

        #pragma omp parallel sections
        {
        #pragma omp section
        {
            for (int k = 0; k<tpb; k++)
            {
                tmpr[indices[0][k]] = h_left[k];
                tmpr[indices[1][k]] = h_right[k];
            }

            CPU_diamond(tmpr, tpb);

            for (int k = 0; k<tpb; k++)
            {
                h_left[k] = tmpr[indices[2][k]];
                h_right[k] = tmpr[indices[3][k]];
            }
        }
        #pragma omp section
        {
            wholeDiamond <<< bks-1,tpb,smem2 >>>(d_right,d_left,false);
        }
        }

        hipMemcpy(d_right, h_right, tpb*sizeof(REALfour), hipMemcpyHostToDevice);
        hipMemcpy(d_left, h_left, tpb*sizeof(REALfour), hipMemcpyHostToDevice);

        swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
        swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

        while(t_eq < t_end)
        {

            wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,true);

            swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
            swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

            //Split Diamond Begin------

            hipMemcpy(h_right,d_left, tpb*sizeof(REALfour), hipMemcpyDeviceToHost);
            hipMemcpy(h_left, d_right, tpb*sizeof(REALfour), hipMemcpyDeviceToHost);

            #pragma omp parallel sections
            {
            #pragma omp section
            {
                for (int k = 0; k<tpb; k++)
                {
                    tmpr[indices[0][k]] = h_left[k];
                    tmpr[indices[1][k]] = h_right[k];
                }
                CPU_diamond(tmpr, tpb);

                for (int k = 0; k<tpb; k++)
                {
                    h_left[k] = tmpr[indices[2][k]];
                    h_right[k] = tmpr[indices[3][k]];
                }
            }
            #pragma omp section
            {
                wholeDiamond <<< bks-1,tpb,smem2 >>>(d_right,d_left,false);
            }
            }

            hipMemcpy(d_right, h_right, tpb*sizeof(REALfour), hipMemcpyHostToDevice);
            hipMemcpy(d_left, h_left, tpb*sizeof(REALfour), hipMemcpyHostToDevice);

            swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
            swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

            //Split Diamond End------

            t_eq += t_fullstep;

    	    if (t_eq > twrite)
    		{
    			downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);

    			hipMemcpy(T_f, d_IC, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);

                fwr << " Density " << t_eq << " ";
                for (int k = 0; k<dv; k++) fwr << T_f[k].x << " ";
                fwr << endl;

                fwr << " Velocity " << t_eq << " ";
                for (int k = 0; k<dv; k++) fwr << (T_f[k].y/T_f[k].x) << " ";
                fwr << endl;

                fwr << " Energy " << t_eq << " ";
                for (int k = 0; k<dv; k++) fwr << energy(T_f[k].w, T_f[k].x, T_f[k].y/T_f[k].x) << " ";
                fwr << endl;

                fwr << " Pressure " << t_eq << " ";
                for (int k = 0; k<dv; k++) fwr << T_f[k].w << " ";
                fwr << endl;

    			upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);

                swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
                swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

    			splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);

                swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
                swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

                t_eq += t_fullstep;

                twrite += freq;
    		}
        }

        hipHostFree(h_right);
        hipHostFree(h_left);
        // free(h_right);
        // free(h_left);
        free(tmpr);
	}
    else
    {
        splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);
        t_eq = t_fullstep;
        swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
        swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

        while(t_eq < t_end)
        {
            wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,true);

            swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
            swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

            splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);

            swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
            swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

            t_eq += t_fullstep;

            if (t_eq > twrite)
            {
                downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);

                hipMemcpy(T_f, d_IC, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);

                fwr << " Density " << t_eq << " ";
            	for (int k = 0; k<dv; k++) fwr << T_f[k].x << " ";
                fwr << endl;

                fwr << " Velocity " << t_eq << " ";
            	for (int k = 0; k<dv; k++) fwr << (T_f[k].y/T_f[k].x) << " ";
                fwr << endl;

                fwr << " Energy " << t_eq << " ";
                for (int k = 0; k<dv; k++) fwr << energy(T_f[k].w, T_f[k].x, T_f[k].y/T_f[k].x) << " ";
                fwr << endl;

                fwr << " Pressure " << t_eq << " ";
                for (int k = 0; k<dv; k++) fwr << T_f[k].w << " ";
                fwr << endl;

                upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);

                swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
                swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

    			splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);

                swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
                swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

                t_eq += t_fullstep;

                twrite += freq;
            }
        }
    }

	downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);

	hipMemcpy(T_f, d_IC, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);

	hipFree(d_IC);
	hipFree(d_right);
	hipFree(d_left);
    hipFree(d_bin);

    return t_eq;
}

int main( int argc, char *argv[] )
{

    //That is there are less than 8 arguments.
    if (argc < 9)
	{
		cout << "The Program takes 9 inputs, #Divisions, #Threads/block, deltat, finish time, output frequency..." << endl;
        cout << "Classic/Swept, CPU sharing Y/N, Variable Output File, Timing Output File (optional)" << endl;
		exit(-1);
	}

	// Choose the GPGPU.  This is device 0 in my machine which has 2 devices.
	hipSetDevice(0);

    bd[0].x = 1.0; //Density
    bd[1].x = 0.125;
    bd[0].y = 0.0; //Velocity
    bd[1].y = 0.0;
    bd[0].w = 1.0; //Pressure
    bd[1].w = 0.1;
    bd[0].z = bd[0].w/m_gamma; //Energy
    bd[1].z = bd[1].w/m_gamma;


    const int dv = atoi(argv[1]); //Number of spatial points
	const int tpb = atoi(argv[2]); //Threads per Block
    const REAL dt = atof(argv[3]);
	const float tf = atof(argv[4]); //Finish time
    const float freq = atof(argv[5]);
    const int scheme = atoi(argv[6]); //1 for Swept 0 for classic
    const int share = atoi(argv[7]);
    const int bks = dv/tpb; //The number of blocks
    const REAL dx = lx/((REAL)dv-1.f);
    char const *prec;
    prec = (sizeof(REAL)<6) ? "Single": "Double";

    //Declare the dimensions in constant memory.
    dimz.x = dt/dx; // dt/dx
    dimz.y = gam; dimz.z = m_gamma;

    cout << "Euler --- #Blocks: " << bks << " | Length: " << lx << " | Precision: " << prec << " | dt/dx: " << dimz.x << endl;



	//Conditions for main input.  Unit testing kinda.
	//dv and tpb must be powers of two.  dv must be larger than tpb and divisible by
	//tpb.

	if ((dv & (tpb-1) !=0) || (tpb&31) != 0)
    {
        cout << "INVALID NUMERIC INPUT!! "<< endl;
        cout << "2nd ARGUMENT MUST BE A POWER OF TWO >= 32 AND FIRST ARGUMENT MUST BE DIVISIBLE BY SECOND" << endl;
        exit(-1);
    }

    if (dimz.x > .1)
    {
        cout << "The value of dt/dx (" << dimz.x << ") is too high.  In general it must be <=.1 for stability." << endl;
        exit(-1);
    }

	// Initialize arrays.
    REALfour *IC, *T_final;
	hipHostAlloc((void **) &IC, dv*sizeof(REALfour), hipHostMallocDefault);
	hipHostAlloc((void **) &T_final, dv*sizeof(REALfour), hipHostMallocDefault);
    // IC = (REALfour *) malloc(dv*sizeof(REALfour));
    // T_final = (REALfour *) malloc(dv*sizeof(REALfour));

	// Some initial condition for the bar temperature, an exponential decay
	// function.
	for (int k = 0; k<dv; k++)
	{
        if (k<dv/2)
        {
            IC[k] = bd[0];
        }
        else
        {
            IC[k] = bd[1];
        }
	}

	// Call out the file before the loop and write out the initial condition.
	ofstream fwr;
	fwr.open(argv[8],ios::trunc);
	// Write out x length and then delta x and then delta t.
	// First item of each line is timestamp.
	fwr << lx << " " << dv << " " << dx << " " << endl;

    fwr << " Density " << 0 << " ";
    for (int k = 0; k<dv; k++) fwr << IC[k].x << " ";
    fwr << endl;

    fwr << " Velocity " << 0 << " ";
    for (int k = 0; k<dv; k++) fwr << IC[k].y << " ";
    fwr << endl;

    fwr << " Energy " << 0 << " ";
    for (int k = 0; k<dv; k++) fwr << energy(IC[k].w, IC[k].x, IC[k].y/IC[k].x) << " ";
    fwr << endl;

    fwr << " Pressure " << 0 << " ";
    for (int k = 0; k<dv; k++) fwr << IC[k].w << " ";
    fwr << endl;

    //Transfer data to GPU.
	// This puts the Fourier number in constant memory.
	hipMemcpyToSymbol(HIP_SYMBOL(dimens),&dimz,sizeof(REALthree));
    hipMemcpyToSymbol(HIP_SYMBOL(dbd),&bd,2*sizeof(REALfour));

	// This initializes the device arrays on the device in global memory.
	// They're all the same size.  Conveniently.

	// Start the counter and start the clock.
	hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

    double tfm;
    if (scheme)
    {
        cout << "Swept" << endl;
        tfm = sweptWrapper(bks, tpb, dv, dt, tf, share, IC, T_final, freq, fwr);
    }
    else
    {
        cout << "Classic" << endl;
        tfm = classicWrapper(bks, tpb, dv, dt, tf, IC, T_final, freq, fwr);
    }

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime( &timed, start, stop);

    timed *= 1.e3;

    double n_timesteps = tfm/dt;

    double per_ts = timed/n_timesteps;

    cout << n_timesteps << " timesteps" << endl;
	cout << "Averaged " << per_ts << " microseconds (us) per timestep" << endl;

    if (argc>8)
    {
        ofstream ftime;
        ftime.open(argv[9],ios::app);
    	ftime << dv << "\t" << tpb << "\t" << per_ts << endl;
    	ftime.close();
    }

	fwr << " Density " << tfm << " ";
	for (int k = 0; k<dv; k++) fwr << T_final[k].x << " ";
    fwr << endl;

    fwr << " Velocity " << tfm << " ";
	for (int k = 0; k<dv; k++) fwr << T_final[k].y/T_final[k].x << " ";
    fwr << endl;

    fwr << " Energy " << tfm << " ";
    for (int k = 0; k<dv; k++) fwr << energy(T_final[k].w, T_final[k].x, T_final[k].y/T_final[k].x) << " ";
    fwr << endl;

    fwr << " Pressure " << tfm << " ";
    for (int k = 0; k<dv; k++) fwr << T_final[k].w << " ";
    fwr << endl;

	fwr.close();

	// Free the memory and reset the device.

    hipDeviceSynchronize();

	hipEventDestroy( start );
	hipEventDestroy( stop );
    hipDeviceReset();

    hipHostFree(IC);
    hipHostFree(T_final);
    // free(IC);
    // free(T_final);

	return 0;

}
