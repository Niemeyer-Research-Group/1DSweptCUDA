#include "hip/hip_runtime.h"
//Based on
//https://en.wikipedia.org/wiki/Sod_shock_tube
//http://wonka.physics.ncsu.edu/pub/VH-1/bproblems.php
//http://www.astro.sunysb.edu/mzingale/codes.html
// http://cococubed.asu.edu/code_pages/exact_riemann.shtml


//COMPILE LINE:
// nvcc -o ./bin/EulerOut Euler1D_SweptShared.cu -gencode arch=compute_35,code=sm_35 -lm -w -std=c++11

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_vector_types.h>
#include <math_functions.h>

#include <ostream>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <omp.h>

//#include "SwR_1DShared.h"


#define REAL        float
#define REALfour    float4
#define REALthree   float3

const REAL gam = 1.4f;
const REAL m_gamma = .4;
const REAL dx = .5;

__constant__ REALfour dbd[2];
__constant__ REALthree dimens;

__device__
__forceinline__
void
pressure(REALfour current)
{
    current.w = dimens.z * (current.z - (0.5 * current.y * current.y/current.x));
}

//This will need to return the ratio to the execFunc

__device__
__forceinline__
REAL
pressureRatio(REAL cvLeft, REAL cvCenter, REAL cvRight)
{
    return (cvRight- cvCenter)/(cvCenter- cvLeft);
}


__device__
REALfour
limitor(REALthree cvCurrent, REALthree cvOther, REAL pRatio)
{
    if (isfinite(pRatio) && pRatio > 0) //If it's finite and positive
    {
        REAL fact = (pRatio < 1) ? pRatio : 1.f;
        return make_float4(cvCurrent + 0.5* fact * (cvOther - cvCurrent));

    }
    else //If it's nan, inf, negative or zero.
    {
        return make_float4(cvCurrent);
    }

}

//Left and Center then Left and right.
__device__
void
eulerFlux(REALfour cvLeft, REALfour cvRight, REALthree flux)
{
    REAL uLeft = cvLeft.y/cvLeft.x;
    REAL uRight = cvRight.y/cvRight.x;
    REAL eLeft = cvLeft.z/cvLeft.x;
    REAL eRight = cvRight.z/cvRight.x;

    flux.x = 0.5 * (cvLeft.x*uLeft + cvRight.x*uRight);
    flux.y = 0.5 * (cvLeft.x*uLeft*uLeft + cvRight.x*uRight*uRight + cvLeft.w + cvRight.w);
    flux.z = 0.5 * (cvLeft.x*uLeft*eLeft + cvRight.x*uRight*eRight + uLeft*cvLeft.w + uRight*cvRight.w);

    printf("FluxL: %.8f %.8f %.8f \n",flux.x,flux.y, flux.z);

    REALfour halfState;
    REAL rhoLeftsqrt = sqrtf(cvLeft.x); REAL rhoRightsqrt = sqrtf(cvRight.x);
    halfState.x = rhoLeftsqrt * rhoRightsqrt;
    halfState.y = (rhoLeftsqrt*uLeft + rhoRightsqrt*uRight)/(rhoLeftsqrt+rhoRightsqrt);
    halfState.z = (rhoLeftsqrt*eLeft + rhoRightsqrt*eRight)/(rhoLeftsqrt+rhoRightsqrt);
    pressure(halfState);

    REAL spectreRadius = sqrtf(dimens.y * halfState.w/halfState.x) + fabs(halfState.y);

    flux += 0.5 * spectreRadius * (make_float3(cvLeft) - make_float3(cvRight));

}


__device__
REALfour
eulerStutterStep(REAL pfarLeft, REALfour stateLeft, REALfour stateCenter, REALfour stateRight, REAL pfarRight)
{
    REALthree fluxL, fluxR, pR;
    REALfour tempStateLeft, tempStateRight;

    pR = make_float3(pressureRatio(pfarLeft,stateLeft.w,stateCenter.w),
        pressureRatio(stateLeft.w,stateCenter.w,stateRight.w),
        pressureRatio(stateCenter.w,stateRight.w,pfarRight));

    tempStateLeft = limitor(make_float3(stateLeft), make_float3(stateCenter), pR.x);
    tempStateRight = limitor(make_float3(stateCenter), make_float3(stateLeft), 1.0/pR.y);
    pressure(tempStateLeft);
    pressure(tempStateRight);
    eulerFlux(tempStateLeft,tempStateRight,fluxL);

    tempStateLeft = limitor(make_float3(stateCenter), make_float3(stateRight), pR.y);
    tempStateRight = limitor(make_float3(stateRight), make_float3(stateCenter), 1.0/pR.z);
    pressure(tempStateLeft);
    pressure(tempStateRight);
    eulerFlux(tempStateLeft,tempStateRight,fluxR);

    stateCenter += make_float4(0.5 * dimens.x * (fluxL-fluxR));
    return pressure(stateCenter);

}

eulerFinalStep(REAL pfarLeft, REALfour stateLeft, REALfour stateCenter, REAL stateCenter_orig REALfour stateRight, REAL pfarRight)
{
    REALthree fluxL, fluxR, pR;
    REALfour tempStateLeft, tempStateRight;

    pR = make_float3(pressureRatio(pfarLeft,stateLeft.w,stateCenter.w),
        pressureRatio(stateLeft.w,stateCenter.w,stateRight.w),
        pressureRatio(stateCenter.w,stateRight.w,pfarRight));

    tempStateLeft = limitor(make_float3(stateLeft), make_float3(stateCenter), pR.x);
    tempStateRight = limitor(make_float3(stateCenter), make_float3(stateLeft), 1.0/pR.y);
    pressure(tempStateLeft);
    pressure(tempStateRight);
    eulerFlux(tempStateLeft,tempStateRight,fluxL);

    tempStateLeft = limitor(make_float3(stateCenter), make_float3(stateRight), pR.y);
    tempStateRight = limitor(make_float3(stateRight), make_float3(stateCenter), 1.0/pR.z);
    pressure(tempStateLeft);
    pressure(tempStateRight);
    eulerFlux(tempStateLeft,tempStateRight,fluxR);

    stateCenter_orig += make_float4(dimens.x * (fluxL-fluxR));
    return pressure(stateCenter_orig);

}


__global__
void
classicDisc(REALfour *IC, REALfour *temp)
{

    int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
    int lastidx = ((blockDim.x*gridDim.x)-1);
    int gidp = gid + 1;
    int gidm = gid - 1;

    if (gid == 0)
    {
        temp[gid] = execFunc(IC[gidp], IC[gidp], IC[gid]);
        printf("IM HERE!\n");
    }
    else if (gid == lastidx)
    {

        temp[gid] = execFunc(IC[gidm], IC[gidm], IC[gid]);
    }
    else
    {
        temp[gid] = execFunc(IC[gidm], IC[gidp], IC[gid]);
    }

    IC[gid] = temp[gid];
}

__global__
void
upTriangle(REALfour *IC, REALfour *right, REALfour *left)
{

	extern __shared__ REALfour temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tid = threadIdx.x; //Block Thread ID

    int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tid + k + blockDim.x;
		tid_bottom[k+2] = tid + k;
	}

	int leftidx = ((tid/4 & 1) * blockDim.x) + (tid/4)*2 + (tid & 3);
	int rightidx = (blockDim.x - 4) + ((tid/4 & 1) * blockDim.x) + (tid & 3) - (tid/4)*2;

	int step2;

    //Assign the initial values to the first row in temper, each block
    //has it's own version of temper shared among its threads.
	temper[tid] = IC[gid];

	if (tid > 1 && tid <(blockDim.x-2))
	{
		temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
			temper[tid_bottom[3]], temper[tid_bottom[4]].w);
	}

	__syncthreads();

	//The initial conditions are timslice 0 so start k at 1.
	for (int k = 4; k<(blockDim.x/2); k+=4)
	{
		if (tid < (blockDim.x-k) && tid >= k)
		{
			temper[tid] = eulerFinalStep(temper[tid_top[0]].w, temper[tid_top[1]], temper[tid_top[2]],
				temper[tid], temper[tid_top[3]], temper[tid_top[4]].w);

		}

		step2 = k + 2;
		__syncthreads();

		if (tid < (blockDim.x-step2) && tid >= step2)
		{
			temper[tid_top[2]] = eulerStutterStep(temper[tid_bottom[0]].w, temper[tid_bottom[1]], temper[tid_bottom[2]],
				temper[tid_bottom[3]], temper[tid_bottom[4]].w);
		}

		//Make sure the threads are synced
		__syncthreads();

	}

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];


}

// Down triangle is only called at the end when data is passed left.  It's never split.
// It returns IC which is a full 1D result at a certain time.
__global__
void
downTriangle(REALfour *IC, REALfour *right, REALfour *left)
{
	extern __shared__ REALfour temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tididx = tid + 2;
	int base = blockDim.x + 4;
	int height = base/2;
	int step2;

	int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tididx + k + base;
		tid_bottom[k+2] = tididx + k;
	}

	int leftidx = height + ((tid/4 & 1) * base) + (tid & 3) - (4 + (tid/4) * 2);
	int rightidx = height + ((tid/4 & 1) * base) + (tid/4)*2 + (tid & 3);
	int gidin = (gid + blockDim.x) & ((blockDim.x*gridDim.x)-1);

	temper[leftidx] = right[gid];
	temper[rightidx] = left[gidin];

	for (int k = (height-2); k>0; k-=4)
	{
		if (tididx < (base-k) && tididx >= k)
		{
			temper[tid_top[2]] = stutterStep(temper[tid_bottom[0]].x, temper[tid_bottom[1]], temper[tid_bottom[2]],
				temper[tid_bottom[3]], temper[tid_bottom[4]].x);

		}

		step2 = k-2;

		if (tididx < (base-step2) && tididx >= step2)
		{
			temper[tididx] = finalStep(temper[tid_top[0]].x, temper[tid_top[1]], temper[tid_top[2]],
				temper[tididx], temper[tid_top[3]], temper[tid_top[4]].x);
		}

		//Make sure the threads are synced
		__syncthreads();
	}

    IC[gid] = temper[tididx];
}


//Full refers to whether or not there is a node run on the CPU.
__global__
void
wholeDiamond(REALfour *right, REALfour *left, bool full)
{

    extern __shared__ REALfour temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tididx = tid + 2;
	int base = blockDim.x + 4;
	int height = base/2;
	int step2;

	int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tididx + k + base;
		tid_bottom[k+2] = tididx + k;
	}

	int leftidx = height + ((tid/4 & 1) * base) + (tid & 3) - (4 + (tid/4) * 2);
	int rightidx = height + ((tid/4 & 1) * base) + (tid/4)*2 + (tid & 3);
	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.

    if (full)
    {
        int gidin = (gid + blockDim.x) & lastidx;
        temper[leftidx] = right[gid];
        temper[rightidx] = left[gidin];
    }
    else
    {
        int gidin = (gid - blockDim.x) & lastidx;
        temper[leftidx] = right[gidin];
        temper[rightidx] = left[gid];
    }

    for (int k = (height-2); k>0; k-=4)
	{
		if (tididx < (base-k) && tididx >= k)
		{
			temper[tid_top[2]] = stutterStep(temper[tid_bottom[0]].x, temper[tid_bottom[1]], temper[tid_bottom[2]],
				temper[tid_bottom[3]], temper[tid_bottom[4]].x);

		}

		step2 = k-2;

		if (tididx < (base-step2) && tididx >= step2)
		{
			temper[tididx] = finalStep(temper[tid_top[0]].x, temper[tid_top[1]], temper[tid_top[2]],
				temper[tididx], temper[tid_top[3]], temper[tid_top[4]].x);
		}

		//Make sure the threads are synced
		__syncthreads();
	}

    //Boundary Conditions! This justifies it.
    if (full)
    {
        if (gid == 0)
        {
            temper[tid] = execFunc(temper[tid2+base], temper[tid2+base], temper[tid1+base]);
        }
        else if (gid == lastidx)
        {
            temper[tid] = execFunc(temper[tid+base], temper[tid+base], temper[tid1+base]);
        }
        else
        {
            temper[tid] = execFunc(temper[tid+base], temper[tid2+base], temper[tid1+base]);
        }
    }
    else
    {
        temper[tid] = execFunc(temper[tid+base], temper[tid2+base], temper[tid1+base]);
    }

    __syncthreads();

    // Then make sure each block of threads are synced.

    // -------------------TOP PART------------------------------------------



    int leftidx = ((tid/4 & 1) * blockDim.x) + (tid/4)*2 + (tid & 3);
	int rightidx = (blockDim.x - 4) + ((tid/4 & 1) * blockDim.x) + (tid & 3) - (tid/4)*2;

    #pragma unroll
    for (int k = -2; k<3; k++)
    {
        tid_top[k+2] = tid + k + blockDim.x;
        tid_bottom[k+2] = tid + k;
    }


	//The initial conditions are timeslice 0 so start k at 1.

    for (int k = 1; k<(height-1); k++)
	{
		//Bitwise even odd. On even iterations write to first row.
		shft_wr = base * (k & 1);
		//On even iterations write to second row (starts at element 32)
		shft_rd = base * ((k + 1) & 1);

		//Each iteration the triangle narrows.  When k = 1, 30 points are
		//computed, k = 2, 28 points.
		if (tid < (blockDim.x-k) && tid >= k)
		{
			temper[tid + shft_wr] = execFunc(temper[tidm+shft_rd], temper[tid1+shft_rd], temper[tid+shft_rd]);
		}

		//Make sure the threads are synced
		__syncthreads();

	}

    right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];

}

//Split one is always first.  Passing left like the downTriangle.  downTriangle
//should be rewritten so it isn't split.  Only write on a non split pass.
__global__
void
splitDiamond(REALfour *right, REALfour *left)
{

    extern __shared__ REALfour temper[];

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
    int lastidx = ((blockDim.x*gridDim.x)-1);
	int tid1 = tid + 1;
	int tid2 = tid + 2;
	int base = blockDim.x + 2;
	int height = base/2;
	int shft_rd;
	int shft_wr;
	int leftidx = base/2 - tid/2 + ((tid/2 & 1) * base) + (tid & 1) - 2;
	int rightidx = base/2 + tid/2 + ((tid/2 & 1) * base) + (tid & 1);
    int gidin = (gid - blockDim.x) & lastidx;
	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.

    temper[leftidx] = right[gidin];
	temper[rightidx] = left[gid];

    //Wind it up!
    //k needs to insert the relevant left right values around the computed values
    //every timestep.  Since it grows larger the loop is reversed.

    for (int k = (height-1); k>0; k--)
    {
        // This tells you if the current row is the first or second.
        shft_wr = base * ((k+1) & 1);
        // Read and write are opposite rows.
        shft_rd = base * (k & 1);

        //Block 0 is split so it needs a different algorithm.  This algorithm
        //is slightly different than top triangle as described in the note above.
        if (blockIdx.x > 0)
        {
            if (tid1 < (base-k) && tid1 >= k)
            {
                temper[tid1 + shft_wr] = execFunc(temper[tid+shft_rd], temper[tid2+shft_rd], temper[tid1+shft_rd]);
            }

        }

        else
        {
            if (tid1 < (base-k) && tid1 >= k)
            {
                if (tid1 == (height-1))
                {
                    temper[tid1 + shft_wr] =execFunc(temper[tid+shft_rd], temper[tid+shft_rd], temper[tid1+shft_rd]);
                }
                else if (tid1 == height)
                {
                    temper[tid1 + shft_wr] = execFunc(temper[tid2+shft_rd], temper[tid2+shft_rd], temper[tid1+shft_rd]);
                }
                else
                {
                    temper[tid1 + shft_wr] = execFunc(temper[tid+shft_rd], temper[tid2+shft_rd], temper[tid1+shft_rd]);
                }
            }

        }

        __syncthreads();
    }

    temper[tid] = temper[tid1];

    //-------------------TOP PART------------------------------------------
    leftidx = tid/2 + ((tid/2 & 1) * blockDim.x) + (tid & 1);
    rightidx = (blockDim.x - 2) + ((tid/2 & 1) * blockDim.x) + (tid & 1) -  tid/2;

    int tidm = tid - 1;

    //The initial conditions are timslice 0 so start k at 1.

	for (int k = 1; k<(height-1); k++)
	{
		//Bitwise even odd. On even iterations write to first row.
		shft_wr = blockDim.x * (k & 1);
		//On even iterations write to second row (starts at element 32)
		shft_rd = blockDim.x * ((k + 1) & 1);

		//Each iteration the triangle narrows.  When k = 1, 30 points are
		//computed, k = 2, 28 points.
        if (blockIdx.x > 0)
        {
            if (tid < (blockDim.x-k) && tid >= k)
    		{
    			temper[tid + shft_wr] = execFunc(temper[tidm + shft_rd], temper[tid1 + shft_rd], temper[tid + shft_rd]);
    		}
        }
        else
        {
            if (tid < (blockDim.x-k) && tid >= k)
            {
                if (tid == (height - 2))
                {
                    temper[tid + shft_wr] = execFunc(temper[tidm + shft_rd], temper[tidm + shft_rd], temper[tid + shft_rd]);
                }
                else if (tid == (height - 1))
                {
                    temper[tid + shft_wr] = execFunc(temper[tid1 + shft_rd], temper[tid1 + shft_rd], temper[tid + shft_rd]);
                }
                else
                {
                    temper[tid + shft_wr] = execFunc(temper[tidm + shft_rd], temper[tid1 + shft_rd], temper[tid + shft_rd]);
                }
            }
        }

		//Make sure the threads are synced
		__syncthreads();
    }

    //After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];
}

//Do the split diamond on the CPU?
// What's the idea?  Say malloc the pointers in the wrapper.
// Calculate left and right idxs in wrapper too, why continually recalculate.
//

// __host__
// void
// CPU_diamond(REALfour *temper, int tpb)
// {
//     int bck, fwd, shft_rd, shft_wr;
//     int base = tpb + 2;
//     int ht = tpb/2;
//
//     //Splitting it is the whole point!
//     for (int k = ht; k>0; k--)
//     {
//         // This tells you if the current row is the first or second.
//         shft_wr = base * ((k+1) & 1);
//         // Read and write are opposite rows.
//         shft_rd = base * (k & 1);
//
//         for(int n = k; n<(base-k); n++)
//         {
//             bck = n - 1;
//             fwd = n + 1;
//             //Double trailing index.
//             if(n == ht)
//             {
//                 temper[n + shft_wr] = execFunc(temper[bck+shft_rd], temper[bck+shft_rd], temper[n+shft_rd]);
//             }
//             //Double leading index.
//             else if(n == ht+1)
//             {
//                 temper[n + shft_wr] = execFunc(temper[fwd+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
//             }
//             else
//             {
//                 temper[n + shft_wr] = execFunc(temper[bck+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
//             }
//         }
//     }
//
//     for (int k = 0; k<tpb; k++) temper[k] = temper[k+1];
//     //Top part.
//     for (int k = 1; k>ht; k++)
//     {
//         // This tells you if the current row is the first or second.
//         shft_wr = base * (k & 1);
//         // Read and write are opposite rows.
//         shft_rd = base * ((k+1) & 1);
//
//         for(int n = k; n<(tpb-k); n++)
//         {
//             bck = n - 1;
//             fwd = n + 1;
//             //Double trailing index.
//             if(n == ht)
//             {
//                 temper[n + shft_wr] = execFunc(temper[bck+shft_rd], temper[bck+shft_rd], temper[n+shft_rd]);
//             }
//             //Double leading index.
//             else if(n == ht+1)
//             {
//                 temper[n + shft_wr] = execFunc(temper[fwd+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
//             }
//             else
//             {
//                 temper[n + shft_wr] = execFunc(temper[bck+shft_rd], temper[fwd+shft_rd], temper[n+shft_rd]);
//             }
//         }
//     }
// }
//
// //The host routine.
double
sweptWrapper(const int bks, int tpb, const int dv, REAL dt, const int t_end,
    const int cpu, REALfour *IC, REALfour *T_f)
{

    const size_t smem1 = 2*tpb*sizeof(REALfour);
    const size_t smem2 = (2*tpb+8)*sizeof(REALfour);

    int indices[4][tpb];
    for (int k = 0; k<tpb; k++)
    {
        indices[0][k] = k/2 + ((k/2 & 1) * tpb) + (k & 1);
        indices[1][k] = (tpb - 2) + ((k/2 & 1) * tpb) + (k & 1) -  k/2;
        indices[2][k] = k/2 + ((k/2 & 1) * tpb) + (k & 1);
        indices[3][k] = (tpb - 1) + ((k/2 & 1) * tpb) + (k & 1) -  k/2;
    }

    REALfour *tmpr;
    tmpr = (REALfour*)malloc(smem2);
	REALfour *d_IC, *d_right, *d_left;
    REALfour right[tpb], left[tpb];

	hipMalloc((void **)&d_IC, sizeof(REALfour)*dv);
	hipMalloc((void **)&d_right, sizeof(REALfour)*dv);
	hipMalloc((void **)&d_left, sizeof(REALfour)*dv);

	// Copy the initial conditions to the device array.
	hipMemcpy(d_IC,IC,sizeof(REALfour)*dv,hipMemcpyHostToDevice);
	// Start the counter and start the clock.
	const double t_fullstep = dt*(double)tpb;

	upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);

    double t_eq;

	// Call the kernels until you reach the iteration limit.
    // Done now juse use streams or omp to optimize.

    // if (cpu)
    // {
    //     t_eq = t_fullstep/2;
    //     omp_set_num_threads( 2 );
    //
    // 	while(t_eq < t_end)
    // 	{
    //
    //         #pragma omp parallel sections
    //         {
    //         #pragma omp section
    //         {
    //             hipMemcpy(right,d_left,tpb*sizeof(REAL),hipMemcpyDeviceToHost);
    //             hipMemcpy(left,d_right+dv-tpb,tpb*sizeof(REAL),hipMemcpyDeviceToHost);
    //
    //             for (int k = 0; k<tpb; k++)
    //             {
    //                 tmpr[indices[0][k]] = right[k];
    //                 tmpr[indices[1][k]] = left[k];
    //             }
    //
    //             CPU_diamond(tmpr, tpb);
    //
    //             for (int k = 0; k<tpb; k++)
    //             {
    //                 right[k] = tmpr[indices[2][k]];
    //                 left[k] = tmpr[indices[3][k]];
    //             }
    //         }
    //         #pragma omp section
    //         {
    //             wholeDiamond <<< bks-1,tpb,smem2 >>>(d_right,d_left,false);
    //             hipMemcpy(d_right, right, tpb*sizeof(REAL), hipMemcpyHostToDevice);
    //             hipMemcpy(d_left, left, tpb*sizeof(REAL), hipMemcpyHostToDevice);
    //         }
    //         }
    //
    //         wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,true);
    //
	// 	    //So it always ends on a left pass since the down triangle is a right pass.
    //
	// 	    t_eq += t_fullstep;
    //
    // 		/* Since the procedure does not store the temperature values, the user
    // 		could input some time interval for which they want the temperature
    // 		values and this loop could copy the values over from the device and
    // 		write them out.  This way the user could see the progression of the
    // 		solution over time, identify an area to be investigated and re-run a
    // 		shorter version of the simulation starting with those intiial conditions.
    //
    // 		-------------------------------------
    // 	 	if (true)
    // 		{
    // 			downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);
    // 			hipMemcpy(T_final, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);
    // 			fwr << t_eq << " ";
    //
    // 			for (int k = 0; k<dv; k++)
    // 			{
    // 					fwr << T_final.x[k] << " ";
    // 			}
    // 				fwr << endl;
    //
    // 			upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);
    // 			wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,-1);
    // 		}
    // 		-------------------------------------
    // 		*/
    //     }
	// }
    // else
    // {
        splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);
        t_eq = t_fullstep;

        while(t_eq < t_end)
        {

            wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,true);

            splitDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);
            //So it always ends on a left pass since the down triangle is a right pass.

            t_eq += t_fullstep;

            /*
            if (true)
            {
                downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);
                hipMemcpy(T_final, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);
                fwr << t_eq << " ";

                for (int k = 0; k<dv; k++)
                {
                        fwr << T_final.x[k] << " ";
                }
                    fwr << endl;

                upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);
                wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left,-1);
            }
            -------------------------------------
            */
        }
    //}

	downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);

	hipMemcpy(T_f, d_IC, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);

	hipFree(d_IC);
	hipFree(d_right);
	hipFree(d_left);

    return t_eq;
}

int main( int argc, char *argv[] )
{
    using namespace std;
	if (argc != 6)
	{
		cout << "The Program takes five inputs: #Divisions, #Threads/block, dt, finish time, and GPU/CPU or all GPU" << endl;
		exit(-1);
	}
	// Choose the GPGPU.  This is device 0 in my machine which has 2 devices.
	hipSetDevice(0);

    REALfour bd[2];
    bd[0].x = 1.0; //Density
    bd[1].x = 0.125;
    bd[0].y = 0.0; //Velocity
    bd[1].y = 0.0;
    bd[0].w = 1.0; //Pressure
    bd[1].w = 0.1;
    bd[0].z = bd[0].w/m_gamma; //Energy
    bd[1].z = bd[1].w/m_gamma;

    //Declare the dimensions in constant memory.

    const REAL dt = atof(argv[3]);
    const int dv = atoi(argv[1]); //Number of spatial points
	const int tpb = atoi(argv[2]); //Threads per Blocks
	const float tf = atof(argv[4]); //Finish time
	const int bks = dv/tpb; //The number of blocks
	const int tst = atoi(argv[5]);
    REAL lx = dx*((float)dv-1.f);

    REALthree dimz;
    dimz.x = dt/dx; // dt/dx
    dimz.y = gam; dimz.z = m_gamma;

	//Conditions for main input.  Unit testing kinda.
	//dv and tpb must be powers of two.  dv must be larger than tpb and divisible by
	//tpb.

	if ((dv & (tpb-1) !=0) || (tpb&31) != 0)
    {
        cout << "INVALID NUMERIC INPUT!! "<< endl;
        cout << "2nd ARGUMENT MUST BE A POWER OF TWO >= 32 AND FIRST ARGUMENT MUST BE DIVISIBLE BY SECOND" << endl;
        exit(-1);
    }

	// Initialize arrays.
    REALfour *IC = (REALfour*)malloc(dv*sizeof(float4));
	REALfour *T_final = (REALfour*)malloc(dv*sizeof(float4));

	// Some initial condition for the bar temperature, an exponential decay
	// function.
	for (int k = 0; k<dv; k++)
	{
        if (k<dv/2)
        {
            IC[k] = bd[0];
        }
        else
        {
            IC[k] = bd[1];
        }
	}

	// Call out the file before the loop and write out the initial condition.
	ofstream fwr, ftime;
	fwr.open("Results/Euler1D_Result.dat",ios::trunc);
	ftime.open("Results/Euler1D_Timing.txt",ios::app);
	// Write out x length and then delta x and then delta t.
	// First item of each line is timestamp.
	fwr << lx << " " << dv << " " << dx << " " << endl << 0 << " ";

	for (int k = 0; k<dv; k++)
	{
		fwr << IC[k].x << " ";

	}

	fwr << endl;

    //Transfer data to GPU.
	// This puts the Fourier number in constant memory.
	hipMemcpyToSymbol(HIP_SYMBOL(dimens),&dimz,sizeof(REALthree));
    hipMemcpyToSymbol(HIP_SYMBOL(dbd),&bd,2*sizeof(REALfour));

	// This initializes the device arrays on the device in global memory.
	// They're all the same size.  Conveniently.

	// Start the counter and start the clock.
	hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

    // Call the kernels until you reach the iteration limit.
	double tfm;

    //--------TEST-----------

     REALfour *d_IC, *d_temp;

    hipMalloc((void **)&d_IC, sizeof(REALfour)*dv);
	hipMalloc((void **)&d_temp, sizeof(REALfour)*dv);

	// Copy the initial conditions to the device array.
	hipMemcpy(d_IC,IC,sizeof(REALfour)*dv,hipMemcpyHostToDevice);

    tfm = 0.0;

    while (tfm < tf)
    {

        classicDisc <<< bks,tpb >>> (d_IC,d_temp);
        tfm += dt;

    }

    hipMemcpy(T_final, d_IC, sizeof(REALfour)*dv, hipMemcpyDeviceToHost);
    hipFree(d_IC);
    hipFree(d_temp);

    //--------TEST-----------

	//tfm = sweptWrapper(bks,tpb,dv,dt,tf,tst,IC,T_final);

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime( &timed, start, stop);

	timed = timed * 1.e-3;

	cout << "That took: " << timed << " seconds" << endl;

	ftime << dv << " " << tpb << " " << timed << endl;

	ftime.close();

	fwr << tfm << " ";
	for (int k = 0; k<dv; k++)
	{
		fwr << T_final[k].x << " ";
	}

    fwr << endl;

    fwr << tfm << " ";
	for (int k = 0; k<dv; k++)
	{
		fwr << T_final[k].w << " ";
	}


	fwr.close();

	// Free the memory and reset the device.
	hipDeviceSynchronize();

	hipEventDestroy( start );
	hipEventDestroy( stop );
    hipDeviceReset();
    free(IC);
    free(T_final);

	return 0;

}
