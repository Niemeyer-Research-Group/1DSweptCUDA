#include "hip/hip_runtime.h"
/* This file is the current iteration of research being done to implement the
swept rule for Partial differential equations in one dimension.  This research
is a collaborative effort between teams at MIT, Oregon State University, and
Purdue University.

Copyright (C) 2015 Kyle Niemeyer, niemeyek@oregonstate.edu AND
Daniel Magee, mageed@oregonstate.edu

This program is free software: you can redistribute it and/or modify
it under the terms of the MIT license.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.

You should have received a copy of the MIT license
along with this program.  If not, see <https://opensource.org/licenses/MIT>.
*/

//COMPILE LINE!
// nvcc -o ./bin/KSOut KS1D_SweptShared.cu -gencode arch=compute_35,code=sm_35 -lm -restrict -Xcompiler -fopenmp --ptxas-options=-v

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

#include <iostream>
#include <ostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>


#ifndef REAL
#define REAL  float
#endif

using namespace std;

const REAL dx = 0.5;

struct discConstants{

	REAL dx;
	REAL dx2;
	REAL dx4;
	REAL dt;
	REAL dt_half;
};

__constant__ discConstants disc;

__host__
REAL initFun(REAL xnode)
{
	return 2.0 * cos(19.0*xnode*M_PI/128.0);
}

__device__
__forceinline__
REAL fourthDer(REAL tfarLeft, REAL tLeft, REAL tCenter, REAL tRight, REAL tfarRight)
{
	return (tfarLeft - 4.f*tLeft + 6.f*tCenter - 4.f*tRight + tfarRight)/(disc.dx4);
}

__device__
__forceinline__
REAL secondDer(REAL tLeft, REAL tRight, REAL tCenter)
{
	return (tLeft + tRight - 2.f*tCenter)/(disc.dx2);
}

__device__
__forceinline__
REAL convect(REAL tLeft, REAL tRight)
{
	return (tRight*tRight - tLeft*tLeft)/(4.f*disc.dx);
}

__device__
REAL stutterStep(REAL tfarLeft, REAL tLeft, REAL tCenter, REAL tRight, REAL tfarRight)
{
	return tCenter - disc.dt_half * (convect(tLeft, tRight) + secondDer(tLeft, tRight, tCenter) +
		fourthDer(tfarLeft, tLeft, tCenter, tRight, tfarRight));
}

__device__
REAL finalStep(REAL tfarLeft, REAL tLeft, REAL tCenter, REAL tRight, REAL tfarRight)
{
	return (-disc.dt * (convect(tLeft, tRight) + secondDer(tLeft, tRight, tCenter) +
			fourthDer(tfarLeft, tLeft, tCenter, tRight, tfarRight)));
}

__global__
void
swapKernel(const REAL *passing_side, REAL *bin, int direction)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
    int lastidx = ((blockDim.x*gridDim.x)-1);
    int gidout = (gid + direction*blockDim.x) & lastidx;

    bin[gidout] = passing_side[gid];

}

//Classic
__global__
void
classicKS(const REAL *ks_in, REAL *ks_out, bool final)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
    int lastidx = ((blockDim.x*gridDim.x)-1);

	if (final)
	{
		ks_out[gid] += finalStep(ks_in[(gid-2)&lastidx],ks_in[(gid-1)&lastidx],ks_in[gid],ks_in[(gid+1)&lastidx],ks_in[(gid+2)&lastidx]);
	}
	else
	{
		ks_out[gid] = stutterStep(ks_in[(gid-2)&lastidx],ks_in[(gid-1)&lastidx],ks_in[gid],ks_in[(gid+1)&lastidx],ks_in[(gid+2)&lastidx]);
	}
}

__global__
void
upTriangle(const REAL *IC, REAL *right, REAL *left)
{
	extern __shared__ REAL temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tid = threadIdx.x; //Block Thread ID

	int tid_top = tid + blockDim.x;

	int leftidx = ((tid/4 & 1) * blockDim.x) + (tid/4)*2 + (tid & 3);
	int rightidx = (blockDim.x - 4) + ((tid/4 & 1) * blockDim.x) + (tid & 3) - (tid/4)*2;

	int step2;

    //Assign the initial values to the first row in temper, each block
    //has it's own version of temper shared among its threads.
	temper[tid] = IC[gid];

	__syncthreads();

	if (tid > 1 && tid <(blockDim.x-2))
	{
		temper[tid_top] = stutterStep(temper[tid - 2], temper[tid - 1], temper[tid],
			temper[tid + 1], temper[tid + 2]);
	}

	__syncthreads();

	//The initial conditions are timslice 0 so start k at 1.
	for (int k = 4; k<(blockDim.x/2); k+=4)
	{
		if (tid < (blockDim.x-k) && tid >= k)
		{
			temper[tid] += finalStep(temper[tid_top - 2], temper[tid_top - 1], temper[tid_top],
				temper[tid_top + 1], temper[tid_top + 2]);
		}

		step2 = k + 2;
		__syncthreads();

		if (tid < (blockDim.x-step2) && tid >= step2)
		{
			temper[tid_top] = stutterStep(temper[tid - 2], temper[tid - 1], temper[tid],
				temper[tid + 1], temper[tid + 2]);
		}

		//Make sure the threads are synced
		__syncthreads();

	}
	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];
}

__global__
void
downTriangle(REAL *IC, const REAL *right, const REAL *left)
{
	extern __shared__ REAL temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tididx = tid + 2;
	int base = blockDim.x + 4;
	int height = base/2;
	int step2;

	int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tididx + k + base;
		tid_bottom[k+2] = tididx + k;
	}

	int leftidx = height + ((tid/4 & 1) * base) + (tid & 3) - (4 + (tid/4) * 2);
	int rightidx = height + ((tid/4 & 1) * base) + (tid/4)*2 + (tid & 3);

	temper[leftidx] = right[gid];
	temper[rightidx] = left[gid];

	__syncthreads();

	for (int k = (height-2); k>0; k-=4)
	{
		if (tididx < (base-k) && tididx >= k)
		{
			temper[tid_top[2]] = stutterStep(temper[tid_bottom[0]], temper[tid_bottom[1]], temper[tid_bottom[2]],
				temper[tid_bottom[3]], temper[tid_bottom[4]]);

		}

		step2 = k-2;
		__syncthreads();

		if (tididx < (base-step2) && tididx >= step2)
		{
			temper[tididx] += finalStep(temper[tid_top[0]], temper[tid_top[1]], temper[tid_top[2]],
				temper[tid_top[3]], temper[tid_top[4]]);
		}

		//Make sure the threads are synced
		__syncthreads();
	}

    IC[gid] = temper[tididx];
}


__global__
void
wholeDiamond(REAL *right, REAL *left)
{
	extern __shared__ REAL temper[];

	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int tididx = tid + 2;
	int base = blockDim.x + 4;
	int height = base/2;
	int step2;

	int tid_top[5], tid_bottom[5];
	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tididx + k + base;
		tid_bottom[k+2] = tididx + k;
	}

	int leftidx = height + ((tid/4 & 1) * base) + (tid & 3) - (4 + (tid/4) * 2);
	int rightidx = height + ((tid/4 & 1) * base) + (tid/4)*2 + (tid & 3);

	temper[leftidx] = right[gid];
	temper[rightidx] = left[gid];

	__syncthreads();

	for (int k = (height-2); k>0; k-=4)
	{
		if (tididx < (base-k) && tididx >= k)
		{
			temper[tid_top[2]] = stutterStep(temper[tid_bottom[0]], temper[tid_bottom[1]], temper[tid_bottom[2]],
				temper[tid_bottom[3]], temper[tid_bottom[4]]);
		}

		step2 = k-2;
		__syncthreads();

		if (tididx < (base-step2) && tididx >= step2)
		{
			temper[tididx] += finalStep(temper[tid_top[0]], temper[tid_top[1]], temper[tid_top[2]],
				temper[tid_top[3]], temper[tid_top[4]]);
		}

		//Make sure the threads are synced
		__syncthreads();
	}

	//Shift the last row to justify it at 0.
	REAL trade = temper[tididx];
	__syncthreads();
	temper[tid] = trade;
	__syncthreads();
    //-------------------TOP PART------------------------------------------

	leftidx = ((tid/4 & 1) * blockDim.x) + (tid/4)*2 + (tid & 3);
	rightidx = (blockDim.x - 4) + ((tid/4 & 1) * blockDim.x) + (tid & 3) - (tid/4)*2;

	#pragma unroll
	for (int k = -2; k<3; k++)
	{
		tid_top[k+2] = tid + k + blockDim.x;
		tid_bottom[k+2] = tid + k;
	}

	__syncthreads();

	if (tid > 1 && tid <(blockDim.x-2))
	{
		temper[tid_top[2]] = stutterStep(temper[tid_bottom[0]], temper[tid_bottom[1]], temper[tid_bottom[2]],
			temper[tid_bottom[3]], temper[tid_bottom[4]]);
	}

	__syncthreads();

	//The initial conditions are timslice 0 so start k at 1.
	for (int k = 4; k<(blockDim.x/2); k+=4)
	{
		if (tid < (blockDim.x-k) && tid >= k)
		{
			temper[tid] += finalStep(temper[tid_top[0]], temper[tid_top[1]], temper[tid_top[2]],
				temper[tid_top[3]], temper[tid_top[4]]);
		}

		step2 = k+2;
		__syncthreads();

		if (tid < (blockDim.x-step2) && tid >= step2)
		{
			temper[tid_top[2]] = stutterStep(temper[tid_bottom[0]], temper[tid_bottom[1]], temper[tid_bottom[2]],
				temper[tid_bottom[3]], temper[tid_bottom[4]]);
		}

		//Make sure the threads are synced
		__syncthreads();

	}

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = temper[rightidx];
	left[gid] = temper[leftidx];

}

double
classicWrapper(const int bks, int tpb, const int dv, const REAL dt, const REAL t_end,
    REAL *IC, REAL *T_f, const REAL freq, ofstream &fwr)
{
    REAL *dks_in, *dks_out, *dks_orig;

    hipMalloc((void **)&dks_in, sizeof(REAL)*dv);
    hipMalloc((void **)&dks_out, sizeof(REAL)*dv);

    // Copy the initial conditions to the device array.
    hipMemcpy(dks_in,IC,sizeof(REAL)*dv,hipMemcpyHostToDevice);

    double t_eq = 0.0;
    double twrite = freq;

    while (t_eq < t_end)
    {
        classicKS <<< bks,tpb >>> (dks_in, dks_out, false)
        classicKS <<< bks,tpb >>> (dks_out, dks_in, true)
        t_eq += dt;

        if (t_eq > twrite)
        {
            hipMemcpy(T_f, dks_in, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

			fwr << " Velocity " << t_eq << " ";
            for (int k = 0; k<dv; k++)
            {
                fwr << T_f[k] << " ";
            }
            fwr << endl;

            twrite += freq;
        }
    }

    hipMemcpy(T_f, dks_in, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

    hipFree(dks_in);
    hipFree(dks_out);

    return t_eq;
}

//The host routine.
double
sweptWrapper(const int bks, int tpb, const int dv, REAL dt, const REAL t_end,
	REAL *IC, REAL *T_f, const REAL freq, ofstream &fwr)
{

	REAL *d_IC, *d_right, *d_left, *d_bin;
	hipMalloc((void **)&d_IC, sizeof(REAL)*dv);
	hipMalloc((void **)&d_right, sizeof(REAL)*dv);
	hipMalloc((void **)&d_left, sizeof(REAL)*dv);
	hipMalloc((void **)&d_bin, sizeof(REAL)*dv);

	// Copy the initial conditions to the device array.
	hipMemcpy(d_IC,IC,sizeof(REAL)*dv,hipMemcpyHostToDevice);
	//Start the counter and start the clock.
	//
	//Every other step is a full timestep and each cycle is half tpb steps.
	const double t_fullstep = 0.25 * dt * (double)tpb;
	double twrite = freq;

	const size_t smem1 = 2*tpb*sizeof(REAL);
	const size_t smem2 = (2*tpb+8)*sizeof(REAL);

	upTriangle <<< bks,tpb,smem1 >>> (d_IC,d_right,d_left);

	swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
	swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

	//Split
	wholeDiamond <<< bks,tpb,smem2 >>> (d_right,d_left);

	swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
	swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

	double t_eq = t_fullstep;

	// Call the kernels until you reach the iteration limit.
	while(t_eq < t_end)
	{

		wholeDiamond <<< bks,tpb,smem2 >>> (d_right,d_left);

		swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
		swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

		//So it always ends on a left pass since the down triangle is a right pass.

		//Split
		wholeDiamond <<< bks,tpb,smem2 >>> (d_right,d_left);

		swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
		swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

		t_eq += t_fullstep;


	 	if (t_eq > twrite)
		{
			downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);

			hipMemcpy(T_f, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

			fwr << " Velocity " << t_eq << " ";

			for (int k = 0; k<dv; k++)	fwr << T_f[k] << " ";

			fwr << endl;

			upTriangle <<< bks,tpb,smem1 >>>(d_IC,d_right,d_left);

			swapKernel <<< bks,tpb >>> (d_right, d_bin, 1);
			swapKernel <<< bks,tpb >>> (d_bin, d_right, 0);

			//Split
			wholeDiamond <<< bks,tpb,smem2 >>>(d_right,d_left);

			swapKernel <<< bks,tpb >>> (d_left, d_bin, -1);
			swapKernel <<< bks,tpb >>> (d_bin, d_left, 0);

			t_eq += t_fullstep;

			twrite += freq;
		}

	}

	downTriangle <<< bks,tpb,smem2 >>>(d_IC,d_right,d_left);

	hipMemcpy(T_f, d_IC, sizeof(REAL)*dv, hipMemcpyDeviceToHost);

	hipFree(d_IC);
	hipFree(d_right);
	hipFree(d_left);
	hipFree(d_bin);

	return t_eq;

}

int main( int argc, char *argv[])
{

	if (argc < 9)
	{
		cout << "The Program takes 9 inputs, #Divisions, #Threads/block, deltat, finish time, output frequency..." << endl;
        cout << "Classic/Swept, CPU sharing Y/N (Ignored), Variable Output File, Timing Output File (optional)" << endl;
		exit(-1);
	}

	// Choose the GPGPU.  This is device 0 in my machine which has 2 devices.
	hipSetDevice(0);

	const int dv = atoi(argv[1]); //Number of spatial points
	const int tpb = atoi(argv[2]); //Threads per Block
    const REAL dt = atof(argv[3]); //delta T timestep
	const float tf = atof(argv[4]); //Finish time
    const float freq = atof(argv[5]); //Output frequency
    const int scheme = atoi(argv[6]); //1 for Swept 0 for classic
    // const int tst = atoi(argv[7]); CPU/GPU share
    const int bks = dv/tpb; //The number of blocks
	const float lx = dv*dx;
	char const *prec;
	prec = (sizeof(REAL)<6) ? "Single": "Double";

	cout << "KS --- #Blocks: " << bks << " | Length: " << lx << " | Precision: " << prec << " | dt/dx: " << dt/dx << endl;

	//Conditions for main input.  Unit testing kinda.
	//dv and tpb must be powers of two.  dv must be larger than tpb and divisible by
	//tpb.

	if ((dv & (tpb-1) !=0) || (tpb&31) != 0)
    {
        cout << "INVALID NUMERIC INPUT!! "<< endl;
        cout << "2nd ARGUMENT MUST BE A POWER OF TWO >= 32 AND FIRST ARGUMENT MUST BE DIVISIBLE BY SECOND" << endl;
        exit(-1);
    }

	discConstants dsc = {
		dx, //dx
		dx*dx, //dx^2
		dx*dx*dx*dx, //dx^4
		dt, //dt
		dt*.05, //dt half
	};

	// Initialize arrays.
    REAL *IC, *T_final;

	hipHostAlloc((void **) &IC, dv*sizeof(REAL), hipHostMallocDefault);
	hipHostAlloc((void **) &T_final, dv*sizeof(REAL), hipHostMallocDefault);

    // IC = (REAL *) malloc(dv*sizeof(REAL));
    // T_final = (REAL *) malloc(dv*sizeof(REAL));

	// Inital condition
	for (int k = 0; k<dv; k++)
	{
		IC[k] = initFun((float)k*dsc.dx);
	}

	// Call out the file before the loop and write out the initial condition.
	ofstream fwr;
	fwr.open(argv[8],ios::trunc);

	// Write out x length and then delta x and then delta t.
	// First item of each line is timestamp.
	fwr << lx << " " << dv << " " << dsc.dx << " " << endl << " Velocity " << 0 << " ";

	for (int k = 0; k<dv; k++) fwr << IC[k] << " ";

	fwr << endl;
	// Transfer data to GPU.

	// This puts the constant part of the equation in constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(disc),&dsc,sizeof(dsc));

	// Start the counter and start the clock.
	hipEvent_t start, stop;
	float timed;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0);

	// Call the kernels until you reach the iteration limit.
	double tfm;
	if (scheme)
    {
		tfm = sweptWrapper(bks, tpb, dv, dsc.dt, tf, IC, T_final, freq, fwr);
	}
	else
	{
		tfm = classicWrapper(bks, tpb, dv, dsc.dt, tf, IC, T_final, freq, fwr);
	}

	// Show the time and write out the final condition.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime( &timed, start, stop);

	timed *= 1.e3;

	double n_timesteps = tfm/dt;

    double per_ts = timed/n_timesteps;

    cout << n_timesteps << " timesteps" << endl;
	cout << "Averaged " << per_ts << " microseconds (us) per timestep" << endl;

    if (argc>8)
    {
        ofstream ftime;
        ftime.open(argv[9],ios::app);
    	ftime << dv << "\t" << tpb << "\t" << per_ts << endl;
    	ftime.close();
    }

	fwr << " Velocity " << tfm << " ";
	for (int k = 0; k<dv; k++) fwr << T_final[k] << " ";

    fwr << endl;

	fwr.close();

	hipDeviceSynchronize();
	// Free the memory and reset the device.

	hipEventDestroy( start );
	hipEventDestroy( stop );
	hipDeviceReset();

	hipHostFree(IC);
    hipHostFree(T_final);
	// free(IC);
	// free(T_final);

	return 0;

}
