#include "hip/hip_runtime.h"
//Now how to link function files.
//Somewhat importantly.  This can only be done with 32 threads in a block.
//Well you could launch more but each triangle is 32.

__global__ void upTriangle(REAL *IC, REAL *right, REAL *left)
{

	REAL temper;
	__shared__ REAL sR[THREADBLK];
	__shared__ REAL sL[THREADBLK];

	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int tid = threadIdx.x % 32; //Warp or node ID.  Fix this.

	temper = IC[gid];

	// There's two ways to do it.  Either just initialize the shared arrays with
	// the first row do it all at the end and make temper be THREADBLK/2 long.
	// Since the warp size can't vary all the indices can be hardwired.
	if (tid<2)
	{
		sL[tid] = temper;
		sR[tid] = __shfl(temper,30+tid);
	}

	//The initial conditions are timslice 0 so start k at 1.
	#pragma unroll
	for (int k = 1; k<16; k++)
	{
		temper = fo * (__shfl_down(temper,1) + __shfl_up(temper,1)) + (1.-2.*fo) * temper;
		//Maybe it works.
		if (tid < 2)
		{
			sL[tid+(2*k)] = __shfl_down(temper,k);
			sR[tid+(2*k)] = __shfl_down(temper,(THREADBLK-1)-k);
		}
		__syncthreads();
	}

	//After the triangle has been computed, the right and left shared arrays are
	//stored in global memory by the global thread ID since (conveniently),
	//they're the same size as a warp!
	right[gid] = sR[tid];
	left[gid] = sL[tid];

}

//The upside down triangle.  This function essentially takes right and left and
//returns IC.

//IMPORTANT note: k and tid were in sync in the first function, but here they're
//out of sync in the loop.  This is because we can't use tid = 33 or 32 and the
//shared temperature array is that long.  BUT in order to fill the arrays, these
//elements must be accessed.  So each element in each row is shifted by +1.
//For instance, thread tid = 16 refers to temper[17].  That being said, tid is
//unique and k is NOT so the index must be referenced by tid.

__global__ void downTriangle(REAL *IC, REAL *right, REAL *left)
{

	//Now temper needs to accommodate a longer row by 2, one on each side.
	//since it has two rows that's 4 extra floats.  The last row will still be
	//32 numbers long.
	__shared__ REAL temper[68];
	__shared__ REAL sR[32];
	__shared__ REAL sL[32];

	//Same as upTriangle
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int shft_rd;
	int shft_wr;

	// Pass to the left so all checks are for block 0 (this reduces arithmetic).
	// The left ridge is always kept by the block.
	sR[tid] = left[gid];

	// The right ridge is passed, each block 1-end gets the right of 0-end-1
	// Block 0 gets the right of the last block.
	if (blockIdx.x > 0)
	{
		sL[tid] = right[gid-blockDim.x];
	}
	else
	{
		sL[tid] = right[blockDim.x*(gridDim.x-1) + tid];
	}

	__syncthreads();

	// Initialize temper. Kind of an unrolled for loop.  This is actually at
	// Timestep 0.
	temper[15] = sL[0];
	temper[16] = sL[1];
	temper[17] = sR[0];
	temper[18] = sR[1];

	//Now we need two counters since we need to use sL and sR EVERY iteration
	//instead of every other iteration and instead of growing smaller with every
	//iteration this grows larger.
	int itr = 2;
	int itr2 = 18;
	//k needs to insert the relevant left right values around the computed values
	//every timestep.  Since it grows larger the loop is reversed.
	for (int k = 17; k>1; k--)
	{
		// This tells you if the current row is the first or second.
		shft_wr = (k & 1);
		// Read and write are opposite rows.
		shft_rd = 34*((shft_wr+1) & 1);

		//Block 0 is split so it needs a different algorithm.  This algorithm
		//is slightly different than top triangle as described in the note above.
		if (blockIdx.x > 0)
		{
			if (tid <= (33-k) && tid >= (k-2))
			{
				temper[tid + 1 + (34*shft_wr)] = fo * (temper[tid+shft_rd] + temper[tid+shft_rd+2]) + (1-2.*fo) * temper[tid+shft_rd+1];
			}

		}

		//Split part.  This exhibits thread divergence and is suboptimal.
		//So it's ripe to be improved.

		else
		{
			if (tid <= (33-k) && tid >= (k-2))
			{
				if (tid == 15)
				{
					temper[tid + 1 + (34*shft_wr)] = 2. * fo * (temper[tid+shft_rd]-temper[tid+shft_rd+1]) + temper[tid+shft_rd+1];
				}
				else if (tid == 16)
				{
					temper[tid + 1 + (34*shft_wr)] = 2. * fo * (temper[tid+shft_rd+2]-temper[tid+shft_rd+1]) + temper[tid+shft_rd+1];
				}
				else
				{
					temper[tid + 1 + (34*shft_wr)] = fo * (temper[tid+shft_rd] + temper[tid+shft_rd+2]) + (1-2.*fo) * temper[tid+shft_rd+1];
				}
			}

		}

		//Fill edges.  Thread 0 never gets used for both operations so the
		//calculation and the filling are conceptually coincident.
		//Threads are synced afterward here because the next timestep is
		//reliant on the entire loop.
		if (k>2 && tid == 0)
		{
			temper[(k-3)+(34*shft_wr)] = sL[itr];
			temper[(k-2)+(34*shft_wr)] = sL[itr+1];
			temper[itr2+(34*shft_wr)] = sR[itr];
			itr2++;
			temper[itr2+(34*shft_wr)] = sR[itr+1];
			itr+=2;

		}
		__syncthreads();

	}

	//Now fill the global unified timestep variable with the final calculated
	//temperatures.

	//Blocks 1 to end hold values 16 to end-16.
	if (blockIdx.x > 0)
	{
		//True if it ends on the first row! The first and last of temper on the final row are empty.
		IC[gid - 16] = temper[tid+1];
	}
	//Block 0 holds values 0 to 15 and end-15 to end.  In that order.
	else
	{
		if (tid>15)
		{
			IC[gid - 16] = temper[tid+1];
		}
		else
		{
			IC[(blockDim.x * gridDim.x) + (tid - 16) ] = temper[tid+1];
		}
	}
}
